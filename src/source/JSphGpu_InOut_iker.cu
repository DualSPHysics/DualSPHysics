#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2020 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or (at your option) any later version. 

 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details. 

 You should have received a copy of the GNU General Public License, along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JSphGpu_InOut_iker.cu \brief Implements functions and CUDA kernels for InOut feature.

#include "JSphGpu_InOut_iker.h"
#include "Functions.h"
#include "FunctionsCuda.h"
#include <cfloat>
#include <hip/hip_math_constants.h>

namespace cusphinout{
#include "FunctionsBasic_iker.h"
#include "FunctionsMath_iker.h"
#include "FunSphKernel_iker.h"
#include "FunctionsGeo3d_iker.h"

#undef _JCellSearch_iker_
#include "JCellSearch_iker.h"

//##############################################################################
//# Kernels for inlet/outlet (JSphInOut).
//# Kernels para inlet/outlet (JSphInOut).
//##############################################################################

//------------------------------------------------------------------------------
/// Mark special fluid particles to ignore.
/// Marca las particulas fluidas especiales para ignorar.
//------------------------------------------------------------------------------
__global__ void KerInOutIgnoreFluidDef(unsigned n,typecode cod,typecode codnew,typecode *code)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(code[p]==cod)code[p]=codnew;
  }
}

//==============================================================================
/// Mark special fluid particles to ignore.
/// Marca las particulas fluidas especiales para ignorar.
//==============================================================================
void InOutIgnoreFluidDef(unsigned n,typecode cod,typecode codnew,typecode *code){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInOutIgnoreFluidDef <<<sgrid,SPHBSIZE>>> (n,cod,codnew,code);
  }
}


//------------------------------------------------------------------------------
/// Returns original position of periodic particle.
//------------------------------------------------------------------------------
__device__ double3 KerInteraction_PosNoPeriodic(double3 posp1)
{
  if(CTE.periactive&1){//-xperi
    if(posp1.x<CTE.maprealposminx)                 { posp1.x-=CTE.xperincx; posp1.y-=CTE.xperincy; posp1.z-=CTE.xperincz; }
    if(posp1.x>CTE.maprealposminx+CTE.maprealsizex){ posp1.x+=CTE.xperincx; posp1.y+=CTE.xperincy; posp1.z+=CTE.xperincz; }
  }
  if(CTE.periactive&2){//-yperi
    if(posp1.y<CTE.maprealposminy)                 { posp1.x-=CTE.yperincx; posp1.y-=CTE.yperincy; posp1.z-=CTE.yperincz; }
    if(posp1.y>CTE.maprealposminy+CTE.maprealsizey){ posp1.x+=CTE.yperincx; posp1.y+=CTE.yperincy; posp1.z+=CTE.yperincz; }
  }
  if(CTE.periactive&4){//-zperi
    if(posp1.z<CTE.maprealposminz)                 { posp1.x-=CTE.zperincx; posp1.y-=CTE.zperincy; posp1.z-=CTE.zperincz; }
    if(posp1.z>CTE.maprealposminz+CTE.maprealsizez){ posp1.x+=CTE.zperincx; posp1.y+=CTE.zperincy; posp1.z+=CTE.zperincz; }
  }
  return(posp1);
}

//------------------------------------------------------------------------------
/// Updates fluid particle position according to current position.
/// Actualizacion de posicion de particulas fluidas segun posicion actual.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerUpdatePosFluid(unsigned n,unsigned pini
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pp<n){
    unsigned p=pp+pini;
    const typecode rcode=code[p];
    const bool outrhop=(CODE_GetSpecialValue(rcode)==CODE_OUTRHOP);
    cusph::KerUpdatePos<periactive>(posxy[p],posz[p],0,0,0,outrhop,p,posxy,posz,dcell,code);
  }
}

//==============================================================================
/// Updates fluid particle position according to current position.
/// Actualizacion de posicion de particulas fluidas segun posicion actual.
//==============================================================================
void UpdatePosFluid(byte periactive,unsigned n,unsigned pini
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    if(periactive)KerUpdatePosFluid<true>  <<<sgrid,SPHBSIZE>>> (n,pini,posxy,posz,dcell,code);
    else          KerUpdatePosFluid<false> <<<sgrid,SPHBSIZE>>> (n,pini,posxy,posz,dcell,code);
  }
}



//------------------------------------------------------------------------------
/// Creates list with current inout particles (normal and periodic).
//------------------------------------------------------------------------------
__global__ void KerInOutCreateListSimple(unsigned n,unsigned pini
  ,const typecode *code,unsigned *listp)
{
  extern __shared__ unsigned slist[];
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned pp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pp<n){
    const unsigned p=pp+pini;
    const typecode rcode=code[p];
    if(CODE_IsNotOut(rcode) && CODE_IsFluidInout(rcode)){//-It includes normal and periodic particles.
      slist[atomicAdd(slist,1)+1]=p; //-Add particle in the list.
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((listp+n),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp=slist[0]+threadIdx.x;
    listp[cp]=slist[threadIdx.x+1];
  }
}
//==============================================================================
/// Creates list with current inout particles (normal and periodic).
/// With stable activated reorders perioc list.
//==============================================================================
unsigned InOutCreateListSimple(bool stable,unsigned n,unsigned pini
  ,const typecode *code,unsigned *listp)
{
  unsigned count=0;
  if(n){
    //-listp size list initialized to zero.
    //-Inicializa tamanho de lista listp a cero.
    hipMemset(listp+n,0,sizeof(unsigned));
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutCreateListSimple <<<sgrid,SPHBSIZE,smem>>> (n,pini,code,listp);
    hipMemcpy(&count,listp+n,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list when stable has been activated.
    //-Reordena lista cuando stable esta activado.
    if(stable && count){ //-Does not affect results.
      thrust::device_ptr<unsigned> dev_list(listp);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}

//------------------------------------------------------------------------------
/// Creates list with current inout particles and normal (no periodic) fluid in 
/// inlet/outlet zones (update its code).
//------------------------------------------------------------------------------
__global__ void KerInOutCreateList(unsigned n,unsigned pini
  ,byte chkinputmask,byte nzone,const byte *cfgzone,const float4 *planes
  ,float3 freemin,float3 freemax
  ,const float2 *boxlimit,const double2 *posxy,const double *posz
  ,typecode *code,unsigned *listp)
{
  extern __shared__ unsigned slist[];
  //float *splanes=(float*)(slist+(n+1));
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned pp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pp<n){
    const unsigned p=pp+pini;
    const typecode rcode=code[p];
    if(CODE_IsNormal(rcode) && CODE_IsFluid(rcode)){//-It includes only normal fluid particles (no periodic).
      bool select=CODE_IsFluidInout(rcode);//-Particles already selected as InOut.
      if(!select){//-Particulas no periodicas y no marcadas como in/out.
        const double2 rxy=posxy[p];
        const double rz=posz[p];
        if(rxy.x<=freemin.x || rxy.y<=freemin.y || rz<=freemin.z || rxy.x>=freemax.x || rxy.y>=freemax.y || rz>=freemax.z){
          byte zone=255;
          if(boxlimit!=NULL){
            for(byte cz=0;cz<nzone && zone==255;cz++)if((cfgzone[cz]&chkinputmask)!=0){
              const float2 xlim=boxlimit[cz];
              const float2 ylim=boxlimit[nzone+cz];
              const float2 zlim=boxlimit[nzone*2+cz];
              if(xlim.x<=rxy.x && rxy.x<=xlim.y && ylim.x<=rxy.y && rxy.y<=ylim.y && zlim.x<=rz && rz<=zlim.y){
                const float4 rpla=planes[cz];
                if((rpla.x*rxy.x+rpla.y*rxy.y+rpla.z*rz+rpla.w)<0)zone=byte(cz);
              }
            }
          }
          else{
            for(byte cz=0;cz<nzone && zone==255;cz++)if((cfgzone[cz]&chkinputmask)!=0){
              const float4 rpla=planes[cz];
              if((rpla.x*rxy.x+rpla.y*rxy.y+rpla.z*rz+rpla.w)<0)zone=byte(cz);
            }        
          }
          if(zone!=255){
            code[p]=CODE_ToFluidInout(rcode,zone)|CODE_TYPE_FLUID_INOUTNUM; //-Adds 16 to indicate new particle in zone.
            select=true;
          }
        }
      }
      if(select)slist[atomicAdd(slist,1)+1]=p; //-Add particle in the list.
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((listp+n),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp=slist[0]+threadIdx.x;
    listp[cp]=slist[threadIdx.x+1];
  }
}

//==============================================================================
/// Creates list with current inout particles and normal (no periodic) fluid in 
/// inlet/outlet zones (update its code).
//==============================================================================
unsigned InOutCreateList(bool stable,unsigned n,unsigned pini
  ,byte chkinputmask,byte nzone,const byte *cfgzone,const float4 *planes
  ,tfloat3 freemin,tfloat3 freemax
  ,const float2 *boxlimit,const double2 *posxy,const double *posz
  ,typecode *code,unsigned *listp)
{
  unsigned count=0;
  if(n){
    //-listp size list initialized to zero.
    //-Inicializa tamanho de lista listp a cero.
    hipMemset(listp+n,0,sizeof(unsigned));
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutCreateList <<<sgrid,SPHBSIZE,smem>>> (n,pini,chkinputmask,nzone,cfgzone
      ,planes,Float3(freemin),Float3(freemax),boxlimit,posxy,posz,code,listp);
    hipMemcpy(&count,listp+n,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list when stable has been activated.
    //-Reordena lista cuando stable esta activado.
    if(stable && count){ //-Does not affect results.
      thrust::device_ptr<unsigned> dev_list(listp);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}


//------------------------------------------------------------------------------
/// Returns velocity according profile configuration (JSphInOutZone::TpVelProfile).
//------------------------------------------------------------------------------
__device__ float KerInOutCalcVel(byte vprof,const float4 &vdata,float posz){
  float vel=0;
  if(vprof==0)vel=vdata.x;  //-InVelP_Uniform
  else if(vprof==1){        //-InVelP_Linear
    const float m=vdata.x;
    const float b=vdata.y;
    vel=m*posz+b;
  }
  else if(vprof==2){        //-InVelP_Parabolic
    const float a=vdata.x;
    const float b=vdata.y;
    const float c=vdata.z;
    vel=a*posz*posz+b*posz+c;
  }
  return(vel);
}

//------------------------------------------------------------------------------
/// Updates velocity and rhop of inlet/outlet particles when it uses an 
/// analytical solution.
//------------------------------------------------------------------------------
__global__ void KerInOutSetAnalyticalData(unsigned n,const unsigned *inoutpart
  ,byte izone,byte rmode,byte vmode,byte vprof,byte refillspfull
  ,float timestep,float zsurfv,float4 veldata,float4 veldata2,float3 dirdata
  ,float coefhydro,float rhopzero,float gamma
  ,const typecode *code,const double *posz,const float *zsurfpart,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    if(izone==byte(CODE_GetIzoneFluidInout(code[p]))){
      const float zsurf=(zsurfpart? zsurfpart[cp]: zsurfv);
      const double rposz=posz[p];
      float4 rvelrhop=velrhop[p];
      //-Compute rhop value.
      if(rmode==0)rvelrhop.w=rhopzero; //-InRhop_Constant
      if(rmode==1){                    //-InRhop_Hydrostatic
        const float depth=float(double(zsurf)-rposz);
        const float rh=1.f+coefhydro*depth;     //rh=1.+rhop0*(-gravity.z)*(Dp*ptdata.GetDepth(p))/vCteB;
        const float frhop=pow(rh,1.f/gamma);    //rhop[id]=rhop0*pow(rh,(1./gamma));
        rvelrhop.w=rhopzero*(frhop<1.f? 1.f: frhop);//-Avoid rhop lower thand rhopzero to prevent suction.
        //rvelrhop.w=rhopzero*pow(rh,1.f/gamma);  //rhop[id]=rhop0*pow(rh,(1./gamma));
      }
      //-Compute velocity value.
      if(vmode<2){//-VelMode InVelM_Fixed or InVelM_Variable.
        float vel=0;
        if(!refillspfull || rposz<=zsurf){
          if(vmode==0){ //-InVelM_Fixed
            vel=KerInOutCalcVel(vprof,veldata,float(rposz));
          }
          else{ //-InVelM_Variable
            const float vel1=KerInOutCalcVel(vprof,veldata,float(rposz));
            const float vel2=KerInOutCalcVel(vprof,veldata2,float(rposz));
            const float time1=veldata.w;
            const float time2=veldata2.w;
            if(timestep<=time1 || time1==time2)vel=vel1;
            else if(timestep>=time2)vel=vel2;
            else vel=(timestep-time1)/(time2-time1)*(vel2-vel1)+vel1;
          }
        }
        rvelrhop.x=vel*dirdata.x;
        rvelrhop.y=vel*dirdata.y;
        rvelrhop.z=vel*dirdata.z;
      }
      velrhop[p]=rvelrhop;
    }
  }
}

//==============================================================================
/// Updates velocity and rhop of inlet/outlet particles when it uses an 
/// analytical solution.
//==============================================================================
void InOutSetAnalyticalData(unsigned n,const unsigned *inoutpart
  ,byte izone,byte rmode,byte vmode,byte vprof,byte refillspfull
  ,float timestep,float zsurfv,tfloat4 veldata,tfloat4 veldata2,tfloat3 dirdata
  ,float coefhydro,float rhopzero,float gamma
  ,const typecode *code,const double *posz,const float *zsurfpart,float4 *velrhop)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInOutSetAnalyticalData <<<sgrid,SPHBSIZE>>> (n,inoutpart,izone,rmode,vmode
      ,vprof,refillspfull,timestep,zsurfv,Float4(veldata),Float4(veldata2)
      ,Float3(dirdata),coefhydro,rhopzero,gamma,code,posz,zsurfpart,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Updates velocity and rhop of inlet/outlet particles when it is not extrapolated. 
/// Actualiza velocidad y densidad de particulas inlet/outlet cuando no es extrapolada.
//------------------------------------------------------------------------------
__global__ void KerInoutClearInteractionVars(unsigned n,unsigned pini,const typecode *code
  ,float3 *ace,float *ar,float *viscdt,float4 *shiftposfs)
{
  const unsigned pp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pp<n){
    const unsigned p=pp+pini;
    if(CODE_IsFluidInout(code[p])){
      ace[p]=make_float3(0,0,0);
      ar[p]=0;
      viscdt[p]=0;
      if(shiftposfs)shiftposfs[p]=make_float4(0,0,0,0);
    }
  }
}

//==============================================================================
/// Updates velocity and rhop of inlet/outlet particles when it is not extrapolated. 
/// Actualiza velocidad y densidad de particulas inlet/outlet cuando no es extrapolada.
//==============================================================================
void InoutClearInteractionVars(unsigned npf,unsigned pini,const typecode *code
  ,float3 *ace,float *ar,float *viscdt,float4 *shiftposfs)
{
  if(npf){
    dim3 sgrid=GetSimpleGridSize(npf,SPHBSIZE);
    KerInoutClearInteractionVars <<<sgrid,SPHBSIZE>>> (npf,pini,code,ace,ar,viscdt,shiftposfs);
  }
}


//------------------------------------------------------------------------------
/// Updates velocity and rhop for M1 variable when Verlet is used. 
/// Actualiza velocidad y densidad de varible M1 cuando se usa Verlet.
//------------------------------------------------------------------------------
__global__ void KerInOutUpdateVelrhopM1(unsigned n,const int *inoutpart
  ,const float4 *velrhop,float4 *velrhopm1)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    velrhopm1[p]=velrhop[p];
  }
}

//==============================================================================
/// Updates velocity and rhop for M1 variable when Verlet is used. 
/// Actualiza velocidad y densidad de varible M1 cuando se usa Verlet.
//==============================================================================
void InOutUpdateVelrhopM1(unsigned n,const int *inoutpart
  ,const float4 *velrhop,float4 *velrhopm1)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInOutUpdateVelrhopM1 <<<sgrid,SPHBSIZE>>> (n,inoutpart,velrhop,velrhopm1);
  }
}


//------------------------------------------------------------------------------
/// Checks particle position.
/// If particle is moved to fluid zone then it changes to fluid particle and 
/// it creates a new in/out particle.
/// If particle is moved out the domain then it changes to ignore particle.
//------------------------------------------------------------------------------
__global__ void KerInOutComputeStep(unsigned n,int *inoutpart,const float4 *planes
  ,const float *width,const byte *cfgupdate,const float *zsurfv,typecode codenewpart
  ,const double2 *posxy,const double *posz,const byte *zsurfok
  ,typecode *code,byte *newizone)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    typecode cod=0;
    byte newiz=255;
    const int p=inoutpart[cp];
    const typecode rcode=code[p];
    const byte izone0=byte(CODE_GetIzoneFluidInout(rcode));
    const byte izone=(izone0&CODE_TYPE_FLUID_INOUT015MASK); //-Substract 16 to obtain the actual zone (0-15).
    const byte cfupdate=cfgupdate[izone];
    const bool refilladvan=(cfupdate&INOUT_RefillAdvanced_MASK)!=0;
    const bool refillsfull=(cfupdate&INOUT_RefillSpFull_MASK)!=0;
    const bool removeinput=(cfupdate&INOUT_RemoveInput_MASK )!=0;
    const bool removezsurf=(cfupdate&INOUT_RemoveZsurf_MASK )!=0;
    const bool converinput=(cfupdate&INOUT_ConvertInput_MASK)!=0;
    const double2 rposxy=posxy[p];
    const float rposz=float(posz[p]);
    const bool zok=(zsurfok? zsurfok[cp]: rposz<=zsurfv[izone]);
    if(izone0>=16){//-Normal fluid particle in zone inlet/outlet.
      if(removeinput || (removezsurf && !zok))cod=CODE_SetOutPos(rcode); //-Normal fluid particle in zone inlet/outlet is removed.
      else cod=(converinput? rcode^0x10: codenewpart); //-Converts to inout particle or not.
    }
    else{//-Previous inout fluid particle.
      const float displane=-cugeo::PlaneDistSign(planes[izone],float(rposxy.x),float(rposxy.y),rposz);
      if(displane>width[izone] || (removezsurf && !zok)){
        cod=CODE_SetOutIgnore(rcode); //-Particle is moved out domain.
      }
      else if(displane<0){
        cod=codenewpart;//-Inout particle changes to fluid particle.
        if(!refilladvan && (refillsfull || zok))newiz=byte(izone); //-A new particle is created.
      }
    }
    newizone[cp]=newiz;
    if(cod!=0)code[p]=cod;
  }
}

//==============================================================================
/// Checks particle position.
/// If particle is moved to fluid zone then it changes to fluid particle and 
/// it creates a new in/out particle.
/// If particle is moved out the domain then it changes to ignore particle.
//==============================================================================
void InOutComputeStep(unsigned n,int *inoutpart,const float4 *planes
  ,const float *width,const byte *cfgupdate,const float *zsurfv,typecode codenewpart
  ,const double2 *posxy,const double *posz,const byte *zsurfok
  ,typecode *code,byte *newizone)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInOutComputeStep <<<sgrid,SPHBSIZE>>> (n,inoutpart,planes,width,cfgupdate,zsurfv
      ,codenewpart,posxy,posz,zsurfok,code,newizone);
  }
}


//------------------------------------------------------------------------------
/// Create list for new inlet particles to create.
/// Crea lista de nuevas particulas inlet a crear.
//------------------------------------------------------------------------------
__global__ void KerInOutListCreate(unsigned n,unsigned nmax,const byte *newizone,int *inoutpart)
{
  extern __shared__ unsigned slist[];
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n && newizone[cp]<16){
    slist[atomicAdd(slist,1)+1]=cp; 
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=n + atomicAdd((inoutpart+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp2=slist[0]+threadIdx.x;
    if(cp2<nmax)inoutpart[cp2]=slist[threadIdx.x+1];
  }
}

//==============================================================================
/// Create list for new inlet particles to create at end of inoutpart[]. 
/// Returns number of new particles to create.
/// 
/// Crea lista de nuevas particulas inlet a crear al final de inoutpart[].
/// Devuelve el numero de las nuevas particulas para crear.
//==============================================================================
unsigned InOutListCreate(bool stable,unsigned n,unsigned nmax,const byte *newizone,int *inoutpart)
{
  unsigned count=0;
  if(n){
    //-inoutpart size list initialized to zero.
    //-Inicializa tamanho de lista inoutpart a cero.
    hipMemset(inoutpart+nmax,0,sizeof(unsigned));
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutListCreate <<<sgrid,SPHBSIZE,smem>>> (n,nmax,newizone,inoutpart);
    hipMemcpy(&count,inoutpart+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list((unsigned*)inoutpart);
      thrust::sort(dev_list+n,dev_list+(n+count));
    }
  }
  return(count);
}


//------------------------------------------------------------------------------
/// Creates new inlet particles to replace the particles moved to fluid domain.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutCreateNewInlet(unsigned newn
  ,const unsigned *inoutpart,unsigned inoutcount,const byte *newizone
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata,const float *width
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<newn){
    const int cp0=inoutpart[inoutcount+cp];
    const int p=inoutpart[cp0];
    const byte izone=newizone[cp0];
    const double dis=width[izone];
    const float3 rdirdata=dirdata[izone];
    double2 rposxy=posxy[p];
    double rposz=posz[p];
    rposxy.x-=dis*rdirdata.x;
    rposxy.y-=dis*rdirdata.y;
    rposz   -=dis*rdirdata.z;
    const unsigned p2=np+cp;
    code[p2]=CODE_ToFluidInout(codenewpart,izone);
    cusph::KerUpdatePos<periactive>(rposxy,rposz,0,0,0,false,p2,posxy,posz,dcell,code);
    idp[p2]=idnext+cp;
    velrhop[p2]=make_float4(0,0,0,1000);
  }
}

//==============================================================================
/// Creates new inlet particles to replace the particles moved to fluid domain.
//==============================================================================
void InOutCreateNewInlet(byte periactive,unsigned newn
  ,const unsigned *inoutpart,unsigned inoutcount,const byte *newizone
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata,const float *width
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  if(newn){
    dim3 sgrid=GetSimpleGridSize(newn,SPHBSIZE);
    if(periactive)KerInOutCreateNewInlet<true>  <<<sgrid,SPHBSIZE>>> (newn,inoutpart,inoutcount,newizone,np,idnext,codenewpart,dirdata,width,posxy,posz,dcell,code,idp,velrhop);
    else          KerInOutCreateNewInlet<false> <<<sgrid,SPHBSIZE>>> (newn,inoutpart,inoutcount,newizone,np,idnext,codenewpart,dirdata,width,posxy,posz,dcell,code,idp,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Move in/out particles according its velocity.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutFillMove(unsigned n,const unsigned *inoutpart
  ,double dt,const float4 *velrhop
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    //-Updates position of particles.
    const float4 rvelrhop=velrhop[p];
    const double dx=double(rvelrhop.x)*dt;
    const double dy=double(rvelrhop.y)*dt;
    const double dz=double(rvelrhop.z)*dt;
    cusph::KerUpdatePos<periactive>(posxy[p],posz[p],dx,dy,dz,false,p,posxy,posz,dcell,code);
  }
}

//==============================================================================
/// Move particles in/out according its velocity.
//==============================================================================
void InOutFillMove(byte periactive,unsigned n,const unsigned *inoutpart
  ,double dt,const float4 *velrhop
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    if(periactive)KerInOutFillMove<true>  <<<sgrid,SPHBSIZE>>> (n,inoutpart,dt,velrhop,posxy,posz,dcell,code);
    else          KerInOutFillMove<false> <<<sgrid,SPHBSIZE>>> (n,inoutpart,dt,velrhop,posxy,posz,dcell,code);
  }
}


//------------------------------------------------------------------------------
/// Computes projection data to filling mode.
//------------------------------------------------------------------------------
__global__ void KerInOutFillProjection(unsigned n,const unsigned *inoutpart
  ,const byte *cfgupdate,const float4 *planes,const double2 *posxy,const double *posz
  ,const typecode *code,float *prodist,double2 *proposxy,double *proposz)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<n){
    const unsigned p=inoutpart[cp];
    float rprodis=0;
    double rpropx=0,rpropy=0,rpropz=0;
    //-Checks if particle was moved to fluid domain.
    const typecode rcode=code[p];
    if(CODE_IsNotOut(rcode) && CODE_IsFluidInout(rcode)){
      const byte izone=byte(CODE_GetIzoneFluidInout(rcode));
      if((cfgupdate[izone]&INOUT_RefillAdvanced_MASK)!=0){
        const double2 rposxy=posxy[p];
        const double rposz=posz[p];
        const float4 rplanes=planes[izone];
        //-Compute distance to plane.
        const double v1=rposxy.x*rplanes.x + rposxy.y*rplanes.y + rposz*rplanes.z + rplanes.w;
        const double v2=rplanes.x*rplanes.x+rplanes.y*rplanes.y+rplanes.z*rplanes.z;
        rprodis=-float(v1/sqrt(v2));//-Equivalent to cugeo::PlaneDistSign().
        //-Calculates point on plane.
        const double t=-v1/v2;
        rpropx=rposxy.x+t*rplanes.x;
        rpropy=rposxy.y+t*rplanes.y;
        rpropz=rposz+t*rplanes.z;
      }
    }
    //-Saves results on GPU memory.
    prodist[cp]=rprodis;
    proposxy[cp]=make_double2(rpropx,rpropy);
    proposz[cp] =rpropz;
  }
}

//==============================================================================
/// Computes projection data to filling mode.
//==============================================================================
void InOutFillProjection(unsigned n,const unsigned *inoutpart
  ,const byte *cfgupdate,const float4 *planes,const double2 *posxy,const double *posz
  ,const typecode *code,float *prodist,double2 *proposxy,double *proposz)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInOutFillProjection <<<sgrid,SPHBSIZE>>> (n,inoutpart,cfgupdate,planes,posxy,posz
      ,code,prodist,proposxy,proposz);
  }
}


//------------------------------------------------------------------------------
/// Compute maximum distance to create points in each PtPos.
/// Create list of selected ptpoints and its distance for new inlet/outlet particles.
//------------------------------------------------------------------------------
__global__ void KerInOutFillListCreate(unsigned npt
  ,const double2 *ptposxy,const double *ptposz,const byte *zsurfok
  ,const byte *ptzone,const byte *cfgupdate,const float *zsurf,const float *width
  ,unsigned npropt,const float *prodist,const double2 *proposxy,const double *proposz
  ,float dpmin,float dpmin2,float dp,float *ptdist,unsigned nmax,unsigned *inoutpart)
{
  extern __shared__ unsigned slist[];
  //float *sdist=(float*)(slist+(blockDim.x+1));
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned cpt=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cpt<npt){
    float distmax=FLT_MAX;
    const byte izone=ptzone[cpt];
    if((cfgupdate[izone]&INOUT_RefillAdvanced_MASK)!=0){
      const double2 rptxy=ptposxy[cpt];
      const double rptz=ptposz[cpt];
      const bool zok=(zsurfok? zsurfok[cpt]: float(rptz)<=zsurf[izone]);
      if(zok){
        distmax=0;
        for(int cpro=0;cpro<npropt;cpro++){
          const double2 propsxy=proposxy[cpro];
          const float disx=rptxy.x-propsxy.x;
          const float disy=rptxy.y-propsxy.y;
          const float disz=rptz   -proposz [cpro];
          if(disx<=dpmin && disy<=dpmin && disz<=dpmin){//-particle near to ptpoint (approx.)
            const float dist2=(disx*disx+disy*disy+disz*disz);
            if(dist2<dpmin2){//-particle near to ptpoint.
              const float dmax=prodist[cpro]+sqrt(dpmin2-dist2);
              distmax=max(distmax,dmax);
            }
          }
        }
      }
    }
    distmax=(distmax==0? dp: distmax);
    //-Creates list of new inlet/outlet particles.
    if(distmax<width[ptzone[cpt]]){
      slist[atomicAdd(slist,1)+1]=cpt; //-Add ptpoint in the list.
      ptdist[cpt]=distmax;             //-Saves distance of ptpoint.
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((inoutpart+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    const unsigned cp2=slist[0]+threadIdx.x;
    if(cp2<nmax)inoutpart[cp2]=slist[threadIdx.x+1];
  }
}

//==============================================================================
/// Compute maximum distance to create points in each PtPos.
/// Create list of selected ptpoints and its distance for new inlet/outlet particles.
/// Returns number of new particles to create.
//==============================================================================
unsigned InOutFillListCreate(bool stable,unsigned npt
  ,const double2 *ptposxy,const double *ptposz,const byte *zsurfok
  ,const byte *ptzone,const byte *cfgupdate,const float *zsurf,const float *width
  ,unsigned npropt,const float *prodist,const double2 *proposxy,const double *proposz
  ,float dpmin,float dpmin2,float dp,float *ptdist,unsigned nmax,unsigned *inoutpart)
{
  unsigned count=0;
  if(npt){
    //-inoutpart size list initialized to zero.
    //-Inicializa tamanho de lista inoutpart a cero.
    hipMemset(inoutpart+nmax,0,sizeof(unsigned));
    dim3 sgrid=GetSimpleGridSize(npt,SPHBSIZE);
    const unsigned smem=(SPHBSIZE+1)*sizeof(unsigned); //-All fluid particles can be in in/out area and one position for counter.
    KerInOutFillListCreate <<<sgrid,SPHBSIZE,smem>>> (npt,ptposxy,ptposz,zsurfok
      ,ptzone,cfgupdate,zsurf,width,npropt,prodist,proposxy,proposz,dpmin,dpmin2
      ,dp,ptdist,nmax,inoutpart);
    hipMemcpy(&count,inoutpart+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list((unsigned*)inoutpart);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}


//------------------------------------------------------------------------------
/// Creates new inlet/outlet particles to fill inlet/outlet domain.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerInOutFillCreate(unsigned newn,const unsigned *newinoutpart
  ,const double2 *ptposxy,const double *ptposz,const byte *ptzone,const float *ptauxdist
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<newn){
    const unsigned cpt=newinoutpart[cp];
    const byte izone=ptzone[cpt];
    const double dis=ptauxdist[cpt];
    const float3 rdirdata=dirdata[izone];
    double2 rposxy=ptposxy[cpt];
    double rposz=ptposz[cpt];
    rposxy.x-=dis*rdirdata.x;
    rposxy.y-=dis*rdirdata.y;
    rposz   -=dis*rdirdata.z;
    const unsigned p=np+cp;
    code[p]=CODE_ToFluidInout(codenewpart,izone);
    cusph::KerUpdatePos<periactive>(rposxy,rposz,0,0,0,false,p,posxy,posz,dcell,code);
    idp[p]=idnext+cp;
    velrhop[p]=make_float4(0,0,0,1000);
  }
}

//==============================================================================
/// Creates new inlet/outlet particles to fill inlet/outlet domain.
//==============================================================================
void InOutFillCreate(byte periactive,unsigned newn,const unsigned *newinoutpart
  ,const double2 *ptposxy,const double *ptposz,const byte *ptzone,const float *ptauxdist
  ,unsigned np,unsigned idnext,typecode codenewpart,const float3 *dirdata
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code,unsigned *idp,float4 *velrhop)
{
  if(newn){
    dim3 sgrid=GetSimpleGridSize(newn,SPHBSIZE);
    if(periactive)KerInOutFillCreate<true>  <<<sgrid,SPHBSIZE>>> (newn,newinoutpart,ptposxy,ptposz,ptzone,ptauxdist,np,idnext,codenewpart,dirdata,posxy,posz,dcell,code,idp,velrhop);
    else          KerInOutFillCreate<false> <<<sgrid,SPHBSIZE>>> (newn,newinoutpart,ptposxy,ptposz,ptzone,ptauxdist,np,idnext,codenewpart,dirdata,posxy,posz,dcell,code,idp,velrhop);
  }
}


//------------------------------------------------------------------------------
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionInOutExtrap_Double
  (unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<inoutcount){
    const unsigned p1=inoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p1]));
    const byte cfg=cfgzone[izone];
    const bool computerhop=((cfg&computerhopmask)!=0);
    const bool computevel= ((cfg&computevelmask )!=0);
    if(computerhop || computevel){
      //-Calculates ghost node position.
      double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
      if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
      const double displane=cumath::DistPlane(planes[izone],pos_p1)*2;
      const float3 rdirdata=dirdata[izone];
      const double3 posp1=make_double3(pos_p1.x+displane*rdirdata.x, pos_p1.y+displane*rdirdata.y, pos_p1.z+displane*rdirdata.z); //-Ghost node position.

      //-Initializes variables for calculation.
      double rhopp1=0;
      double3 gradrhopp1=make_double3(0,0,0);
      double3 velp1=make_double3(0,0,0);
      tmatrix3d gradvelp1; cumath::Tmatrix3dReset(gradvelp1); //-Only for velocity.
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(posp1.x,posp1.y,posp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Interaction with fluids.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const double2 p2xy=posxy[p2];
          const double drx=double(posp1.x-p2xy.x);
          const double dry=double(posp1.y-p2xy.y);
          const double drz=double(posp1.z-posz[p2]);
          const double rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluidNotInout(code[p2])){//-Only with fluid particles but not inout particles.
            //-Only Wendland or Cubic Spline kernel.
            //-Computes kernel.
            float fac;
            const double wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const double frx=drx*fac,fry=dry*fac,frz=drz*fac; //-Gradients.

            const float4 velrhopp2=velrhop[p2];
            //===== Get mass and volume of particle p2 =====
            double massp2=CTE.massf;
            double volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const double vwab=wab*volp2;
            const double vfrx=frx*volp2;
            const double vfry=fry*volp2;
            const double vfrz=frz*volp2;

            //===== Velocity and its gradient =====
            if(computevel){
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
              gradvelp1.a11+=vfrx*velrhopp2.x;    // du/dx
              gradvelp1.a12+=vfry*velrhopp2.x;    // du/dy
              gradvelp1.a13+=vfrz*velrhopp2.x;    // du/dz
              gradvelp1.a21+=vfrx*velrhopp2.y;    // dv/dx
              gradvelp1.a22+=vfry*velrhopp2.y;    // dv/dx
              gradvelp1.a23+=vfrz*velrhopp2.y;    // dv/dx
              gradvelp1.a31+=vfrx*velrhopp2.z;    // dw/dx
              gradvelp1.a32+=vfry*velrhopp2.z;    // dw/dx
              gradvelp1.a33+=vfrz*velrhopp2.z;    // dw/dx
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      float4 velrhopfinal=velrhop[p1];
      const double3 dpos=make_double3(pos_p1.x-posp1.x, pos_p1.y-posp1.y, pos_p1.z-posp1.z); //-Inlet/outlet particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3(a_corr2);
        if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const double rhoghost=rhopp1*invacorr2.a11 + gradrhopp1.x*invacorr2.a12 + gradrhopp1.z*invacorr2.a13;
            const double grx=-(rhopp1*invacorr2.a21 + gradrhopp1.x*invacorr2.a22 + gradrhopp1.z*invacorr2.a23);
            const double grz=-(rhopp1*invacorr2.a31 + gradrhopp1.x*invacorr2.a32 + gradrhopp1.z*invacorr2.a33);
            velrhopfinal.w=float(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const double velghost_x=velp1.x*invacorr2.a11 + gradvelp1.a11*invacorr2.a12 + gradvelp1.a13*invacorr2.a13;
            const double velghost_z=velp1.z*invacorr2.a11 + gradvelp1.a31*invacorr2.a12 + gradvelp1.a33*invacorr2.a13;
            const double a11=-(velp1.x*invacorr2.a21 + gradvelp1.a11*invacorr2.a22 + gradvelp1.a13*invacorr2.a23);
            const double a13=-(velp1.z*invacorr2.a21 + gradvelp1.a31*invacorr2.a22 + gradvelp1.a33*invacorr2.a23);
            const double a31=-(velp1.x*invacorr2.a31 + gradvelp1.a11*invacorr2.a32 + gradvelp1.a13*invacorr2.a33);
            const double a33=-(velp1.z*invacorr2.a31 + gradvelp1.a31*invacorr2.a32 + gradvelp1.a33*invacorr2.a33);
            velrhopfinal.x=float(velghost_x + a11*dpos.x + a31*dpos.z);
            velrhopfinal.z=float(velghost_z + a13*dpos.x + a33*dpos.z);
            velrhopfinal.y=0;
          }
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr2.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
      }
      else{
        const double determ=cumath::Determinant4x4(a_corr3);
        if(fabs(determ)>=determlimit){
          const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const double rhoghost=rhopp1*invacorr3.a11 + gradrhopp1.x*invacorr3.a12 + gradrhopp1.y*invacorr3.a13 + gradrhopp1.z*invacorr3.a14;
            const double grx=   -(rhopp1*invacorr3.a21 + gradrhopp1.x*invacorr3.a22 + gradrhopp1.y*invacorr3.a23 + gradrhopp1.z*invacorr3.a24);
            const double gry=   -(rhopp1*invacorr3.a31 + gradrhopp1.x*invacorr3.a32 + gradrhopp1.y*invacorr3.a33 + gradrhopp1.z*invacorr3.a34);
            const double grz=   -(rhopp1*invacorr3.a41 + gradrhopp1.x*invacorr3.a42 + gradrhopp1.y*invacorr3.a43 + gradrhopp1.z*invacorr3.a44);
            velrhopfinal.w=float(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const double velghost_x=velp1.x*invacorr3.a11 + gradvelp1.a11*invacorr3.a12 + gradvelp1.a12*invacorr3.a13 + gradvelp1.a13*invacorr3.a14;
            const double velghost_y=velp1.y*invacorr3.a11 + gradvelp1.a21*invacorr3.a12 + gradvelp1.a22*invacorr3.a13 + gradvelp1.a23*invacorr3.a14;
            const double velghost_z=velp1.z*invacorr3.a11 + gradvelp1.a31*invacorr3.a12 + gradvelp1.a32*invacorr3.a13 + gradvelp1.a33*invacorr3.a14;
            const double a11=-(velp1.x*invacorr3.a21 + gradvelp1.a11*invacorr3.a22 + gradvelp1.a12*invacorr3.a23 + gradvelp1.a13*invacorr3.a24);
            const double a12=-(velp1.y*invacorr3.a21 + gradvelp1.a21*invacorr3.a22 + gradvelp1.a22*invacorr3.a23 + gradvelp1.a23*invacorr3.a24);
            const double a13=-(velp1.z*invacorr3.a21 + gradvelp1.a31*invacorr3.a22 + gradvelp1.a32*invacorr3.a23 + gradvelp1.a33*invacorr3.a24);
            const double a21=-(velp1.x*invacorr3.a31 + gradvelp1.a11*invacorr3.a32 + gradvelp1.a12*invacorr3.a33 + gradvelp1.a13*invacorr3.a34);
            const double a22=-(velp1.y*invacorr3.a31 + gradvelp1.a21*invacorr3.a32 + gradvelp1.a22*invacorr3.a33 + gradvelp1.a23*invacorr3.a34);
            const double a23=-(velp1.z*invacorr3.a31 + gradvelp1.a31*invacorr3.a32 + gradvelp1.a32*invacorr3.a33 + gradvelp1.a33*invacorr3.a34);
            const double a31=-(velp1.x*invacorr3.a41 + gradvelp1.a11*invacorr3.a42 + gradvelp1.a12*invacorr3.a43 + gradvelp1.a13*invacorr3.a44);
            const double a32=-(velp1.y*invacorr3.a41 + gradvelp1.a21*invacorr3.a42 + gradvelp1.a22*invacorr3.a43 + gradvelp1.a23*invacorr3.a44);
            const double a33=-(velp1.z*invacorr3.a41 + gradvelp1.a31*invacorr3.a42 + gradvelp1.a32*invacorr3.a43 + gradvelp1.a33*invacorr3.a44);
            velrhopfinal.x=float(velghost_x + a11*dpos.x + a21*dpos.y + a31*dpos.z);
            velrhopfinal.y=float(velghost_y + a12*dpos.x + a22*dpos.y + a32*dpos.z);
            velrhopfinal.z=float(velghost_z + a13*dpos.x + a23*dpos.y + a33*dpos.z);
          }
        }
        else if(a_corr3.a11>0){ // Determinant is small but a11 is nonzero, 0th order ANGELO
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr3.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
      }
      velrhop[p1]=velrhopfinal;
    }
  }
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionInOutExtrap_Single
  (unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<inoutcount){
    const unsigned p1=inoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p1]));
    const byte cfg=cfgzone[izone];
    const bool computerhop=((cfg&computerhopmask)!=0);
    const bool computevel= ((cfg&computevelmask )!=0);
    if(computerhop || computevel){
      //-Calculates ghost node position.
      double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
      if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
      const double displane=cumath::DistPlane(planes[izone],pos_p1)*2;
      const float3 rdirdata=dirdata[izone];
      const double3 posp1=make_double3(pos_p1.x+displane*rdirdata.x, pos_p1.y+displane*rdirdata.y, pos_p1.z+displane*rdirdata.z); //-Ghost node position.

      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);
      tmatrix3f gradvelp1; cumath::Tmatrix3fReset(gradvelp1); //-Only for velocity.
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(posp1.x,posp1.y,posp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Interaction with fluids.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const double2 p2xy=posxy[p2];
          const float drx=float(posp1.x-p2xy.x);
          const float dry=float(posp1.y-p2xy.y);
          const float drz=float(posp1.z-posz[p2]);
          const float rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluidNotInout(code[p2])){//-Only with fluid particles but not inout particles.
            //-Computes kernel.
            float fac;
            const float wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

            const float4 velrhopp2=velrhop[p2];
            //===== Get mass and volume of particle p2 =====
            float massp2=CTE.massf;
            float volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const float vwab=wab*volp2;
            const float vfrx=frx*volp2;
            const float vfry=fry*volp2;
            const float vfrz=frz*volp2;

            //===== Velocity and its gradient =====
            if(computevel){
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
              gradvelp1.a11+=vfrx*velrhopp2.x;    // du/dx
              gradvelp1.a12+=vfry*velrhopp2.x;    // du/dy
              gradvelp1.a13+=vfrz*velrhopp2.x;    // du/dz
              gradvelp1.a21+=vfrx*velrhopp2.y;    // dv/dx
              gradvelp1.a22+=vfry*velrhopp2.y;    // dv/dx
              gradvelp1.a23+=vfrz*velrhopp2.y;    // dv/dx
              gradvelp1.a31+=vfrx*velrhopp2.z;    // dw/dx
              gradvelp1.a32+=vfry*velrhopp2.z;    // dw/dx
              gradvelp1.a33+=vfrz*velrhopp2.z;    // dw/dx
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      float4 velrhopfinal=velrhop[p1];
      const float3 dpos=make_float3(float(pos_p1.x-posp1.x),float(pos_p1.y-posp1.y),float(pos_p1.z-posp1.z)); //-Inlet/outlet particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3(a_corr2);
        if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr2.a11*velp1.x + invacorr2.a12*gradvelp1.a11 + invacorr2.a13*gradvelp1.a13);
            const float velghost_z=float(invacorr2.a11*velp1.z + invacorr2.a12*gradvelp1.a31 + invacorr2.a13*gradvelp1.a33);
            const float a11=-float(invacorr2.a21*velp1.x + invacorr2.a22*gradvelp1.a11 + invacorr2.a23*gradvelp1.a13);
            const float a13=-float(invacorr2.a21*velp1.z + invacorr2.a22*gradvelp1.a31 + invacorr2.a23*gradvelp1.a33);
            const float a31=-float(invacorr2.a31*velp1.x + invacorr2.a32*gradvelp1.a11 + invacorr2.a33*gradvelp1.a13);
            const float a33=-float(invacorr2.a31*velp1.z + invacorr2.a32*gradvelp1.a31 + invacorr2.a33*gradvelp1.a33);
            velrhopfinal.x=(velghost_x + a11*dpos.x + a31*dpos.z);
            velrhopfinal.z=(velghost_z + a13*dpos.x + a33*dpos.z);
            velrhopfinal.y=0;
          }
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr2.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
      }
      else{
        const double determ=cumath::Determinant4x4(a_corr3);
        if(fabs(determ)>=determlimit){
          const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr3.a11*velp1.x + invacorr3.a12*gradvelp1.a11 + invacorr3.a13*gradvelp1.a12 + invacorr3.a14*gradvelp1.a13);
            const float velghost_y=float(invacorr3.a11*velp1.y + invacorr3.a12*gradvelp1.a21 + invacorr3.a13*gradvelp1.a22 + invacorr3.a14*gradvelp1.a23);
            const float velghost_z=float(invacorr3.a11*velp1.z + invacorr3.a12*gradvelp1.a31 + invacorr3.a13*gradvelp1.a32 + invacorr3.a14*gradvelp1.a33);
            const float a11=      -float(invacorr3.a21*velp1.x + invacorr3.a22*gradvelp1.a11 + invacorr3.a23*gradvelp1.a12 + invacorr3.a24*gradvelp1.a13);
            const float a12=      -float(invacorr3.a21*velp1.y + invacorr3.a22*gradvelp1.a21 + invacorr3.a23*gradvelp1.a22 + invacorr3.a24*gradvelp1.a23);
            const float a13=      -float(invacorr3.a21*velp1.z + invacorr3.a22*gradvelp1.a31 + invacorr3.a23*gradvelp1.a32 + invacorr3.a24*gradvelp1.a33);
            const float a21=      -float(invacorr3.a31*velp1.x + invacorr3.a32*gradvelp1.a11 + invacorr3.a33*gradvelp1.a12 + invacorr3.a34*gradvelp1.a13);
            const float a22=      -float(invacorr3.a31*velp1.y + invacorr3.a32*gradvelp1.a21 + invacorr3.a33*gradvelp1.a22 + invacorr3.a34*gradvelp1.a23);
            const float a23=      -float(invacorr3.a31*velp1.z + invacorr3.a32*gradvelp1.a31 + invacorr3.a33*gradvelp1.a32 + invacorr3.a34*gradvelp1.a33);
            const float a31=      -float(invacorr3.a41*velp1.x + invacorr3.a42*gradvelp1.a11 + invacorr3.a43*gradvelp1.a12 + invacorr3.a44*gradvelp1.a13);
            const float a32=      -float(invacorr3.a41*velp1.y + invacorr3.a42*gradvelp1.a21 + invacorr3.a43*gradvelp1.a22 + invacorr3.a44*gradvelp1.a23);
            const float a33=      -float(invacorr3.a41*velp1.z + invacorr3.a42*gradvelp1.a31 + invacorr3.a43*gradvelp1.a32 + invacorr3.a44*gradvelp1.a33);
            velrhopfinal.x=(velghost_x + a11*dpos.x + a21*dpos.y + a31*dpos.z);
            velrhopfinal.y=(velghost_y + a12*dpos.x + a22*dpos.y + a32*dpos.z);
            velrhopfinal.z=(velghost_z + a13*dpos.x + a23*dpos.y + a33*dpos.z);
          }
        }
        else if(a_corr3.a11>0){ // Determinant is small but a11 is nonzero, 0th order ANGELO
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr3.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
      }
      velrhop[p1]=velrhopfinal;
    }
  }
}


//------------------------------------------------------------------------------
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//------------------------------------------------------------------------------
template<bool sim2d,TpKernel tker> __global__ void KerInteractionInOutExtrap_FastSingle
  (unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<inoutcount){
    const unsigned p1=inoutpart[cp];
    const byte izone=byte(CODE_GetIzoneFluidInout(code[p1]));
    const byte cfg=cfgzone[izone];
    const bool computerhop=((cfg&computerhopmask)!=0);
    const bool computevel= ((cfg&computevelmask )!=0);
    if(computerhop || computevel){
      //-Calculates ghost node position.
      double3 pos_p1=make_double3(posxy[p1].x,posxy[p1].y,posz[p1]);
      if(CODE_IsPeriodic(code[p1]))pos_p1=KerInteraction_PosNoPeriodic(pos_p1);
      const double displane=cumath::DistPlane(planes[izone],pos_p1)*2;
      const float3 rdirdata=dirdata[izone];
      const double3 posp1=make_double3(pos_p1.x+displane*rdirdata.x, pos_p1.y+displane*rdirdata.y, pos_p1.z+displane*rdirdata.z); //-Ghost node position.

      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);
      tmatrix3f gradvelp1; cumath::Tmatrix3fReset(gradvelp1); //-Only for velocity.
      tmatrix3f a_corr2; if(sim2d) cumath::Tmatrix3fReset(a_corr2); //-Only for 2D.
      tmatrix4f a_corr3; if(!sim2d)cumath::Tmatrix4fReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(posp1.x,posp1.y,posp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Interaction with fluids.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const double2 p2xy=posxy[p2];
          const float drx=float(posp1.x-p2xy.x);
          const float dry=float(posp1.y-p2xy.y);
          const float drz=float(posp1.z-posz[p2]);
          const float rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluidNotInout(code[p2])){//-Only with fluid particles but not inout particles.
            //-Computes kernel.
            float fac;
            const float wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

            const float4 velrhopp2=velrhop[p2];
            //===== Get mass and volume of particle p2 =====
            float massp2=CTE.massf;
            float volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const float vwab=wab*volp2;
            const float vfrx=frx*volp2;
            const float vfry=fry*volp2;
            const float vfrz=frz*volp2;

            //===== Velocity and its gradient =====
            if(computevel){
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
              gradvelp1.a11+=vfrx*velrhopp2.x;    // du/dx
              gradvelp1.a12+=vfry*velrhopp2.x;    // du/dy
              gradvelp1.a13+=vfrz*velrhopp2.x;    // du/dz
              gradvelp1.a21+=vfrx*velrhopp2.y;    // dv/dx
              gradvelp1.a22+=vfry*velrhopp2.y;    // dv/dx
              gradvelp1.a23+=vfrz*velrhopp2.y;    // dv/dx
              gradvelp1.a31+=vfrx*velrhopp2.z;    // dw/dx
              gradvelp1.a32+=vfry*velrhopp2.z;    // dw/dx
              gradvelp1.a33+=vfrz*velrhopp2.z;    // dw/dx
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      float4 velrhopfinal=velrhop[p1];
      const float3 dpos=make_float3(float(pos_p1.x-posp1.x),float(pos_p1.y-posp1.y),float(pos_p1.z-posp1.z)); //-Inlet/outlet particle position - ghost node position.
      if(sim2d){
        const double determ=cumath::Determinant3x3dbl(a_corr2);
        if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
          const tmatrix3f invacorr2=cumath::InverseMatrix3x3dbl(a_corr2,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr2.a11*velp1.x + invacorr2.a12*gradvelp1.a11 + invacorr2.a13*gradvelp1.a13);
            const float velghost_z=float(invacorr2.a11*velp1.z + invacorr2.a12*gradvelp1.a31 + invacorr2.a13*gradvelp1.a33);
            const float a11=-float(invacorr2.a21*velp1.x + invacorr2.a22*gradvelp1.a11 + invacorr2.a23*gradvelp1.a13);
            const float a13=-float(invacorr2.a21*velp1.z + invacorr2.a22*gradvelp1.a31 + invacorr2.a23*gradvelp1.a33);
            const float a31=-float(invacorr2.a31*velp1.x + invacorr2.a32*gradvelp1.a11 + invacorr2.a33*gradvelp1.a13);
            const float a33=-float(invacorr2.a31*velp1.z + invacorr2.a32*gradvelp1.a31 + invacorr2.a33*gradvelp1.a33);
            velrhopfinal.x=(velghost_x + a11*dpos.x + a31*dpos.z);
            velrhopfinal.z=(velghost_z + a13*dpos.x + a33*dpos.z);
            velrhopfinal.y=0;
          }
        }
        else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr2.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
      }
      else{
        const double determ=cumath::Determinant4x4dbl(a_corr3);
        if(fabs(determ)>=determlimit){
          const tmatrix4f invacorr3=cumath::InverseMatrix4x4dbl(a_corr3,determ);
          //-GHOST NODE DENSITY IS MIRRORED BACK TO THE INFLOW OR OUTFLOW PARTICLES.
          if(computerhop){
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            velrhopfinal.w=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          //-GHOST NODE VELOCITY ARE MIRRORED BACK TO THE OUTFLOW PARTICLES.
          if(computevel){
            const float velghost_x=float(invacorr3.a11*velp1.x + invacorr3.a12*gradvelp1.a11 + invacorr3.a13*gradvelp1.a12 + invacorr3.a14*gradvelp1.a13);
            const float velghost_y=float(invacorr3.a11*velp1.y + invacorr3.a12*gradvelp1.a21 + invacorr3.a13*gradvelp1.a22 + invacorr3.a14*gradvelp1.a23);
            const float velghost_z=float(invacorr3.a11*velp1.z + invacorr3.a12*gradvelp1.a31 + invacorr3.a13*gradvelp1.a32 + invacorr3.a14*gradvelp1.a33);
            const float a11=      -float(invacorr3.a21*velp1.x + invacorr3.a22*gradvelp1.a11 + invacorr3.a23*gradvelp1.a12 + invacorr3.a24*gradvelp1.a13);
            const float a12=      -float(invacorr3.a21*velp1.y + invacorr3.a22*gradvelp1.a21 + invacorr3.a23*gradvelp1.a22 + invacorr3.a24*gradvelp1.a23);
            const float a13=      -float(invacorr3.a21*velp1.z + invacorr3.a22*gradvelp1.a31 + invacorr3.a23*gradvelp1.a32 + invacorr3.a24*gradvelp1.a33);
            const float a21=      -float(invacorr3.a31*velp1.x + invacorr3.a32*gradvelp1.a11 + invacorr3.a33*gradvelp1.a12 + invacorr3.a34*gradvelp1.a13);
            const float a22=      -float(invacorr3.a31*velp1.y + invacorr3.a32*gradvelp1.a21 + invacorr3.a33*gradvelp1.a22 + invacorr3.a34*gradvelp1.a23);
            const float a23=      -float(invacorr3.a31*velp1.z + invacorr3.a32*gradvelp1.a31 + invacorr3.a33*gradvelp1.a32 + invacorr3.a34*gradvelp1.a33);
            const float a31=      -float(invacorr3.a41*velp1.x + invacorr3.a42*gradvelp1.a11 + invacorr3.a43*gradvelp1.a12 + invacorr3.a44*gradvelp1.a13);
            const float a32=      -float(invacorr3.a41*velp1.y + invacorr3.a42*gradvelp1.a21 + invacorr3.a43*gradvelp1.a22 + invacorr3.a44*gradvelp1.a23);
            const float a33=      -float(invacorr3.a41*velp1.z + invacorr3.a42*gradvelp1.a31 + invacorr3.a43*gradvelp1.a32 + invacorr3.a44*gradvelp1.a33);
            velrhopfinal.x=(velghost_x + a11*dpos.x + a21*dpos.y + a31*dpos.z);
            velrhopfinal.y=(velghost_y + a12*dpos.x + a22*dpos.y + a32*dpos.z);
            velrhopfinal.z=(velghost_z + a13*dpos.x + a23*dpos.y + a33*dpos.z);
          }
        }
        else if(a_corr3.a11>0){ // Determinant is small but a11 is nonzero, 0th order ANGELO
          if(computerhop)velrhopfinal.w=float(rhopp1/a_corr3.a11);
          if(computevel){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
      }
      velrhop[p1]=velrhopfinal;
    }
  }
}

//==============================================================================
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//==============================================================================
template<TpKernel tker> void Interaction_InOutExtrapT(byte doublemode,bool simulate2d
  ,unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,const StDivDataGpu &dvd,const double2 *posxy,const double *posz,const typecode *code
  ,const unsigned *idp,float4 *velrhop)
{
  const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
  //-Interaction GhostBoundaryNodes-Fluid.
  if(inoutcount){
    const unsigned bsize=128;
    dim3 sgrid=GetSimpleGridSize(inoutcount,bsize);
    if(simulate2d){ const bool sim2d=true;
      switch(doublemode){
        case 1:  KerInteractionInOutExtrap_FastSingle<sim2d,tker> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,posxy,posz,code,idp,velrhop);  break;
        case 2:  KerInteractionInOutExtrap_Single    <sim2d,tker> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,posxy,posz,code,idp,velrhop);  break;
        case 3:  KerInteractionInOutExtrap_Double    <sim2d,tker> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,posxy,posz,code,idp,velrhop);  break;
      }
    }
    else{           const bool sim2d=false;
      switch(doublemode){
        case 1:  KerInteractionInOutExtrap_FastSingle<sim2d,tker> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,posxy,posz,code,idp,velrhop);  break;
        case 2:  KerInteractionInOutExtrap_Single    <sim2d,tker> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,posxy,posz,code,idp,velrhop);  break;
        case 3:  KerInteractionInOutExtrap_Double    <sim2d,tker> <<<sgrid,bsize>>> (inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask,planes,width,dirdata,determlimit,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,posxy,posz,code,idp,velrhop);  break;
      }
    }
  }
}

//==============================================================================
/// Perform interaction between ghost inlet/outlet nodes and fluid particles. GhostNodes-Fluid
/// Realiza interaccion entre ghost inlet/outlet nodes y particulas de fluido. GhostNodes-Fluid
//==============================================================================
void Interaction_InOutExtrap(byte doublemode,bool simulate2d,TpKernel tkernel
  ,unsigned inoutcount,const int *inoutpart,const byte *cfgzone,byte computerhopmask,byte computevelmask
  ,const float4 *planes,const float* width,const float3 *dirdata,float determlimit
  ,const StDivDataGpu &dvd,const double2 *posxy,const double *posz,const typecode *code
  ,const unsigned *idp,float4 *velrhop)
{
  switch(tkernel){
    case KERNEL_Wendland:
      Interaction_InOutExtrapT<KERNEL_Wendland>(doublemode,simulate2d
        ,inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask
        ,planes,width,dirdata,determlimit,dvd,posxy,posz,code,idp,velrhop);
    break;
#ifndef DISABLE_KERNELS_EXTRA
    case KERNEL_Cubic:
      Interaction_InOutExtrapT<KERNEL_Cubic>(doublemode,simulate2d
        ,inoutcount,inoutpart,cfgzone,computerhopmask,computevelmask
        ,planes,width,dirdata,determlimit,dvd,posxy,posz,code,idp,velrhop);
    break;
#endif
    default: throw "Kernel unknown at Interaction_InOutExtrap().";
  }
}


//##############################################################################
//# Kernels to interpolate velocity (JSphInOutGridDataTime).
//# Kernels para interpolar valores de velocidad (JSphInOutGridDataTime).
//##############################################################################
//------------------------------------------------------------------------------
/// Interpolate data between time0 and time1.
//------------------------------------------------------------------------------
__global__ void KerInOutInterpolateTime(unsigned npt,double fxtime
  ,const float *vel0,const float *vel1,float *vel)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<npt){
    const float v0=vel0[p];
    vel[p]=float(fxtime*(vel1[p]-v0)+v0);
  }
}

//==============================================================================
/// Interpolate data between time0 and time1.
//==============================================================================
void InOutInterpolateTime(unsigned npt,double time,double t0,double t1
  ,const float *velx0,const float *velx1,float *velx
  ,const float *velz0,const float *velz1,float *velz)
{
  if(npt){
    const double fxtime=((time-t0)/(t1-t0));
    dim3 sgrid=GetSimpleGridSize(npt,SPHBSIZE);
    KerInOutInterpolateTime <<<sgrid,SPHBSIZE>>> (npt,fxtime,velx0,velx1,velx);
    if(velz0)KerInOutInterpolateTime <<<sgrid,SPHBSIZE>>> (npt,fxtime,velz0,velz1,velz);
  }
}

//------------------------------------------------------------------------------
/// Interpolate velocity in time and Z-position of selected partiles in a list.
//------------------------------------------------------------------------------
__global__ void KerInOutInterpolateZVel(unsigned izone,double posminz,double dpz,int nz1
  ,const float *velx,const float *velz,unsigned np,const int *plist,const double *posz
  ,const typecode *code,float4 *velrhop,float velcorr)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<np){
    const unsigned p=plist[cp];
    if(izone==CODE_GetIzoneFluidInout(code[p])){
      const double pz=posz[p]-posminz;
      int cz=int(pz/dpz);
      cz=max(cz,0);
      cz=min(cz,nz1);
      const double fz=(pz/dpz-cz);  //const double fz=(pz-Dpz*cz)/Dpz;
      //-Interpolation in Z.
      const unsigned cp=cz;
      const float v00=velx[cp];
      const float v01=(cz<nz1? velx[cp+1]: v00);
      const float v=float(fz*(v01-v00)+v00);
      velrhop[p]=make_float4(v-velcorr,0,0,velrhop[p].w);
      if(velz!=NULL){
        const float v00=velz[cp];
        const float v01=(cz<nz1? velz[cp+1]:    v00);
        const float v=float(fz*(v01-v00)+v00);
        velrhop[p].z=v;
      }
    }
  }
}

//==============================================================================
/// Interpolate velocity in time and Z-position of selected partiles in a list.
//==============================================================================
void InOutInterpolateZVel(unsigned izone,double posminz,double dpz,int nz1
  ,const float *velx,const float *velz,unsigned np,const int *plist
  ,const double *posz,const typecode *code,float4 *velrhop,float velcorr)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    KerInOutInterpolateZVel <<<sgrid,SPHBSIZE>>> (izone,posminz,dpz,nz1,velx,velz,np,plist,posz,code,velrhop,velcorr);
  }
}

//------------------------------------------------------------------------------
/// Removes interpolated Z velocity of inlet/outlet particles.
//------------------------------------------------------------------------------
__global__ void KerInOutInterpolateResetZVel(unsigned izone,unsigned np,const int *plist
  ,const typecode *code,float4 *velrhop)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(cp<np){
    const unsigned p=plist[cp];
    if(izone==CODE_GetIzoneFluidInout(code[p]))velrhop[p].z=0;
  }
}

//==============================================================================
/// Removes interpolated Z velocity of inlet/outlet particles.
//==============================================================================
void InOutInterpolateResetZVel(unsigned izone,unsigned np,const int *plist
  ,const typecode *code,float4 *velrhop)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    KerInOutInterpolateResetZVel <<<sgrid,SPHBSIZE>>> (izone,np,plist,code,velrhop);
  }
}



}



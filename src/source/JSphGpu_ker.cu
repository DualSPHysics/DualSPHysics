#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2020 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public License 
 as published by the Free Software Foundation; either version 2.1 of the License, or (at your option) any later version.
 
 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more details. 

 You should have received a copy of the GNU Lesser General Public License along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JSphGpu_ker.cu \brief Implements functions and CUDA kernels for the Particle Interaction and System Update.

#include "JSphGpu_ker.h"
#include "Functions.h"
#include "FunctionsCuda.h"
#include "JLog2.h"
#include <cfloat>
#include <hip/hip_math_constants.h>
//:#include "JDgKerPrint.h"
//:#include "JDgKerPrint_ker.h"

#pragma warning(disable : 4267) //Cancels "warning C4267: conversion from 'size_t' to 'int', possible loss of data"
#pragma warning(disable : 4244) //Cancels "warning C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data"
#pragma warning(disable : 4503) //Cancels "warning C4503: decorated name length exceeded, name was truncated"
#include <thrust/device_vector.h>
#include <thrust/sort.h>

__constant__ StCteInteraction CTE;
#define CTE_AVAILABLE

namespace cusph{
#include "FunctionsBasic_iker.h"
#include "FunctionsMath_iker.h"
#include "FunSphKernel_iker.h"
#include "FunSphEos_iker.h"
#include "JCellSearch_iker.h"


//==============================================================================
/// Reduction using maximum of float values in shared memory for a warp.
/// Reduccion mediante maximo de valores float en memoria shared para un warp.
//==============================================================================
template <unsigned blockSize> __device__ void KerReduMaxFloatWarp(volatile float* sdat,unsigned tid){
  if(blockSize>=64)sdat[tid]=max(sdat[tid],sdat[tid+32]);
  if(blockSize>=32)sdat[tid]=max(sdat[tid],sdat[tid+16]);
  if(blockSize>=16)sdat[tid]=max(sdat[tid],sdat[tid+8]);
  if(blockSize>=8)sdat[tid]=max(sdat[tid],sdat[tid+4]);
  if(blockSize>=4)sdat[tid]=max(sdat[tid],sdat[tid+2]);
  if(blockSize>=2)sdat[tid]=max(sdat[tid],sdat[tid+1]);
}

//==============================================================================
/// Accumulates the maximum of n values of array dat[], storing the result in 
/// the beginning of res[].(Many positions of res[] are used as blocks, 
/// storing the final result in res[0]).
///
/// Acumula el maximo de n valores del vector dat[], guardando el resultado al 
/// principio de res[] (Se usan tantas posiciones del res[] como bloques, 
/// quedando el resultado final en res[0]).
//==============================================================================
template <unsigned blockSize> __global__ void KerReduMaxFloat(unsigned n,unsigned ini,const float *dat,float *res){
  extern __shared__ float sdat[];
  unsigned tid=threadIdx.x;
  unsigned c=blockIdx.x*blockDim.x + threadIdx.x;
  sdat[tid]=(c<n? dat[c+ini]: -FLT_MAX);
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sdat[tid]=max(sdat[tid],sdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sdat[tid]=max(sdat[tid],sdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sdat[tid]=max(sdat[tid],sdat[tid+64]);   __syncthreads(); }
  if(tid<32)KerReduMaxFloatWarp<blockSize>(sdat,tid);
  if(tid==0)res[blockIdx.x]=sdat[0];
}

//==============================================================================
/// Returns the maximum of an array, using resu[] as auxiliar array.
/// Size of resu[] must be >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE)
///
/// Devuelve el maximo de un vector, usando resu[] como vector auxiliar. El tamanho
/// de resu[] debe ser >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE)
//==============================================================================
float ReduMaxFloat(unsigned ndata,unsigned inidata,float* data,float* resu){
  float resf=0;
  if(ndata>=1){
    unsigned n=ndata,ini=inidata;
    unsigned smemSize=SPHBSIZE*sizeof(float);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    unsigned n_blocks=sgrid.x*sgrid.y;
    float *dat=data;
    float *resu1=resu,*resu2=resu+n_blocks;
    float *res=resu1;
    while(n>1){
      KerReduMaxFloat<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,dat,res);
      n=n_blocks; ini=0;
      sgrid=GetSimpleGridSize(n,SPHBSIZE);  
      n_blocks=sgrid.x*sgrid.y;
      if(n>1){
        dat=res; res=(dat==resu1? resu2: resu1); 
      }
    }
    if(ndata>1)hipMemcpy(&resf,res,sizeof(float),hipMemcpyDeviceToHost);
    else hipMemcpy(&resf,data,sizeof(float),hipMemcpyDeviceToHost);
  }
  //else{//-Using Thrust library is slower than ReduMasFloat() with ndata < 5M.
  //  thrust::device_ptr<float> dev_ptr(data);
  //  resf=thrust::reduce(dev_ptr,dev_ptr+ndata,-FLT_MAX,thrust::maximum<float>());
  //}
  return(resf);
}

//==============================================================================
/// Accumulates the sum of n values of array dat[], storing the result in 
/// the beginning of res[].(Many positions of res[] are used as blocks, 
/// storing the final result in res[0]).
///
/// Acumula la suma de n valores del vector dat[].w, guardando el resultado al 
/// principio de res[] (Se usan tantas posiciones del res[] como bloques, 
/// quedando el resultado final en res[0]).
//==============================================================================
template <unsigned blockSize> __global__ void KerReduMaxFloat_w(unsigned n,unsigned ini,const float4 *dat,float *res){
  extern __shared__ float sdat[];
  unsigned tid=threadIdx.x;
  unsigned c=blockIdx.x*blockDim.x + threadIdx.x;
  sdat[tid]=(c<n? dat[c+ini].w: -FLT_MAX);
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sdat[tid]=max(sdat[tid],sdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sdat[tid]=max(sdat[tid],sdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sdat[tid]=max(sdat[tid],sdat[tid+64]);   __syncthreads(); }
  if(tid<32)KerReduMaxFloatWarp<blockSize>(sdat,tid);
  if(tid==0)res[blockIdx.x]=sdat[0];
}

//==============================================================================
/// Returns the maximum of an array, using resu[] as auxiliar array.
/// Size of resu[] must be >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE).
///
/// Devuelve el maximo de la componente w de un vector float4, usando resu[] como 
/// vector auxiliar. El tamanho de resu[] debe ser >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE).
//==============================================================================
float ReduMaxFloat_w(unsigned ndata,unsigned inidata,float4* data,float* resu){
  unsigned n=ndata,ini=inidata;
  unsigned smemSize=SPHBSIZE*sizeof(float);
  dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
  unsigned n_blocks=sgrid.x*sgrid.y;
  float *dat=NULL;
  float *resu1=resu,*resu2=resu+n_blocks;
  float *res=resu1;
  while(n>1){
    if(!dat)KerReduMaxFloat_w<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,data,res);
    else KerReduMaxFloat<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,dat,res);
    n=n_blocks; ini=0;
    sgrid=GetSimpleGridSize(n,SPHBSIZE);  
    n_blocks=sgrid.x*sgrid.y;
    if(n>1){
      dat=res; res=(dat==resu1? resu2: resu1); 
    }
  }
  float resf;
  if(ndata>1)hipMemcpy(&resf,res,sizeof(float),hipMemcpyDeviceToHost);
  else{
    float4 resf4;
    hipMemcpy(&resf4,data,sizeof(float4),hipMemcpyDeviceToHost);
    resf=resf4.w;
  }
  return(resf);
}

//==============================================================================
/// Stores constants for the GPU interaction.
/// Graba constantes para la interaccion a la GPU.
//==============================================================================
void CteInteractionUp(const StCteInteraction *cte){
  hipMemcpyToSymbol(HIP_SYMBOL(CTE),cte,sizeof(StCteInteraction));
}

//------------------------------------------------------------------------------
/// Initialises array with the indicated value.
/// Inicializa array con el valor indicado.
//------------------------------------------------------------------------------
__global__ void KerInitArray(unsigned n,float3 *v,float3 value)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p]=value;
}

//==============================================================================
/// Initialises array with the indicated value.
/// Inicializa array con el valor indicado.
//==============================================================================
void InitArray(unsigned n,float3 *v,tfloat3 value){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerInitArray <<<sgrid,SPHBSIZE>>> (n,v,Float3(value));
  }
}

//------------------------------------------------------------------------------
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//------------------------------------------------------------------------------
__global__ void KerResety(unsigned n,unsigned ini,float3 *v)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p+ini].y=0;
}

//==============================================================================
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//==============================================================================
void Resety(unsigned n,unsigned ini,float3 *v){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerResety <<<sgrid,SPHBSIZE>>> (n,ini,v);
  }
}

//------------------------------------------------------------------------------
/// Calculates module^2 of ace.
//------------------------------------------------------------------------------
__global__ void KerComputeAceMod(unsigned n,const float3 *ace,float *acemod)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float3 r=ace[p];
    acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of ace.
//==============================================================================
void ComputeAceMod(unsigned n,const float3 *ace,float *acemod){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeAceMod <<<sgrid,SPHBSIZE>>> (n,ace,acemod);
  }
}

//------------------------------------------------------------------------------
/// Calculates module^2 of ace, comprobando que la particula sea normal.
/// Uses zero for periodic particles.
//------------------------------------------------------------------------------
__global__ void KerComputeAceMod(unsigned n,const typecode *code,const float3 *ace,float *acemod)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const typecode rcod=code[p];
    const float3 r=(CODE_IsNormal(rcod) && !CODE_IsFluidInout(rcod)? ace[p]: make_float3(0,0,0));
    acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of ace, comprobando que la particula sea normal.
/// Uses zero for periodic particles.
//==============================================================================
void ComputeAceMod(unsigned n,const typecode *code,const float3 *ace,float *acemod){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeAceMod <<<sgrid,SPHBSIZE>>> (n,code,ace,acemod);
  }
}


//##############################################################################
//# Other kernels...
//# Otros kernels...
//##############################################################################
//------------------------------------------------------------------------------
/// Calculates module^2 of vel.
//------------------------------------------------------------------------------
__global__ void KerComputeVelMod(unsigned n,const float4 *vel,float *velmod)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float4 r=vel[p];
    velmod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of vel.
//==============================================================================
void ComputeVelMod(unsigned n,const float4 *vel,float *velmod){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeVelMod <<<sgrid,SPHBSIZE>>> (n,vel,velmod);
  }
}


//##############################################################################
//# Kernels para cambiar la posicion.
//# Kernels for changing the position.
//##############################################################################
//------------------------------------------------------------------------------
/// Updates pos, dcell and code from the indicated displacement.
/// The code may be CODE_OUTRHOP because in ComputeStepVerlet / Symplectic this is evaluated
/// and is executed before ComputeStepPos.
/// Checks limits depending on maprealposmin and maprealsize, this is valid 
/// for single-GPU because maprealpos and domrealpos are equal. For multi-gpu it is
/// important to mark particles that leave the domain without leaving the map.
///
/// Actualiza pos, dcell y code a partir del desplazamiento indicado.
/// Code puede ser CODE_OUTRHOP pq en ComputeStepVerlet/Symplectic se evalua esto 
/// y se ejecuta antes que ComputeStepPos.
/// Comprueba los limites en funcion de maprealposmin y maprealsize esto es valido
/// para single-gpu pq domrealpos y maprealpos son iguales. Para multi-gpu seria 
/// necesario marcar las particulas q salgan del dominio sin salir del mapa.
//------------------------------------------------------------------------------
template<bool periactive> __device__ void KerUpdatePos
  (double2 rxy,double rz,double movx,double movy,double movz
  ,bool outrhop,unsigned p,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  //-Checks validity of displacement. | Comprueba validez del desplazamiento.
  const bool outmove=(fmaxf(fabsf(float(movx)),fmaxf(fabsf(float(movy)),fabsf(float(movz))))>CTE.movlimit);
  //-Applies diplacement.
  double3 rpos=make_double3(rxy.x,rxy.y,rz);
  rpos.x+=movx; rpos.y+=movy; rpos.z+=movz;
  if(rpos.y<0 && CTE.symmetry)rpos.y=-rpos.y; //<vs_syymmetry>
  //-Checks limits of real domain. | Comprueba limites del dominio reales.
  double dx=rpos.x-CTE.maprealposminx;
  double dy=rpos.y-CTE.maprealposminy;
  double dz=rpos.z-CTE.maprealposminz;
  bool out=(dx!=dx || dy!=dy || dz!=dz || dx<0 || dy<0 || dz<0 || dx>=CTE.maprealsizex || dy>=CTE.maprealsizey || dz>=CTE.maprealsizez);
  if(periactive && out){
    bool xperi=(CTE.periactive&1),yperi=(CTE.periactive&2),zperi=(CTE.periactive&4);
    if(xperi){
      if(dx<0)                { dx-=CTE.xperincx; dy-=CTE.xperincy; dz-=CTE.xperincz; }
      if(dx>=CTE.maprealsizex){ dx+=CTE.xperincx; dy+=CTE.xperincy; dz+=CTE.xperincz; }
    }
    if(yperi){
      if(dy<0)                { dx-=CTE.yperincx; dy-=CTE.yperincy; dz-=CTE.yperincz; }
      if(dy>=CTE.maprealsizey){ dx+=CTE.yperincx; dy+=CTE.yperincy; dz+=CTE.yperincz; }
    }
    if(zperi){
      if(dz<0)                { dx-=CTE.zperincx; dy-=CTE.zperincy; dz-=CTE.zperincz; }
      if(dz>=CTE.maprealsizez){ dx+=CTE.zperincx; dy+=CTE.zperincy; dz+=CTE.zperincz; }
    }
    bool outx=!xperi && (dx<0 || dx>=CTE.maprealsizex);
    bool outy=!yperi && (dy<0 || dy>=CTE.maprealsizey);
    bool outz=!zperi && (dz<0 || dz>=CTE.maprealsizez);
    out=(outx||outy||outz);
    rpos=make_double3(dx+CTE.maprealposminx,dy+CTE.maprealposminy,dz+CTE.maprealposminz);
  }
  //-Stores updated position.
  posxy[p]=make_double2(rpos.x,rpos.y);
  posz[p]=rpos.z;
  //-Stores cell and check. | Guarda celda y check.
  if(outrhop || outmove || out){//-Particle out. Only brands as excluded normal particles (not periodic). | Particle out. Solo las particulas normales (no periodicas) se pueden marcar como excluidas.
    typecode rcode=code[p];
    if(out)rcode=CODE_SetOutPos(rcode);
    else if(outrhop)rcode=CODE_SetOutRhop(rcode);
    else rcode=CODE_SetOutMove(rcode);
    code[p]=rcode;
    dcell[p]=PC__CodeMapOut;
  }
  else{//-Particle in.
    if(periactive){
      dx=rpos.x-CTE.domposminx;
      dy=rpos.y-CTE.domposminy;
      dz=rpos.z-CTE.domposminz;
    }
    const unsigned cx=unsigned(dx/CTE.scell);
    const unsigned cy=unsigned(dy/CTE.scell);
    const unsigned cz=unsigned(dz/CTE.scell);
    dcell[p]=PC__Cell(CTE.cellcode,cx,cy,cz);
  }
}

//------------------------------------------------------------------------------
/// Returns the corrected position after applying periodic conditions.
/// Devuelve la posicion corregida tras aplicar condiciones periodicas.
//------------------------------------------------------------------------------
__device__ double3 KerUpdatePeriodicPos(double3 ps)
{
  double dx=ps.x-CTE.maprealposminx;
  double dy=ps.y-CTE.maprealposminy;
  double dz=ps.z-CTE.maprealposminz;
  const bool out=(dx!=dx || dy!=dy || dz!=dz || dx<0 || dy<0 || dz<0 || dx>=CTE.maprealsizex || dy>=CTE.maprealsizey || dz>=CTE.maprealsizez);
  //-Adjusts position according to periodic conditions and rechecks domain limits.
  //-Ajusta posicion segun condiciones periodicas y vuelve a comprobar los limites del dominio.
  if(out){
    bool xperi=(CTE.periactive&1),yperi=(CTE.periactive&2),zperi=(CTE.periactive&4);
    if(xperi){
      if(dx<0)                { dx-=CTE.xperincx; dy-=CTE.xperincy; dz-=CTE.xperincz; }
      if(dx>=CTE.maprealsizex){ dx+=CTE.xperincx; dy+=CTE.xperincy; dz+=CTE.xperincz; }
    }
    if(yperi){
      if(dy<0)                { dx-=CTE.yperincx; dy-=CTE.yperincy; dz-=CTE.yperincz; }
      if(dy>=CTE.maprealsizey){ dx+=CTE.yperincx; dy+=CTE.yperincy; dz+=CTE.yperincz; }
    }
    if(zperi){
      if(dz<0)                { dx-=CTE.zperincx; dy-=CTE.zperincy; dz-=CTE.zperincz; }
      if(dz>=CTE.maprealsizez){ dx+=CTE.zperincx; dy+=CTE.zperincy; dz+=CTE.zperincz; }
    }
    ps=make_double3(dx+CTE.maprealposminx,dy+CTE.maprealposminy,dz+CTE.maprealposminz);
  }
  return(ps);
}


//##############################################################################
//# Kernels for calculating forces (Pos-Double).
//# Kernels para calculo de fuerzas (Pos-Double).
//##############################################################################
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. Bound-Fluid/Float
/// Realiza la interaccion de una particula con un conjunto de ellas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool symm>
  __device__ void KerInteractionForcesBoundBox
  (unsigned p1,const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned* idp
  ,float massf,const float4 &pscellp1,const float4 &velrhop1,float &arp1,float &visc)
{
  for(int p2=pini;p2<pfin;p2++){
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x + CTE.poscellsize*(CEL_GetfX(pscellp1.w)-CEL_GetfX(pscellp2.w));
    float dry=pscellp1.y-pscellp2.y + CTE.poscellsize*(CEL_GetfY(pscellp1.w)-CEL_GetfY(pscellp2.w));
    float drz=pscellp1.z-pscellp2.z + CTE.poscellsize*(CEL_GetfZ(pscellp1.w)-CEL_GetfZ(pscellp2.w));
    if(symm)dry=pscellp1.y+pscellp2.y + CTE.poscellsize*CEL_GetfY(pscellp2.w); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO){
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>
      //-Obtains particle mass p2 if there are floating bodies.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massf si es fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING){
        const typecode cod=code[p2];
        bool ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2? ftomassp[CODE_GetTypeValue(cod)]: massf);
        compute=!(USE_FTEXTERNAL && ftp2); //-Deactivated when DEM or Chrono is used and is bound-float. | Se desactiva cuando se usa DEM o Chrono y es bound-float.
      }

      if(compute){
        //-Density derivative (Continuity equation).
        const float dvx=velrhop1.x-velrhop2.x, dvy=velrhop1.y-velrhop2.y, dvz=velrhop1.z-velrhop2.z;
        arp1+=(USE_FLOATING? ftmassp2: massf)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);

        {//===== Viscosity ===== 
          const float dot=drx*dvx + dry*dvy + drz*dvz;
          const float dot_rr2=dot/(rr2+CTE.eta2);
          visc=max(dot_rr2,visc); 
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Particle interaction. Bound-Fluid/Float
/// Realiza interaccion entre particulas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool symm> 
  __global__ void KerInteractionForcesBound(unsigned n,unsigned pinit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid,const unsigned *dcell
  ,const float *ftomassp
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of thread.
  if(p<n){
    const unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0;

    //-Loads particle p1 data.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>
    
    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Boundary-Fluid interaction.
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
      if(pfin){
                          KerInteractionForcesBoundBox<tker,ftmode,false> (p1,pini,pfin,ftomassp,poscell,velrhop,code,idp,CTE.massf,pscellp1,velrhop1,arp1,visc);
        if(symm && rsymp1)KerInteractionForcesBoundBox<tker,ftmode,true > (p1,pini,pfin,ftomassp,poscell,velrhop,code,idp,CTE.massf,pscellp1,velrhop1,arp1,visc);
      }
    }
    //-Stores results.
    if(arp1 || visc){
      ar[p1]+=arp1;
      if(visc>viscdt[p1])viscdt[p1]=visc;
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool lamsps,TpDensity tdensity,bool shift,bool symm>
  __device__ void KerInteractionForcesFluidBox(bool boundp2,unsigned p1
  ,const unsigned &pini,const unsigned &pfin,float visco
  ,const float *ftomassp,const float2 *tauff
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float massp2,bool ftp1
  ,const float4 &pscellp1,const float4 &velrhop1,float pressp1
  ,const float2 &taup1_xx_xy,const float2 &taup1_xz_yy,const float2 &taup1_yz_zz
  ,float2 &grap1_xx_xy,float2 &grap1_xz_yy,float2 &grap1_yz_zz
  ,float3 &acep1,float &arp1,float &visc,float &deltap1
  ,TpShifting shiftmode,float4 &shiftposfsp1)
{
  for(int p2=pini;p2<pfin;p2++){
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x + CTE.poscellsize*(CEL_GetfX(pscellp1.w)-CEL_GetfX(pscellp2.w));
    float dry=pscellp1.y-pscellp2.y + CTE.poscellsize*(CEL_GetfY(pscellp1.w)-CEL_GetfY(pscellp2.w));
    float drz=pscellp1.z-pscellp2.z + CTE.poscellsize*(CEL_GetfZ(pscellp1.w)-CEL_GetfZ(pscellp2.w));
    if(symm)dry=pscellp1.y+pscellp2.y + CTE.poscellsize*CEL_GetfY(pscellp2.w); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO){
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 if any floating bodies exist.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2=false;         //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING){
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2? ftomassp[CODE_GetTypeValue(cod)]: massp2);
        #ifdef DELTA_HEAVYFLOATING
          if(ftp2 && tdensity==DDT_DDT && ftmassp2<=(massp2*1.2f))deltap1=FLT_MAX;
        #else
          if(ftp2 && tdensity==DDT_DDT)deltap1=FLT_MAX;
        #endif
        if(ftp2 && shift && shiftmode==SHIFT_NoBound)shiftposfsp1.x=FLT_MAX; //-Cancels shifting with floating bodies. | Con floatings anula shifting.
        compute=!(USE_FTEXTERNAL && ftp1 && (boundp2 || ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>
      
      //-Velocity derivative (Momentum equation).
      if(compute){
        const float pressp2=cufsph::ComputePressCte(velrhop2.w);
        const float prs=(pressp1+pressp2)/(velrhop1.w*velrhop2.w)
          +(tker==KERNEL_Cubic? cufsph::GetKernelCubic_Tensil(rr2,velrhop1.w,pressp1,velrhop2.w,pressp2): 0);
        const float p_vpm=-prs*(USE_FLOATING? ftmassp2: massp2);
        acep1.x+=p_vpm*frx; acep1.y+=p_vpm*fry; acep1.z+=p_vpm*frz;
      }

      //-Density derivative (Continuity equation).
      const float dvx=velrhop1.x-velrhop2.x, dvy=velrhop1.y-velrhop2.y, dvz=velrhop1.z-velrhop2.z;
      if(compute)arp1+=(USE_FLOATING? ftmassp2: massp2)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);

      const float cbar=CTE.cs0;
      const float dot3=(tdensity!=DDT_None || shift? drx*frx+dry*fry+drz*frz: 0);
      //-Density Diffusion Term (Molteni and Colagrossi 2009).
      if(tdensity==DDT_DDT && deltap1!=FLT_MAX){
        const float rhop1over2=velrhop1.w/velrhop2.w;
        const float visc_densi=CTE.ddtkh*cbar*(rhop1over2-1.f)/(rr2+CTE.eta2);
        const float delta=visc_densi*dot3*(USE_FLOATING? ftmassp2: massp2);
        //deltap1=(boundp2? FLT_MAX: deltap1+delta);
        deltap1=(boundp2 && CTE.tboundary==BC_DBC? FLT_MAX: deltap1+delta);
      }
      //-Density Diffusion Term (Fourtakas et al 2019).
      if((tdensity==DDT_DDT2 || (tdensity==DDT_DDT2Full && !boundp2)) && deltap1!=FLT_MAX && !ftp2){
        const float rh=1.f+CTE.ddtgz*drz;
        const float drhop=CTE.rhopzero*pow(rh,1.f/CTE.gamma)-CTE.rhopzero;  
        const float visc_densi=CTE.ddtkh*cbar*((velrhop2.w-velrhop1.w)-drhop)/(rr2+CTE.eta2);
        const float delta=visc_densi*dot3*massp2/velrhop2.w;
        deltap1=(boundp2? FLT_MAX: deltap1-delta); //-blocks it makes it boil - bloody DBC
      }

      //-Shifting correction.
      if(shift && shiftposfsp1.x!=FLT_MAX){
        const float massrhop=(USE_FLOATING? ftmassp2: massp2)/velrhop2.w;
        const bool noshift=(boundp2 && (shiftmode==SHIFT_NoBound || (shiftmode==SHIFT_NoFixed && CODE_IsFixed(code[p2]))));
        shiftposfsp1.x=(noshift? FLT_MAX: shiftposfsp1.x+massrhop*frx); //-Removes shifting for the boundaries. | Con boundary anula shifting.
        shiftposfsp1.y+=massrhop*fry;
        shiftposfsp1.z+=massrhop*frz;
        shiftposfsp1.w-=massrhop*dot3;
      }

      //===== Viscosity ===== 
      if(compute){
        const float dot=drx*dvx + dry*dvy + drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);
        if(!lamsps){//-Artificial viscosity.
          if(dot<0){
            const float amubar=CTE.kernelh*dot_rr2;  //amubar=CTE.kernelh*dot/(rr2+CTE.eta2);
            const float robar=(velrhop1.w+velrhop2.w)*0.5f;
            const float pi_visc=(-visco*cbar*amubar/robar)*(USE_FLOATING? ftmassp2: massp2);
            acep1.x-=pi_visc*frx; acep1.y-=pi_visc*fry; acep1.z-=pi_visc*frz;
          }
        }
        else{//-Laminar+SPS viscosity.
          {//-Laminar contribution.
            const float robar2=(velrhop1.w+velrhop2.w);
            const float temp=4.f*visco/((rr2+CTE.eta2)*robar2);  //-Simplication of temp=2.0f*visco/((rr2+CTE.eta2)*robar); robar=(rhopp1+velrhop2.w)*0.5f;
            const float vtemp=(USE_FLOATING? ftmassp2: massp2)*temp*(drx*frx+dry*fry+drz*frz);  
            acep1.x+=vtemp*dvx; acep1.y+=vtemp*dvy; acep1.z+=vtemp*dvz;
          }
          //-SPS turbulence model.
          float2 taup2_xx_xy=taup1_xx_xy; //-taup1 is always zero when p1 is not fluid. | taup1 siempre es cero cuando p1 no es fluid.
          float2 taup2_xz_yy=taup1_xz_yy;
          float2 taup2_yz_zz=taup1_yz_zz;
          if(!boundp2 && (USE_NOFLOATING || !ftp2)){//-When p2 is fluid.
            float2 taup2=tauff[p2*3];     taup2_xx_xy.x+=taup2.x; taup2_xx_xy.y+=taup2.y;
                   taup2=tauff[p2*3+1];   taup2_xz_yy.x+=taup2.x; taup2_xz_yy.y+=taup2.y;
                   taup2=tauff[p2*3+2];   taup2_yz_zz.x+=taup2.x; taup2_yz_zz.y+=taup2.y;
          }
          acep1.x+=(USE_FLOATING? ftmassp2: massp2)*(taup2_xx_xy.x*frx+taup2_xx_xy.y*fry+taup2_xz_yy.x*frz);
          acep1.y+=(USE_FLOATING? ftmassp2: massp2)*(taup2_xx_xy.y*frx+taup2_xz_yy.y*fry+taup2_yz_zz.x*frz);
          acep1.z+=(USE_FLOATING? ftmassp2: massp2)*(taup2_xz_yy.x*frx+taup2_yz_zz.x*fry+taup2_yz_zz.y*frz);
          //-Velocity gradients.
          if(USE_NOFLOATING || !ftp1){//-When p1 is fluid.
            const float volp2=-(USE_FLOATING? ftmassp2: massp2)/velrhop2.w;
            float dv=dvx*volp2; grap1_xx_xy.x+=dv*frx; grap1_xx_xy.y+=dv*fry; grap1_xz_yy.x+=dv*frz;
                  dv=dvy*volp2; grap1_xx_xy.y+=dv*frx; grap1_xz_yy.y+=dv*fry; grap1_yz_zz.x+=dv*frz;
                  dv=dvz*volp2; grap1_xz_yy.x+=dv*frx; grap1_yz_zz.x+=dv*fry; grap1_yz_zz.y+=dv*frz;
            // to compute tau terms we assume that gradvel.xy=gradvel.dudy+gradvel.dvdx, gradvel.xz=gradvel.dudz+gradvel.dwdx, gradvel.yz=gradvel.dvdz+gradvel.dwdy
            // so only 6 elements are needed instead of 3x3.
          }
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,bool lamsps,TpDensity tdensity,bool shift,bool symm>
  __global__ void KerInteractionForcesFluid(unsigned n,unsigned pinit,float viscob,float viscof
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float *ftomassp,const float2 *tauff,float2 *gradvelff
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar,float3 *ace,float *delta
  ,TpShifting shiftmode,float4 *shiftposfs)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0,deltap1=0;
    float3 acep1=make_float3(0,0,0);

    //-Variables for Shifting.
    float4 shiftposfsp1;
    if(shift)shiftposfsp1=shiftposfs[p1];

    //-Obtains data of particle p1 in case there are floating bodies.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    if(USE_FLOATING){
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
      if(ftp1 && tdensity!=DDT_None)deltap1=FLT_MAX; //-DDT is not applied to floating particles.
      if(ftp1 && shift)shiftposfsp1.x=FLT_MAX; //-Shifting is not calculated for floating bodies. | Para floatings no se calcula shifting.
    }

    //-Obtains basic data of particle p1.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const float pressp1=cufsph::ComputePressCte(velrhop1.w);
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //-Variables for Laminar+SPS.
    float2 taup1_xx_xy,taup1_xz_yy,taup1_yz_zz;
    if(lamsps){
      taup1_xx_xy=tauff[p1*3];
      taup1_xz_yy=tauff[p1*3+1];
      taup1_yz_zz=tauff[p1*3+2];
    }
    //-Variables for Laminar+SPS (computation).
    float2 grap1_xx_xy,grap1_xz_yy,grap1_yz_zz;
    if(lamsps){
      grap1_xx_xy=make_float2(0,0);
      grap1_xz_yy=make_float2(0,0);
      grap1_yz_zz=make_float2(0,0);
    }

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin){
                          KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,false> (false,p1,pini,pfin,viscof,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1);
        if(symm && rsymp1)KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,true > (false,p1,pini,pfin,viscof,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1); //<vs_syymmetry>
      }
    }
    //-Interaction with boundaries.
    ini3-=cellfluid; fin3-=cellfluid;
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin){
                        KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,false> (true ,p1,pini,pfin,viscob,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massb,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1);
      if(symm && rsymp1)KerInteractionForcesFluidBox<tker,ftmode,lamsps,tdensity,shift,true > (true ,p1,pini,pfin,viscob,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massb,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1);
      }
    }

    //-Stores results.
    if(shift||arp1||acep1.x||acep1.y||acep1.z||visc){
      if(tdensity!=DDT_None){
        if(delta){
          const float rdelta=delta[p1];
          delta[p1]=(rdelta==FLT_MAX || deltap1==FLT_MAX? FLT_MAX: rdelta+deltap1);
        }
        else if(deltap1!=FLT_MAX)arp1+=deltap1;
      }
      ar[p1]+=arp1;
      float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      if(visc>viscdt[p1])viscdt[p1]=visc;
      if(lamsps){
        float2 rg;
        rg=gradvelff[p1*3  ];  rg=make_float2(rg.x+grap1_xx_xy.x,rg.y+grap1_xx_xy.y);  gradvelff[p1*3  ]=rg;
        rg=gradvelff[p1*3+1];  rg=make_float2(rg.x+grap1_xz_yy.x,rg.y+grap1_xz_yy.y);  gradvelff[p1*3+1]=rg;
        rg=gradvelff[p1*3+2];  rg=make_float2(rg.x+grap1_yz_zz.x,rg.y+grap1_yz_zz.y);  gradvelff[p1*3+2]=rg;
      }
      if(shift)shiftposfs[p1]=shiftposfsp1;
    }
  }
}

#ifndef DISABLE_BSMODES
//==============================================================================
/// Collects kernel information.
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,bool lamsps,TpDensity tdensity,bool shift,bool symm> 
  void Interaction_ForcesT_KerInfo(StKerInfo *kerinfo)
{
 #if CUDART_VERSION >= 6050
  {
    typedef void (*fun_ptr)(unsigned,unsigned,float,float,int,int4,int3,const int2*,unsigned,const unsigned*,const float*,const float2*,float2*,const float4*,const float4*,const typecode*,const unsigned*,float*,float*,float3*,float*,TpShifting,float4*);
    fun_ptr ptr=&KerInteractionForcesFluid<tker,ftmode,lamsps,tdensity,shift,symm>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesfluid_bs=qblocksize;
    kerinfo->forcesfluid_rg=attr.numRegs;
    kerinfo->forcesfluid_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesFluid  blocksize:%u (%u)\n",qblocksize,0);
  }
  {
    typedef void (*fun_ptr)(unsigned,unsigned,int,int4,int3,const int2*,const unsigned*,const float*,const float4*,const float4*,const typecode*,const unsigned*,float*,float*);
    fun_ptr ptr=&KerInteractionForcesBound<tker,ftmode,symm>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesbound_bs=qblocksize;
    kerinfo->forcesbound_rg=attr.numRegs;
    kerinfo->forcesbound_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesBound  blocksize:%u (%u)\n",qblocksize,0);
  }
  fcuda::Check_CudaErroorFun("Error collecting kernel information.");
 #endif
}
#endif

//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,bool lamsps,TpDensity tdensity,bool shift> 
  void Interaction_ForcesGpuT(const StInterParmsg &t)
{
  //-Collects kernel information.
#ifndef DISABLE_BSMODES
  if(t.kerinfo){
    Interaction_ForcesT_KerInfo<tker,ftmode,lamsps,tdensity,shift,false>(t.kerinfo);
    return;
  }
#endif
  const StDivDataGpu &dvd=t.divdatag;
  const int2* beginendcell=dvd.beginendcell;
  //-Interaction Fluid-Fluid & Fluid-Bound.
  if(t.fluidnum){
    dim3 sgridf=GetSimpleGridSize(t.fluidnum,t.bsfluid);
    if(t.symmetry) //<vs_syymmetry_ini>
      KerInteractionForcesFluid<tker,ftmode,lamsps,tdensity,shift,true> <<<sgridf,t.bsfluid,0,t.stm>>> 
      (t.fluidnum,t.fluidini,t.viscob,t.viscof,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
      ,t.ftomassp,(const float2*)t.tau,(float2*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp
      ,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs);
    else //<vs_syymmetry_end>
      KerInteractionForcesFluid<tker,ftmode,lamsps,tdensity,shift,false> <<<sgridf,t.bsfluid,0,t.stm>>> 
      (t.fluidnum,t.fluidini,t.viscob,t.viscof,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
      ,t.ftomassp,(const float2*)t.tau,(float2*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp
      ,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs);
      //KerInteractionForcesFluid<tker,ftmode,lamsps,tdensity,shift,false> <<<sgridf,t.bsfluid,0,t.stm>>> (t.fluidnum,t.fluidini,t.scelldiv,t.nc,t.cellfluid,t.viscob,t.viscof,t.begincell,Int3(t.cellmin),t.dcell,t.ftomassp,(const float2*)t.tau,(float2*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs);
  }
  //-Interaction Boundary-Fluid.
  if(t.boundnum){
    const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
    dim3 sgridb=GetSimpleGridSize(t.boundnum,t.bsbound);
    //printf("bsbound:%u\n",bsbound);
    if(t.symmetry) //<vs_syymmetry_ini>
      KerInteractionForcesBound<tker,ftmode,true > <<<sgridb,t.bsbound,0,t.stm>>> 
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcell+dvd.cellfluid,t.dcell
        ,t.ftomassp,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar);
    else //<vs_syymmetry_end>
      KerInteractionForcesBound<tker,ftmode,false> <<<sgridb,t.bsbound,0,t.stm>>> 
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,t.dcell
        ,t.ftomassp,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar);
  }
}

//==============================================================================
//#define FAST_COMPILATION
template<TpKernel tker,TpFtMode ftmode,bool lamsps> void Interaction_Forces_gt2(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.shiftmode || t.tdensity!=DDT_DDT2Full)throw "Shifting and extra DDT are disabled for FastCompilation...";
  Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT2Full,false> (t);
#else
  if(t.shiftmode){               const bool shift=true;
    if(t.tdensity==DDT_None)    Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_None    ,shift> (t);
    if(t.tdensity==DDT_DDT)     Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT     ,shift> (t);
    if(t.tdensity==DDT_DDT2)    Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT2    ,shift> (t);
    if(t.tdensity==DDT_DDT2Full)Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT2Full,shift> (t);
  }
  else{                           const bool shift=false;
    if(t.tdensity==DDT_None)    Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_None    ,shift> (t);
    if(t.tdensity==DDT_DDT)     Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT     ,shift> (t);
    if(t.tdensity==DDT_DDT2)    Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT2    ,shift> (t);
    if(t.tdensity==DDT_DDT2Full)Interaction_ForcesGpuT<tker,ftmode,lamsps,DDT_DDT2Full,shift> (t);
  }
#endif
}
//==============================================================================
template<TpKernel tker,TpFtMode ftmode> void Interaction_Forces_gt1(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.lamsps)throw "Extra viscosity options are disabled for FastCompilation...";
  Interaction_Forces_gt2<tker,ftmode,false> (t);
#else
  if(t.lamsps)Interaction_Forces_gt2<tker,ftmode,true>  (t);
  else        Interaction_Forces_gt2<tker,ftmode,false> (t);
#endif
}
//==============================================================================
template<TpKernel tker> void Interaction_Forces_gt0(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.ftmode!=FTMODE_None)throw "Extra FtMode options are disabled for FastCompilation...";
  Interaction_Forces_gt1<tker,FTMODE_None> (t);
#else
  if(t.ftmode==FTMODE_None)    Interaction_Forces_gt1<tker,FTMODE_None> (t);
  else if(t.ftmode==FTMODE_Sph)Interaction_Forces_gt1<tker,FTMODE_Sph>  (t);
  else if(t.ftmode==FTMODE_Ext)Interaction_Forces_gt1<tker,FTMODE_Ext>  (t);
#endif
}
//==============================================================================
void Interaction_Forces(const StInterParmsg &t){
#ifdef FAST_COMPILATION
  if(t.tkernel!=KERNEL_Wendland)throw "Extra kernels are disabled for FastCompilation...";
  Interaction_Forces_gt0<KERNEL_Wendland> (t);
#else
  if(t.tkernel==KERNEL_Wendland)     Interaction_Forces_gt0<KERNEL_Wendland> (t);
 #ifndef DISABLE_KERNELS_EXTRA
  else if(t.tkernel==KERNEL_Cubic)   Interaction_Forces_gt0<KERNEL_Cubic   > (t);
 #endif
#endif
}

//------------------------------------------------------------------------------
/// Returns the corrected position after applying periodic conditions.
/// Devuelve la posicion corregida tras aplicar condiciones periodicas.
//------------------------------------------------------------------------------
__device__ float4 KerComputePosCell(const double3 &ps,const double3 &mapposmin,float poscellsize)
{
  const double dx=ps.x-mapposmin.x;
  const double dy=ps.y-mapposmin.y;
  const double dz=ps.z-mapposmin.z;
  int cx=int(dx/poscellsize);
  int cy=int(dy/poscellsize);
  int cz=int(dz/poscellsize);
  cx=(cx>=0? cx: 0);
  cy=(cy>=0? cy: 0);
  cz=(cz>=0? cz: 0);
  const float px=float(dx-(double(poscellsize)*cx));
  const float py=float(dy-(double(poscellsize)*cy));
  const float pz=float(dz-(double(poscellsize)*cz));
  const float pw=__uint_as_float(CEL_Code(cx,cy,cz));
  return(make_float4(px,py,pz,pw));
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected bondary and fluid.
//------------------------------------------------------------------------------
template<TpKernel tker,bool sim2d,TpSlipMode tslip> __global__ void KerInteractionMdbcCorrection_Fast
  (unsigned n,unsigned nbound,float determlimit,float mdbcthreshold
  ,double3 mapposmin,float poscellsize,const float4 *poscell
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  const unsigned p1=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<n){
    const float3 bnormalp1=boundnormal[p1];
    if(bnormalp1.x!=0 || bnormalp1.y!=0 || bnormalp1.z!=0){
      float rhopfinal=FLT_MAX;
      float3 velrhopfinal=make_float3(0,0,0);
      float sumwab=0;

      //-Calculates ghost node position.
      double3 gposp1=make_double3(posxy[p1].x+bnormalp1.x,posxy[p1].y+bnormalp1.y,posz[p1]+bnormalp1.z);
      gposp1=(CTE.periactive!=0? KerUpdatePeriodicPos(gposp1): gposp1); //-Corrected interface Position.
      const float4 gpscellp1=KerComputePosCell(gposp1,mapposmin,poscellsize);

      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);                              // -Only for velocity
      tmatrix3f a_corr2; if(sim2d) cumath::Tmatrix3fReset(a_corr2); //-Only for 2D.
      tmatrix4f a_corr3; if(!sim2d)cumath::Tmatrix4fReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(gposp1.x,gposp1.y,gposp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Boundary-Fluid interaction.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const float4 pscellp2=poscell[p2];
          float drx=gpscellp1.x-pscellp2.x + CTE.poscellsize*(CEL_GetfX(gpscellp1.w)-CEL_GetfX(pscellp2.w));
          float dry=gpscellp1.y-pscellp2.y + CTE.poscellsize*(CEL_GetfY(gpscellp1.w)-CEL_GetfY(pscellp2.w));
          float drz=gpscellp1.z-pscellp2.z + CTE.poscellsize*(CEL_GetfZ(gpscellp1.w)-CEL_GetfZ(pscellp2.w));
          const float rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
            //-Computes kernel.
            float fac;
            const float wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

            //===== Get mass and volume of particle p2 =====
            const float4 velrhopp2=velrhop[p2];
            float massp2=CTE.massf;
            const float volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const float vwab=wab*volp2;
            sumwab+=vwab;
            const float vfrx=frx*volp2;
            const float vfry=fry*volp2;
            const float vfrz=frz*volp2;

            //===== Velocity =====
            if(tslip!=SLIP_Vel0) {
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      if(sumwab>=mdbcthreshold){
        const float3 dpos=make_float3(-bnormalp1.x,-bnormalp1.y,-bnormalp1.z); //-Boundary particle position - ghost node position.
        if(sim2d){
          const double determ=cumath::Determinant3x3dbl(a_corr2);
          if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
            const tmatrix3f invacorr2=cumath::InverseMatrix3x3dbl(a_corr2,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr2.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
        else{
          const double determ=cumath::Determinant4x4dbl(a_corr3);
          if(fabs(determ)>=determlimit){
            const tmatrix4f invacorr3=cumath::InverseMatrix4x4dbl(a_corr3,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr3.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
        //-Store the results.
        rhopfinal=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
        if(tslip==SLIP_Vel0){//-DBC vel=0
          velrhop[p1].w=rhopfinal;
        }
        if(tslip==SLIP_NoSlip){//-No-Slip
          const float3 v=motionvel[p1];
          velrhop[p1]=make_float4(v.x+v.x-velrhopfinal.x,v.y+v.y-velrhopfinal.y,v.z+v.z-velrhopfinal.z,rhopfinal);
        }
        if(tslip==SLIP_FreeSlip){//-No-Penetration and free slip    SHABA
          float3 FSVelFinal; // final free slip boundary velocity
          const float3 v=motionvel[p1];
          float motion=sqrt(v.x*v.x+v.y*v.y+v.z*v.z); // to check if boundary moving
          float norm=sqrt(bnormalp1.x*bnormalp1.x+bnormalp1.y*bnormalp1.y+bnormalp1.z*bnormalp1.z);
          float3 normal; // creating a normailsed boundary normal
          normal.x=fabs(bnormalp1.x)/norm; normal.y=fabs(bnormalp1.y)/norm; normal.z=fabs(bnormalp1.z)/norm;

          // finding the velocity componants normal and tangential to boundary 
          float3 normvel=make_float3(velrhopfinal.x*normal.x,velrhopfinal.y*normal.y,velrhopfinal.z*normal.z); // velocity in direction of normal pointin ginto fluid)
          float3 tangvel=make_float3(velrhopfinal.x-normvel.x,velrhopfinal.y-normvel.y,velrhopfinal.z-normvel.z); // velocity tangential to normal

          if (motion > 0) { // if moving boundary
              float3 normmot=make_float3(v.x*normal.x,v.y*normal.y,v.z*normal.z); // boundary motion in direction normal to boundary 
              FSVelFinal=make_float3(normmot.x+normmot.x-normvel.x,normmot.y+normmot.y-normvel.y,normmot.z+normmot.z-normvel.z);
              // only velocity in normal direction for no-penetration
              // fluid sees zero velocity in the tangetial direction
          }
          else {
              FSVelFinal=make_float3(tangvel.x-normvel.x,tangvel.y-normvel.y,tangvel.z-normvel.z);
              // tangential velocity equal to fluid velocity for free slip
              // normal velocity reversed for no-penetration
          }

          // Save the velocity and density
          velrhop[p1]=make_float4(FSVelFinal.x,FSVelFinal.y,FSVelFinal.z,rhopfinal);
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Perform interaction between ghost node of selected bondary and fluid.
//------------------------------------------------------------------------------
template<TpKernel tker,bool sim2d,TpSlipMode tslip> __global__ void KerInteractionMdbcCorrection_Dbl
  (unsigned n,unsigned nbound,float determlimit,float mdbcthreshold
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid
  ,const double2 *posxy,const double *posz,const typecode *code,const unsigned *idp
  ,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  const unsigned p1=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<n){
    const float3 bnormalp1=boundnormal[p1];
    if(bnormalp1.x!=0 || bnormalp1.y!=0 || bnormalp1.z!=0){
      float rhopfinal=FLT_MAX;
      float3 velrhopfinal=make_float3(0,0,0);
      float sumwab=0;

      //-Calculates ghost node position.
      double3 gposp1=make_double3(posxy[p1].x+bnormalp1.x,posxy[p1].y+bnormalp1.y,posz[p1]+bnormalp1.z);
      gposp1=(CTE.periactive!=0? KerUpdatePeriodicPos(gposp1): gposp1); //-Corrected interface Position.
      //-Initializes variables for calculation.
      float rhopp1=0;
      float3 gradrhopp1=make_float3(0,0,0);
      float3 velp1=make_float3(0,0,0);                              // -Only for velocity
      tmatrix3d a_corr2; if(sim2d) cumath::Tmatrix3dReset(a_corr2); //-Only for 2D.
      tmatrix4d a_corr3; if(!sim2d)cumath::Tmatrix4dReset(a_corr3); //-Only for 3D.
    
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(gposp1.x,gposp1.y,gposp1.z,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Boundary-Fluid interaction.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const double2 p2xy=posxy[p2];
          const float drx=float(gposp1.x-p2xy.x);
          const float dry=float(gposp1.y-p2xy.y);
          const float drz=float(gposp1.z-posz[p2]);
          const float rr2=drx*drx+dry*dry+drz*drz;
          if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){//-Only with fluid particles (including inout).
            //-Computes kernel.
            float fac;
            const float wab=cufsph::GetKernel_WabFac<tker>(rr2,fac);
            const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

            //===== Get mass and volume of particle p2 =====
            const float4 velrhopp2=velrhop[p2];
            float massp2=CTE.massf;
            const float volp2=massp2/velrhopp2.w;

            //===== Density and its gradient =====
            rhopp1+=massp2*wab;
            gradrhopp1.x+=massp2*frx;
            gradrhopp1.y+=massp2*fry;
            gradrhopp1.z+=massp2*frz;

            //===== Kernel values multiplied by volume =====
            const float vwab=wab*volp2;
            sumwab+=vwab;
            const float vfrx=frx*volp2;
            const float vfry=fry*volp2;
            const float vfrz=frz*volp2;

            //===== Velocity =====
            if(tslip!=SLIP_Vel0) {
              velp1.x+=vwab*velrhopp2.x;
              velp1.y+=vwab*velrhopp2.y;
              velp1.z+=vwab*velrhopp2.z;
            }

            //===== Matrix A for correction =====
            if(sim2d){
              a_corr2.a11+=vwab;  a_corr2.a12+=drx*vwab;  a_corr2.a13+=drz*vwab;
              a_corr2.a21+=vfrx;  a_corr2.a22+=drx*vfrx;  a_corr2.a23+=drz*vfrx;
              a_corr2.a31+=vfrz;  a_corr2.a32+=drx*vfrz;  a_corr2.a33+=drz*vfrz;
            }
            else{
              a_corr3.a11+=vwab;  a_corr3.a12+=drx*vwab;  a_corr3.a13+=dry*vwab;  a_corr3.a14+=drz*vwab;
              a_corr3.a21+=vfrx;  a_corr3.a22+=drx*vfrx;  a_corr3.a23+=dry*vfrx;  a_corr3.a24+=drz*vfrx;
              a_corr3.a31+=vfry;  a_corr3.a32+=drx*vfry;  a_corr3.a33+=dry*vfry;  a_corr3.a34+=drz*vfry;
              a_corr3.a41+=vfrz;  a_corr3.a42+=drx*vfrz;  a_corr3.a43+=dry*vfrz;  a_corr3.a44+=drz*vfrz;
            }
          }
        }
      }

      //-Store the results.
      //--------------------
      if(sumwab>=mdbcthreshold){
        const float3 dpos=make_float3(-bnormalp1.x,-bnormalp1.y,-bnormalp1.z); //-Boundary particle position - ghost node position.
        if(sim2d){
          const double determ=cumath::Determinant3x3(a_corr2);
          if(fabs(determ)>=determlimit){//-Use 1e-3f (first_order) or 1e+3f (zeroth_order).
            const tmatrix3d invacorr2=cumath::InverseMatrix3x3(a_corr2,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr2.a11*rhopp1 + invacorr2.a12*gradrhopp1.x + invacorr2.a13*gradrhopp1.z);
            const float grx=    -float(invacorr2.a21*rhopp1 + invacorr2.a22*gradrhopp1.x + invacorr2.a23*gradrhopp1.z);
            const float grz=    -float(invacorr2.a31*rhopp1 + invacorr2.a32*gradrhopp1.x + invacorr2.a33*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + grz*dpos.z);
          }
          else if(a_corr2.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr2.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr2.a11);
            velrhopfinal.z=float(velp1.z/a_corr2.a11);
            velrhopfinal.y=0;
          }
        }
        else{
          const double determ=cumath::Determinant4x4(a_corr3);
          if(fabs(determ)>=determlimit){
            const tmatrix4d invacorr3=cumath::InverseMatrix4x4(a_corr3,determ);
            //-GHOST NODE DENSITY IS MIRRORED BACK TO THE BOUNDARY PARTICLES.
            const float rhoghost=float(invacorr3.a11*rhopp1 + invacorr3.a12*gradrhopp1.x + invacorr3.a13*gradrhopp1.y + invacorr3.a14*gradrhopp1.z);
            const float grx=    -float(invacorr3.a21*rhopp1 + invacorr3.a22*gradrhopp1.x + invacorr3.a23*gradrhopp1.y + invacorr3.a24*gradrhopp1.z);
            const float gry=    -float(invacorr3.a31*rhopp1 + invacorr3.a32*gradrhopp1.x + invacorr3.a33*gradrhopp1.y + invacorr3.a34*gradrhopp1.z);
            const float grz=    -float(invacorr3.a41*rhopp1 + invacorr3.a42*gradrhopp1.x + invacorr3.a43*gradrhopp1.y + invacorr3.a44*gradrhopp1.z);
            rhopfinal=(rhoghost + grx*dpos.x + gry*dpos.y + grz*dpos.z);
          }
          else if(a_corr3.a11>0){//-Determinant is small but a11 is nonzero, 0th order ANGELO.
            rhopfinal=float(rhopp1/a_corr3.a11);
          }
          //-Ghost node velocity (0th order).
          if(tslip!=SLIP_Vel0){
            velrhopfinal.x=float(velp1.x/a_corr3.a11);
            velrhopfinal.y=float(velp1.y/a_corr3.a11);
            velrhopfinal.z=float(velp1.z/a_corr3.a11);
          }
        }
        //-Store the results.
        rhopfinal=(rhopfinal!=FLT_MAX? rhopfinal: CTE.rhopzero);
        if(tslip==SLIP_Vel0){//-DBC vel=0
          velrhop[p1].w=rhopfinal;
        }
        if(tslip==SLIP_NoSlip){//-No-Slip
          const float3 v=motionvel[p1];
          velrhop[p1]=make_float4(v.x+v.x-velrhopfinal.x,v.y+v.y-velrhopfinal.y,v.z+v.z-velrhopfinal.z,rhopfinal);
        }
        if(tslip==SLIP_FreeSlip){//-No-Penetration and free slip    SHABA
          float3 FSVelFinal; // final free slip boundary velocity
          const float3 v=motionvel[p1];
          float motion=sqrt(v.x*v.x+v.y*v.y+v.z*v.z); // to check if boundary moving
          float norm=sqrt(bnormalp1.x*bnormalp1.x+bnormalp1.y*bnormalp1.y+bnormalp1.z*bnormalp1.z);
          float3 normal; // creating a normailsed boundary normal
          normal.x=fabs(bnormalp1.x)/norm; normal.y=fabs(bnormalp1.y)/norm; normal.z=fabs(bnormalp1.z)/norm;

          // finding the velocity componants normal and tangential to boundary 
          float3 normvel=make_float3(velrhopfinal.x*normal.x,velrhopfinal.y*normal.y,velrhopfinal.z*normal.z); // velocity in direction of normal pointin ginto fluid)
          float3 tangvel=make_float3(velrhopfinal.x-normvel.x,velrhopfinal.y-normvel.y,velrhopfinal.z-normvel.z); // velocity tangential to normal

          if (motion > 0) { // if moving boundary
              float3 normmot=make_float3(v.x*normal.x,v.y*normal.y,v.z*normal.z); // boundary motion in direction normal to boundary 
              FSVelFinal=make_float3(normmot.x+normmot.x-normvel.x,normmot.y+normmot.y-normvel.y,normmot.z+normmot.z-normvel.z);
              // only velocity in normal direction for no-penetration
              // fluid sees zero velocity in the tangetial direction
          }
          else {
              FSVelFinal=make_float3(tangvel.x-normvel.x,tangvel.y-normvel.y,tangvel.z-normvel.z);
              // tangential velocity equal to fluid velocity for free slip
              // normal velocity reversed for no-penetration
          }

          // Save the velocity and density
          velrhop[p1]=make_float4(FSVelFinal.x,FSVelFinal.y,FSVelFinal.z,rhopfinal);
        }
      }
    }
  }
}


//==============================================================================
/// Calculates extrapolated data on boundary particles from fluid domain for mDBC.
/// Calcula datos extrapolados en el contorno para mDBC.
//==============================================================================
template<TpKernel tker,bool sim2d,TpSlipMode tslip> void Interaction_MdbcCorrectionT2(
  bool fastsingle,unsigned n,unsigned nbound,float mdbcthreshold,const StDivDataGpu &dvd
  ,const tdouble3 &mapposmin,const double2 *posxy,const double *posz,const float4 *poscell
  ,const typecode *code,const unsigned *idp,const float3 *boundnormal,const float3 *motionvel
  ,float4 *velrhop)
{
  const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
  const float determlimit=1e-3f;
  //-Interaction GhostBoundaryNodes-Fluid.
  if(n){
    const unsigned bsbound=128;
    dim3 sgridb=cusph::GetSimpleGridSize(n,bsbound);
    if(fastsingle){//-mDBC-Fast_v2
      KerInteractionMdbcCorrection_Fast <tker,sim2d,tslip> <<<sgridb,bsbound>>> (n,nbound
        ,determlimit,mdbcthreshold,Double3(mapposmin),dvd.poscellsize,poscell
        ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
        ,posxy,posz,code,idp,boundnormal,motionvel,velrhop);
    }
    else{//-mDBC_v0
      KerInteractionMdbcCorrection_Dbl <tker,sim2d,tslip> <<<sgridb,bsbound>>> (n,nbound
        ,determlimit,mdbcthreshold,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
        ,posxy,posz,code,idp,boundnormal,motionvel,velrhop);
    }
  }
}
//==============================================================================
template<TpKernel tker> void Interaction_MdbcCorrectionT(bool simulate2d
  ,TpSlipMode slipmode,bool fastsingle,unsigned n,unsigned nbound
  ,float mdbcthreshold,const StDivDataGpu &dvd,const tdouble3 &mapposmin
  ,const double2 *posxy,const double *posz,const float4 *poscell,const typecode *code
  ,const unsigned *idp,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  switch(slipmode){
    case SLIP_Vel0:{ const TpSlipMode tslip=SLIP_Vel0;
      if(simulate2d)Interaction_MdbcCorrectionT2 <tker,true ,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
      else          Interaction_MdbcCorrectionT2 <tker,false,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#ifndef DISABLE_MDBC_EXTRAMODES
    case SLIP_NoSlip:{ const TpSlipMode tslip=SLIP_NoSlip;
      if(simulate2d)Interaction_MdbcCorrectionT2 <tker,true ,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
      else          Interaction_MdbcCorrectionT2 <tker,false,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
    case SLIP_FreeSlip:{ const TpSlipMode tslip=SLIP_FreeSlip;
      if(simulate2d)Interaction_MdbcCorrectionT2 <tker,true ,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
      else          Interaction_MdbcCorrectionT2 <tker,false,tslip> (fastsingle,n,nbound,mdbcthreshold,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#endif
    default: throw "SlipMode unknown at Interaction_MdbcCorrectionT().";
  }
}
//==============================================================================
/// Calculates extrapolated data on boundary particles from fluid domain for mDBC.
/// Calcula datos extrapolados en el contorno para mDBC.
//==============================================================================
void Interaction_MdbcCorrection(TpKernel tkernel,bool simulate2d,TpSlipMode slipmode
  ,bool fastsingle,unsigned n,unsigned nbound,float mdbcthreshold
  ,const StDivDataGpu &dvd,const tdouble3 &mapposmin
  ,const double2 *posxy,const double *posz,const float4 *poscell,const typecode *code
  ,const unsigned *idp,const float3 *boundnormal,const float3 *motionvel,float4 *velrhop)
{
  switch(tkernel){
    case KERNEL_Wendland:{ const TpKernel tker=KERNEL_Wendland;
      Interaction_MdbcCorrectionT <tker> (simulate2d,slipmode,fastsingle,n,nbound,mdbcthreshold
        ,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#ifndef DISABLE_KERNELS_EXTRA
    case KERNEL_Cubic:{ const TpKernel tker=KERNEL_Cubic;
      Interaction_MdbcCorrectionT <tker> (simulate2d,slipmode,fastsingle,n,nbound,mdbcthreshold
        ,dvd,mapposmin,posxy,posz,poscell,code,idp,boundnormal,motionvel,velrhop);
    }break;
#endif
    default: throw "Kernel unknown at Interaction_MdbcCorrection().";
  }
}


//##############################################################################
//# Kernels for DEM interaction.
//# Kernels para interaccion DEM.
//##############################################################################
//------------------------------------------------------------------------------
/// DEM interaction of a particle with a set of particles. (Float-Float/Bound)
/// Realiza la interaccion DEM de una particula con un conjunto de ellas. (Float-Float/Bound)
//------------------------------------------------------------------------------
__device__ void KerInteractionForcesDemBox 
  (bool boundp2,const unsigned &pini,const unsigned &pfin
  ,const float4 *demdata,float dtforce
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float4 &pscellp1,const float4 &velp1,typecode tavp1,float masstotp1
  ,float ftmassp1,float taup1,float kfricp1,float restitup1
  ,float3 &acep1,float &demdtp1)
{
  for(int p2=pini;p2<pfin;p2++){
    const typecode codep2=code[p2];
    if(CODE_IsNotFluid(codep2) && tavp1!=CODE_GetTypeAndValue(codep2)){
      const float4 pscellp2=poscell[p2];
      const float drx=pscellp1.x-pscellp2.x + CTE.poscellsize*(CEL_GetfX(pscellp1.w)-CEL_GetfX(pscellp2.w));
      const float dry=pscellp1.y-pscellp2.y + CTE.poscellsize*(CEL_GetfY(pscellp1.w)-CEL_GetfY(pscellp2.w));
      const float drz=pscellp1.z-pscellp2.z + CTE.poscellsize*(CEL_GetfZ(pscellp1.w)-CEL_GetfZ(pscellp2.w));
      const float rr2=drx*drx+dry*dry+drz*drz;
      const float rad=sqrt(rr2);

      //-Computes maximum value of demdt.
      float4 demdatap2=demdata[CODE_GetTypeAndValue(codep2)];
      const float nu_mass=(boundp2? masstotp1/2: masstotp1*demdatap2.x/(masstotp1+demdatap2.x)); //-With boundary takes the actual mass of floating 1. | Con boundary toma la propia masa del floating 1.
      const float kn=4/(3*(taup1+demdatap2.y))*sqrt(CTE.dp/4); //-Generalized rigidity - Lemieux 2008.
      const float dvx=velp1.x-velrhop[p2].x, dvy=velp1.y-velrhop[p2].y, dvz=velp1.z-velrhop[p2].z; //vji
      const float nx=drx/rad, ny=dry/rad, nz=drz/rad; //-normal_ji             
      const float vn=dvx*nx+dvy*ny+dvz*nz; //-vji.nji    
      const float demvisc=0.2f/(3.21f*(pow(nu_mass/kn,0.4f)*pow(fabs(vn),-0.2f))/40.f);
      if(demdtp1<demvisc)demdtp1=demvisc;

      const float over_lap=1.0f*CTE.dp-rad; //-(ri+rj)-|dij|
      if(over_lap>0.0f){ //-Contact.
        //-Normal.
        const float eij=(restitup1+demdatap2.w)/2;
        const float gn=-(2.0f*log(eij)*sqrt(nu_mass*kn))/(sqrt(float(PI)+log(eij)*log(eij))); //-Generalized damping - Cummins 2010.
        //const float gn=0.08f*sqrt(nu_mass*sqrt(CTE.dp/2)/((taup1+demdatap2.y)/2)); //-generalized damping - Lemieux 2008.
        const float rep=kn*pow(over_lap,1.5f);
        const float fn=rep-gn*pow(over_lap,0.25f)*vn;
        float acef=fn/ftmassp1; //-Divides by the mass of particle to obtain the acceleration.
        acep1.x+=(acef*nx); acep1.y+=(acef*ny); acep1.z+=(acef*nz); //-Force is applied in the normal between the particles.
        //-Tangencial.
        const float dvxt=dvx-vn*nx, dvyt=dvy-vn*ny, dvzt=dvz-vn*nz; //Vji_t
        const float vt=sqrt(dvxt*dvxt + dvyt*dvyt + dvzt*dvzt);
        const float tx=(vt!=0? dvxt/vt: 0), ty=(vt!=0? dvyt/vt: 0), tz=(vt!=0? dvzt/vt: 0); //-Tang vel unit vector.
        const float ft_elast=2*(kn*dtforce-gn)*vt/7; //-Elastic frictional string -->  ft_elast=2*(kn*fdispl-gn*vt)/7; fdispl=dtforce*vt;
        const float kfric_ij=(kfricp1+demdatap2.z)/2;
        float ft=kfric_ij*fn*tanh(8*vt);  //-Coulomb.
        ft=(ft<ft_elast? ft: ft_elast);   //-Not above yield criteria, visco-elastic model.
        acef=ft/ftmassp1; //-Divides by the mass of particle to obtain the acceleration.
        acep1.x+=(acef*tx); acep1.y+=(acef*ty); acep1.z+=(acef*tz);
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
__global__ void KerInteractionForcesDem(unsigned nfloat
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,const float *ftomassp,float dtforce
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float3 *ace)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<nfloat){
    const unsigned p1=ftridp[p]; //-Number of particle.
    if(p1!=UINT_MAX){
      float demdtp1=0;
      float3 acep1=make_float3(0,0,0);

      //-Obtains basic data of particle p1.
      const float4 pscellp1=poscell[p1];
      const float4 velp1=velrhop[p1];
      const typecode cod=code[p1];
      const typecode tavp1=CODE_GetTypeAndValue(cod);
      const float4 rdata=demdata[tavp1];
      const float masstotp1=rdata.x;
      const float taup1=rdata.y;
      const float kfricp1=rdata.z;
      const float restitup1=rdata.w;
      const float ftmassp1=ftomassp[CODE_GetTypeValue(cod)];

      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

      //-Interaction with boundaries.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin)KerInteractionForcesDemBox (true ,pini,pfin,demdata,dtforce,poscell,velrhop,code,idp,pscellp1,velp1,tavp1,masstotp1,ftmassp1,taup1,kfricp1,restitup1,acep1,demdtp1);
      }

      //-Interaction with fluids.
      ini3+=cellfluid; fin3+=cellfluid;
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin)KerInteractionForcesDemBox (false,pini,pfin,demdata,dtforce,poscell,velrhop,code,idp,pscellp1,velp1,tavp1,masstotp1,ftmassp1,taup1,kfricp1,restitup1,acep1,demdtp1);
      }

      //-Stores results.
      if(acep1.x || acep1.y || acep1.z || demdtp1){
        float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
        if(viscdt[p1]<demdtp1)viscdt[p1]=demdtp1;
      }
    }
  }
}

#ifndef DISABLE_BSMODES
//==============================================================================
/// Collects kernel information.
//==============================================================================
void Interaction_ForcesDemT_KerInfo(StKerInfo *kerinfo)
{
#if CUDART_VERSION >= 6050
  {
    typedef void (*fun_ptr)(unsigned,int,int4,int3,const int2*,unsigned,const unsigned*,const unsigned*,const float4*,const float*,float,const float4*,const float4*,const typecode*,const unsigned*,float*,float3*);
    fun_ptr ptr=&KerInteractionForcesDem;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesdem_bs=qblocksize;
    kerinfo->forcesdem_rg=attr.numRegs;
    kerinfo->forcesdem_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesDem  blocksize:%u (%u)\n",qblocksize,0);
  }
  fcuda::Check_CudaErroorFun("Error collecting kernel information.");
#endif
}
#endif

//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
void Interaction_ForcesDem(unsigned bsize,unsigned nfloat
  ,const StDivDataGpu &dvd,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,const float *ftomassp,float dtforce
  ,const float4 *poscell,const float4 *velrhop
  ,const typecode *code,const unsigned *idp,float *viscdt,float3 *ace,StKerInfo *kerinfo)
{
  const int2* beginendcell=dvd.beginendcell;
  //-Collects kernel information.
#ifndef DISABLE_BSMODES
  if(kerinfo){
    Interaction_ForcesDemT_KerInfo(kerinfo);
    return;
  }
#endif
  //-Interaction Fluid-Fluid & Fluid-Bound.
  if(nfloat){
    dim3 sgrid=GetSimpleGridSize(nfloat,bsize);
    KerInteractionForcesDem <<<sgrid,bsize>>> (nfloat
      ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcell,dvd.cellfluid,dcell
      ,ftridp,demdata,ftomassp,dtforce,poscell,velrhop,code,idp,viscdt,ace);
  }
}


//##############################################################################
//# Kernels for Laminar+SPS.
//##############################################################################
//------------------------------------------------------------------------------
/// Computes sub-particle stress tensor (Tau) for SPS turbulence model.
//------------------------------------------------------------------------------
__global__ void KerComputeSpsTau(unsigned n,unsigned pini,float smag,float blin
  ,const float4 *velrhop,const float2 *gradvelff,float2 *tauff)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; 
  if(p<n){
    const unsigned p1=p+pini;
    float2 rr=gradvelff[p1*3];   const float grad_xx=rr.x,grad_xy=rr.y;
           rr=gradvelff[p1*3+1]; const float grad_xz=rr.x,grad_yy=rr.y;
           rr=gradvelff[p1*3+2]; const float grad_yz=rr.x,grad_zz=rr.y;
    const float pow1=grad_xx*grad_xx + grad_yy*grad_yy + grad_zz*grad_zz;
    const float prr= grad_xy*grad_xy + grad_xz*grad_xz + grad_yz*grad_yz + pow1+pow1;
    const float visc_sps=smag*sqrt(prr);
    const float div_u=grad_xx+grad_yy+grad_zz;
    const float sps_k=(2.0f/3.0f)*visc_sps*div_u;
    const float sps_blin=blin*prr;
    const float sumsps=-(sps_k+sps_blin);
    const float twovisc_sps=(visc_sps+visc_sps);
    float one_rho2=1.0f/velrhop[p1].w;
    //-Computes new values of tau[].
    const float tau_xx=one_rho2*(twovisc_sps*grad_xx +sumsps);
    const float tau_xy=one_rho2*(visc_sps   *grad_xy);
    tauff[p1*3]=make_float2(tau_xx,tau_xy);
    const float tau_xz=one_rho2*(visc_sps   *grad_xz);
    const float tau_yy=one_rho2*(twovisc_sps*grad_yy +sumsps);
    tauff[p1*3+1]=make_float2(tau_xz,tau_yy);
    const float tau_yz=one_rho2*(visc_sps   *grad_yz);
    const float tau_zz=one_rho2*(twovisc_sps*grad_zz +sumsps);
    tauff[p1*3+2]=make_float2(tau_yz,tau_zz);
  }
}

//==============================================================================
/// Computes sub-particle stress tensor (Tau) for SPS turbulence model.
//==============================================================================
void ComputeSpsTau(unsigned np,unsigned npb,float smag,float blin
  ,const float4 *velrhop,const tsymatrix3f *gradvelg,tsymatrix3f *tau,hipStream_t stm)
{
  const unsigned npf=np-npb;
  if(npf){
    dim3 sgridf=GetSimpleGridSize(npf,SPHBSIZE);
    KerComputeSpsTau <<<sgridf,SPHBSIZE,0,stm>>> (npf,npb,smag,blin,velrhop,(const float2*)gradvelg,(float2*)tau);
  }
}


//##############################################################################
//# Kernels for Delta-SPH.
//# Kernels para Delta-SPH.
//##############################################################################
//------------------------------------------------------------------------------
/// Adds value of delta[] to ar[] provided it is not FLT_MAX.
/// Anhade valor de delta[] a ar[] siempre que no sea FLT_MAX.
//------------------------------------------------------------------------------
__global__ void KerAddDelta(unsigned n,const float *delta,float *ar)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    float rdelta=delta[p];
    if(rdelta!=FLT_MAX)ar[p]+=rdelta;
  }
}

//==============================================================================
/// Adds value of delta[] to ar[] provided it is not FLT_MAX.
/// Anhade valor de delta[] a ar[] siempre que no sea FLT_MAX.
//==============================================================================
void AddDelta(unsigned n,const float *delta,float *ar,hipStream_t stm){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerAddDelta <<<sgrid,SPHBSIZE,0,stm>>> (n,delta,ar);
  }
}


//##############################################################################
//# Kernels para ComputeStep (position)
//# Kernels for ComputeStep (position)
//##############################################################################
//------------------------------------------------------------------------------
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//------------------------------------------------------------------------------
template<bool periactive,bool floatings> __global__ void KerComputeStepPos(unsigned n,unsigned pini
  ,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pt=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pt<n){
    unsigned p=pt+pini;
    const typecode rcode=code[p];
    const bool outrhop=CODE_IsOutRhop(rcode);
    const bool fluid=(!floatings || CODE_IsFluid(rcode));
    const bool normal=(!periactive || outrhop || CODE_IsNormal(rcode));
    if(normal && fluid){ //-Does not apply to periodic or floating particles. | No se aplica a particulas periodicas o floating.
      const double2 rmovxy=movxy[p];
      KerUpdatePos<periactive>(posxy[p],posz[p],rmovxy.x,rmovxy.y,movz[p],outrhop,p,posxy,posz,dcell,code);
    }
    //-In case of floating maintains the original position.
    //-En caso de floating mantiene la posicion original.
  }
}

//==============================================================================
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//==============================================================================
void ComputeStepPos(byte periactive,bool floatings,unsigned np,unsigned npb
  ,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pini=npb;
  const unsigned npf=np-pini;
  if(npf){
    dim3 sgrid=GetSimpleGridSize(npf,SPHBSIZE);
    if(periactive){ const bool peri=true;
      if(floatings)KerComputeStepPos<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
    }
    else{ const bool peri=false;
      if(floatings)KerComputeStepPos<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
    }
  }
}

//------------------------------------------------------------------------------
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//------------------------------------------------------------------------------
template<bool periactive,bool floatings> __global__ void KerComputeStepPos2(unsigned n,unsigned pini
  ,const double2 *posxypre,const double *poszpre,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pt=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pt<n){
    unsigned p=pt+pini;
    const typecode rcode=code[p];
    const bool outrhop=CODE_IsOutRhop(rcode);
    const bool fluid=(!floatings || CODE_IsFluid(rcode));
    const bool normal=(!periactive || outrhop || CODE_IsNormal(rcode));
    if(normal){//-Does not apply to periodic particles. | No se aplica a particulas periodicas
      if(fluid){//-Only applied for fluid displacement. | Solo se aplica desplazamiento al fluido.
        const double2 rmovxy=movxy[p];
        KerUpdatePos<periactive>(posxypre[p],poszpre[p],rmovxy.x,rmovxy.y,movz[p],outrhop,p,posxy,posz,dcell,code);
      }
      else{
        posxy[p]=posxypre[p];
        posz[p]=poszpre[p];
      }
    }
  }
}

//==============================================================================
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//==============================================================================
void ComputeStepPos2(byte periactive,bool floatings,unsigned np,unsigned npb
  ,const double2 *posxypre,const double *poszpre,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pini=npb;
  const unsigned npf=np-pini;
  if(npf){
    dim3 sgrid=GetSimpleGridSize(npf,SPHBSIZE);
    if(periactive){ const bool peri=true;
      if(floatings)KerComputeStepPos2<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos2<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
    }
    else{ const bool peri=false;
      if(floatings)KerComputeStepPos2<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
      else         KerComputeStepPos2<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
    }
  }
}



//##############################################################################
//# Kernels for motion.
//# Kernels para Motion
//##############################################################################
//------------------------------------------------------------------------------
/// Computes for a range of particles, their position according to idp[].
/// Calcula para un rango de particulas calcula su posicion segun idp[].
//------------------------------------------------------------------------------
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const typecode *code,const unsigned *idp,unsigned *ridp)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    p+=ini;
    unsigned id=idp[p];
    if(idini<=id && id<idfin){
      if(CODE_IsNormal(code[p]))ridp[id-idini]=p;
    }
  }
}
//------------------------------------------------------------------------------
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const unsigned *idp,unsigned *ridp)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    p+=ini;
    const unsigned id=idp[p];
    if(idini<=id && id<idfin)ridp[id-idini]=p;
  }
}

//==============================================================================
/// Calculate particle position according to idp[]. When it does not find UINT_MAX.
/// When periactive is false it means there are no duplicate particles (periodic)
/// and all are CODE_NORMAL.
///
/// Calcula posicion de particulas segun idp[]. Cuando no la encuentra es UINT_MAX.
/// Cuando periactive es False sumpone que no hay particulas duplicadas (periodicas)
/// y todas son CODE_NORMAL.
//==============================================================================
void CalcRidp(bool periactive,unsigned np,unsigned pini,unsigned idini,unsigned idfin,const typecode *code,const unsigned *idp,unsigned *ridp){
  //-Assigns values UINT_MAX
  const unsigned nsel=idfin-idini;
  hipMemset(ridp,255,sizeof(unsigned)*nsel); 
  //-Computes position according to id. | Calcula posicion segun id.
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(periactive)KerCalcRidp <<<sgrid,SPHBSIZE>>> (np,pini,idini,idfin,code,idp,ridp);
    else          KerCalcRidp <<<sgrid,SPHBSIZE>>> (np,pini,idini,idfin,idp,ridp);
  }
}

//------------------------------------------------------------------------------
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento lineal a un conjunto de particulas.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerMoveLinBound(unsigned n,unsigned ini,double3 mvpos,float3 mvvel
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p+ini];
    if(pid>=0){
      //-Computes displacement and updates position.
      KerUpdatePos<periactive>(posxy[pid],posz[pid],mvpos.x,mvpos.y,mvpos.z,false,pid,posxy,posz,dcell,code);
      //-Computes velocity.
      velrhop[pid]=make_float4(mvvel.x,mvvel.y,mvvel.z,velrhop[pid].w);
    }
  }
}

//==============================================================================
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento lineal a un conjunto de particulas.
//==============================================================================
void MoveLinBound(byte periactive,unsigned np,unsigned ini,tdouble3 mvpos,tfloat3 mvvel
  ,const unsigned *ridp,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
  if(periactive)KerMoveLinBound<true>  <<<sgrid,SPHBSIZE>>> (np,ini,Double3(mvpos),Float3(mvvel),ridp,posxy,posz,dcell,velrhop,code);
  else          KerMoveLinBound<false> <<<sgrid,SPHBSIZE>>> (np,ini,Double3(mvpos),Float3(mvvel),ridp,posxy,posz,dcell,velrhop,code);
}



//------------------------------------------------------------------------------
/// Applies a matrix movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//------------------------------------------------------------------------------
template<bool periactive,bool simulate2d> __global__ void KerMoveMatBound(unsigned n,unsigned ini,tmatrix4d m,double dt
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code,float3 *boundnormal)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p+ini];
    if(pid>=0){
      double2 rxy=posxy[pid];
      double3 rpos=make_double3(rxy.x,rxy.y,posz[pid]);
      //-Computes new position.
      double3 rpos2;
      rpos2.x= rpos.x*m.a11 + rpos.y*m.a12 + rpos.z*m.a13 + m.a14;
      rpos2.y= rpos.x*m.a21 + rpos.y*m.a22 + rpos.z*m.a23 + m.a24;
      rpos2.z= rpos.x*m.a31 + rpos.y*m.a32 + rpos.z*m.a33 + m.a34;
      if(simulate2d)rpos2.y=rpos.y;
      //-Computes displacement and updates position.
      const double dx=rpos2.x-rpos.x;
      const double dy=rpos2.y-rpos.y;
      const double dz=rpos2.z-rpos.z;
      KerUpdatePos<periactive>(make_double2(rpos.x,rpos.y),rpos.z,dx,dy,dz,false,pid,posxy,posz,dcell,code);
      //-Computes velocity.
      velrhop[pid]=make_float4(float(dx/dt),float(dy/dt),float(dz/dt),velrhop[pid].w);
      //-Computes normal.
      if(boundnormal){
        const float3 bnor=boundnormal[pid];
        const double3 gs=make_double3(rpos.x+bnor.x,rpos.y+bnor.y,rpos.z+bnor.z);
        const double gs2x=gs.x*m.a11 + gs.y*m.a12 + gs.z*m.a13 + m.a14;
        const double gs2y=gs.x*m.a21 + gs.y*m.a22 + gs.z*m.a23 + m.a24;
        const double gs2z=gs.x*m.a31 + gs.y*m.a32 + gs.z*m.a33 + m.a34;
        boundnormal[pid]=make_float3(gs2x-rpos2.x,gs2y-rpos2.y,gs2z-rpos2.z);
      }
    }
  }
}

//==============================================================================
/// Applies a matrix movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//==============================================================================
void MoveMatBound(byte periactive,bool simulate2d,unsigned np,unsigned ini,tmatrix4d m,double dt
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code,float3 *boundnormal)
{
  dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
  if(periactive){ const bool peri=true;
    if(simulate2d)KerMoveMatBound<peri,true>  <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal);
    else          KerMoveMatBound<peri,false> <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal);
  }
  else{ const bool peri=false;
    if(simulate2d)KerMoveMatBound<peri,true>  <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal);
    else          KerMoveMatBound<peri,false> <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code,boundnormal);
  }
}

//------------------------------------------------------------------------------
/// Copy motion velocity to MotionVel[].
/// Copia velocidad de movimiento a MotionVel[].
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerCopyMotionVel(unsigned n
  ,const unsigned *ridpmv,const float4 *velrhop,float3 *motionvel)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p];
    if(pid>=0){
      //-Computes velocity.
      const float4 v=velrhop[pid];
      motionvel[pid]=make_float3(v.x,v.y,v.z);
    }
  }
}

//==============================================================================
/// Copy motion velocity to MotionVel[].
/// Copia velocidad de movimiento a MotionVel[].
//==============================================================================
void CopyMotionVel(unsigned nmoving,const unsigned *ridp,const float4 *velrhop,float3 *motionvel)
{
  dim3 sgrid=GetSimpleGridSize(nmoving,SPHBSIZE);
  KerCopyMotionVel<true>  <<<sgrid,SPHBSIZE>>> (nmoving,ridp,velrhop,motionvel);
}



//##############################################################################
//# Kernels for MLPistons motion.
//##############################################################################
//------------------------------------------------------------------------------
/// Applies movement and velocity of piston 1D to a group of particles.
/// Aplica movimiento y velocidad de piston 1D a conjunto de particulas.
//------------------------------------------------------------------------------
template<byte periactive> __global__ void KerMovePiston1d(unsigned n,unsigned idini
  ,double dp,double poszmin,unsigned poszcount,const byte *pistonid,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(p<n){
    const unsigned id=p+idini;
    int pid=ridpmv[id];
    if(pid>=0){
      const unsigned pisid=pistonid[CODE_GetTypeValue(code[pid])];
      if(pisid<255){
        const double2 rpxy=posxy[pid];
        const double rpz=posz[pid];
        const unsigned cz=unsigned((rpz-poszmin)/dp);
        const double rmovx=(cz<poszcount? movx[pisid*poszcount+cz]: 0);
        const float rvelx=float(cz<poszcount? velx[pisid*poszcount+cz]: 0);
        //-Updates position.
        KerUpdatePos<periactive>(rpxy,rpz,rmovx,0,0,false,pid,posxy,posz,dcell,code);
        //-Updates velocity.
        velrhop[pid].x=rvelx;
      }
    }
  }
}

//==============================================================================
/// Applies movement and velocity of piston 1D to a group of particles.
/// Aplica movimiento y velocidad de piston 1D a conjunto de particulas.
//==============================================================================
void MovePiston1d(bool periactive,unsigned np,unsigned idini
  ,double dp,double poszmin,unsigned poszcount,const byte *pistonid,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(periactive)KerMovePiston1d<true>  <<<sgrid,SPHBSIZE>>> (np,idini,dp,poszmin,poszcount,pistonid,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
    else          KerMovePiston1d<false> <<<sgrid,SPHBSIZE>>> (np,idini,dp,poszmin,poszcount,pistonid,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
  }
}

//------------------------------------------------------------------------------
/// Applies movement and velocity of piston 2D to a group of particles.
/// Aplica movimiento y velocidad de piston 2D a conjunto de particulas.
//------------------------------------------------------------------------------
template<byte periactive> __global__ void KerMovePiston2d(unsigned n,unsigned idini
  ,double dp,double posymin,double poszmin,unsigned poszcount,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(p<n){
    const unsigned id=p+idini;
    int pid=ridpmv[id];
    if(pid>=0){
      const double2 rpxy=posxy[pid];
      const double rpz=posz[pid];
      const unsigned cy=unsigned((rpxy.y-posymin)/dp);
      const unsigned cz=unsigned((rpz-poszmin)/dp);
      const double rmovx=(cz<poszcount? movx[cy*poszcount+cz]: 0);
      const float rvelx=float(cz<poszcount? velx[cy*poszcount+cz]: 0);
      //-Actualiza posicion.
      KerUpdatePos<periactive>(rpxy,rpz,rmovx,0,0,false,pid,posxy,posz,dcell,code);
      //-Actualiza velocidad.
      velrhop[pid].x=rvelx;
    }
  }
}

//==============================================================================
/// Applies movement and velocity of piston 2D to a group of particles.
/// Aplica movimiento y velocidad de piston 2D a conjunto de particulas.
//==============================================================================
void MovePiston2d(bool periactive,unsigned np,unsigned idini
  ,double dp,double posymin,double poszmin,unsigned poszcount,const double* movx,const double* velx
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(periactive)KerMovePiston2d<true>  <<<sgrid,SPHBSIZE>>> (np,idini,dp,posymin,poszmin,poszcount,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
    else          KerMovePiston2d<false> <<<sgrid,SPHBSIZE>>> (np,idini,dp,posymin,poszmin,poszcount,movx,velx,ridpmv,posxy,posz,dcell,velrhop,code);
  }
}


//##############################################################################
//# Kernels for Floating bodies.
//##############################################################################
//==============================================================================
/// Computes distance between floating and centre particles according to periodic conditions.
/// Calcula distancia entre pariculas floating y centro segun condiciones periodicas.
//==============================================================================
template<bool periactive> __device__ void KerFtPeriodicDist(double px,double py,double pz,double cenx,double ceny,double cenz,float radius,float &dx,float &dy,float &dz){
  if(periactive){
    double ddx=px-cenx;
    double ddy=py-ceny;
    double ddz=pz-cenz;
    const unsigned peri=CTE.periactive;
    if(PERI_AxisX(peri) && fabs(ddx)>radius){
      if(ddx>0){ ddx+=CTE.xperincx; ddy+=CTE.xperincy; ddz+=CTE.xperincz; }
      else{      ddx-=CTE.xperincx; ddy-=CTE.xperincy; ddz-=CTE.xperincz; }
    }
    if(PERI_AxisY(peri) && fabs(ddy)>radius){
      if(ddy>0){ ddx+=CTE.yperincx; ddy+=CTE.yperincy; ddz+=CTE.yperincz; }
      else{      ddx-=CTE.yperincx; ddy-=CTE.yperincy; ddz-=CTE.yperincz; }
    }
    if(PERI_AxisZ(peri) && fabs(ddz)>radius){
      if(ddz>0){ ddx+=CTE.zperincx; ddy+=CTE.zperincy; ddz+=CTE.zperincz; }
      else{      ddx-=CTE.zperincx; ddy-=CTE.zperincy; ddz-=CTE.zperincz; }
    }
    dx=float(ddx);
    dy=float(ddy);
    dz=float(ddz);
  }
  else{
    dx=float(px-cenx);
    dy=float(py-ceny);
    dz=float(pz-cenz);
  }
}

//------------------------------------------------------------------------------
/// Calculate summation: face, fomegaace in ftoforcessum[].
/// Calcula suma de face y fomegaace a partir de particulas floating en ftoforcessum[].
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerFtCalcForcesSum( //ftodatp={pini,np,radius,massp}
  const float4 *ftodatp,const double3 *ftocenter,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,const float3 *ace
  ,float3 *ftoforcessum)
{
  extern __shared__ float rfacex[];
  float *rfacey=rfacex+blockDim.x;
  float *rfacez=rfacey+blockDim.x;
  float *rfomegaacex=rfacez+blockDim.x;
  float *rfomegaacey=rfomegaacex+blockDim.x;
  float *rfomegaacez=rfomegaacey+blockDim.x;

  const unsigned tid=threadIdx.x;  //-Thread number.
  const unsigned cf=blockIdx.x;    //-Floating number.
  
  //-Loads floating data.
  const float4 rfdata=ftodatp[cf];
  const unsigned fpini=(unsigned)__float_as_int(rfdata.x);
  const unsigned fnp=(unsigned)__float_as_int(rfdata.y);
  const float fradius=rfdata.z;
  const float fmassp=rfdata.w;
  const double3 rcenter=ftocenter[cf];

  //-Initialises shared memory to zero.
  const unsigned ntid=(fnp<blockDim.x? fnp: blockDim.x); //-Number of used threads. | Numero de threads utilizados.
  if(tid<ntid){
    rfacex[tid]=rfacey[tid]=rfacez[tid]=0;
    rfomegaacex[tid]=rfomegaacey[tid]=rfomegaacez[tid]=0;
  }

  //-Computes data in shared memory. | Calcula datos en memoria shared.
  const unsigned nfor=unsigned((fnp+blockDim.x-1)/blockDim.x);
  for(unsigned cfor=0;cfor<nfor;cfor++){
    unsigned p=cfor*blockDim.x+tid;
    if(p<fnp){
      const unsigned rp=ftridp[p+fpini];
      if(rp!=UINT_MAX){
        float3 force=ace[rp];
        force.x*=fmassp; force.y*=fmassp; force.z*=fmassp;
        rfacex[tid]+=force.x; rfacey[tid]+=force.y; rfacez[tid]+=force.z;
        //-Computes distance from the centre. | Calcula distancia al centro.
        double2 rposxy=posxy[rp];
        float dx,dy,dz;
        KerFtPeriodicDist<periactive>(rposxy.x,rposxy.y,posz[rp],rcenter.x,rcenter.y,rcenter.z,fradius,dx,dy,dz);
        //-Computes omegaace.
        rfomegaacex[tid]+=(force.z*dy - force.y*dz);
        rfomegaacey[tid]+=(force.x*dz - force.z*dx);
        rfomegaacez[tid]+=(force.y*dx - force.x*dy);
      }
    }
  }

  //-Reduces data in shared memory and stores results.
  //-Reduce datos de memoria shared y guarda resultados.
  __syncthreads();
  if(!tid){
    float3 face=make_float3(0,0,0);
    float3 fomegaace=make_float3(0,0,0);
    for(unsigned c=0;c<ntid;c++){
      face.x+=rfacex[c];  face.y+=rfacey[c];  face.z+=rfacez[c];
      fomegaace.x+=rfomegaacex[c]; fomegaace.y+=rfomegaacey[c]; fomegaace.z+=rfomegaacez[c];
    }
    //-Stores results in ftoforcessum[].
    ftoforcessum[cf*2]=face;
    ftoforcessum[cf*2+1]=fomegaace;
  }
}

//==============================================================================
/// Calculate summation: face, fomegaace in ftoforcessum[].
/// Calcula suma de face y fomegaace a partir de particulas floating en ftoforcessum[].
//==============================================================================
void FtCalcForcesSum(bool periactive,unsigned ftcount
  ,const float4 *ftodatp,const double3 *ftocenter,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,const float3 *ace
  ,float3 *ftoforcessum)
{
  if(ftcount){
    const unsigned bsize=256;
    const unsigned smem=sizeof(float)*(3+3)*bsize;
    dim3 sgrid=GetSimpleGridSize(ftcount*bsize,bsize);
    if(periactive)KerFtCalcForcesSum<true>  <<<sgrid,bsize,smem>>> (ftodatp,ftocenter,ftridp,posxy,posz,ace,ftoforcessum);
    else          KerFtCalcForcesSum<false> <<<sgrid,bsize,smem>>> (ftodatp,ftocenter,ftridp,posxy,posz,ace,ftoforcessum);
  }
}

//------------------------------------------------------------------------------
/// Carga valores de matriz 3x3 en bloques de 4, 4 y 1.
/// Loads values of matrix 3x3 in blocks of 4, 4 y 1.
//------------------------------------------------------------------------------
__device__ void KerLoadMatrix3f(unsigned c,const float4 *data8,const float *data1,tmatrix3f &v)
{
  float4 v4=data8[c*2];
  v.a11=v4.x; v.a12=v4.y; v.a13=v4.z; v.a21=v4.w;
  v4=data8[c*2+1];
  v.a22=v4.x; v.a23=v4.y; v.a31=v4.z; v.a32=v4.w;
  v.a33=data1[c];
}

//------------------------------------------------------------------------------
/// Adds acceleration from particles and from external forces to ftoforces[].
/// Anhade aceleracion de particulas y de fuerzas externas en ftoforces[].
//------------------------------------------------------------------------------
__global__ void KerFtCalcForces(unsigned ftcount,float3 gravity
  ,const float *ftomass,const float3 *ftoangles
  ,const float4 *ftoinertiaini8,const float *ftoinertiaini1
  ,const float3 *ftoforcessum,float3 *ftoforces,const float3 *ftoextforces) //fdata={pini,np,radius,mass}
{
  const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Number of floating.
  if(cf<ftcount){
    //-Loads floating data.
    const float fmass=ftomass[cf];
    const float3 fang=ftoangles[cf];
    tmatrix3f inert;
    KerLoadMatrix3f(cf,ftoinertiaini8,ftoinertiaini1,inert);

    //-Compute a cumulative rotation matrix.
    const tmatrix3f frot=cumath::RotMatrix3x3(fang);
    //-Compute the intertia tensor by rotating the initial tensor to the curent orientation I=(R*I_0)*R^T.
    inert=cumath::MulMatrix3x3(cumath::MulMatrix3x3(frot,inert),cumath::TrasMatrix3x3(frot));
    //-Calculates the inverse of the intertia matrix to compute the I^-1 * L= W
    const tmatrix3f invinert=cumath::InverseMatrix3x3(inert);

    //-Loads traslational and rotational velocities.
    const unsigned cf2=cf*2;
    float3 face=ftoforcessum[cf2];
    float3 fomegaace=ftoforcessum[cf2+1];

    //-Sums the external forces.
    if(ftoextforces!=NULL){
      //-Linear force.
      const float3 rflin=ftoextforces[cf2];
      face.x=face.x+rflin.x; 
      face.y=face.y+rflin.y; 
      face.z=face.z+rflin.z; 
      //-Angular force.
      const float3 rfang=ftoextforces[cf2+1];
      fomegaace.x=fomegaace.x+rfang.x; 
      fomegaace.y=fomegaace.y+rfang.y; 
      fomegaace.z=fomegaace.z+rfang.z; 
    }

    //-Calculate omega starting from fomegaace & invinert. | Calcula omega a partir de fomegaace y invinert.
    {
      float3 omegaace;
      omegaace.x=(fomegaace.x*invinert.a11+fomegaace.y*invinert.a12+fomegaace.z*invinert.a13);
      omegaace.y=(fomegaace.x*invinert.a21+fomegaace.y*invinert.a22+fomegaace.z*invinert.a23);
      omegaace.z=(fomegaace.x*invinert.a31+fomegaace.y*invinert.a32+fomegaace.z*invinert.a33);
      fomegaace=omegaace;
    }
    //-Add gravity force and divide by mass. | Suma fuerza de gravedad y divide por la masa.
    face.x=(face.x + fmass*gravity.x) / fmass;
    face.y=(face.y + fmass*gravity.y) / fmass;
    face.z=(face.z + fmass*gravity.z) / fmass;
    //-Stores results in ftoforces[].
    const float3 rface=ftoforces[cf2];
    const float3 rfome=ftoforces[cf2+1];
    face.x+=rface.x;      face.y+=rface.y;      face.z+=rface.z;
    fomegaace.x+=rfome.x; fomegaace.y+=rfome.y; fomegaace.z+=rfome.z;
    //-Stores final results.
    ftoforces[cf2]  =face;
    ftoforces[cf2+1]=fomegaace;
  }
}

//==============================================================================
/// Adds acceleration from particles and from external forces to ftoforces[].
/// Anhade aceleracion de particulas y de fuerzas externas en ftoforces[].
//==============================================================================
void FtCalcForces(unsigned ftcount,tfloat3 gravity
  ,const float *ftomass,const float3 *ftoangles
  ,const float4 *ftoinertiaini8,const float *ftoinertiaini1
  ,const float3 *ftoforcessum,float3 *ftoforces,const float3 *ftoextforces)
{
  if(ftcount){
    dim3 sgrid=GetSimpleGridSize(ftcount,SPHBSIZE);
    KerFtCalcForces <<<sgrid,SPHBSIZE>>> (ftcount,Float3(gravity),ftomass
      ,ftoangles,ftoinertiaini8,ftoinertiaini1,ftoforcessum,ftoforces,ftoextforces);
  }
}


//------------------------------------------------------------------------------
/// Calculate data to update floatings.
/// Calcula datos para actualizar floatings.
//------------------------------------------------------------------------------
__global__ void KerFtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt
  ,const float3 *ftovelace,const double3 *ftocenter,const float3 *ftoforces
  ,float3 *ftoforcesres,double3 *ftocenterres)
{
  const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Floating number.
  if(cf<ftcount){
    //-Compute fomega.
    float3 fomega=ftovelace[ftcount+cf];
    {
      const float3 omegaace=ftoforces[cf*2+1];
      fomega.x=float(dt*omegaace.x+fomega.x);
      fomega.y=float(dt*omegaace.y+fomega.y);
      fomega.z=float(dt*omegaace.z+fomega.z);
    }
    float3 fvel=ftovelace[cf];
    //-Zero components for 2-D simulation. | Anula componentes para 2D.
    float3 face=ftoforces[cf*2];
    if(simulate2d){ face.y=0; fomega.x=0; fomega.z=0; fvel.y=0; }
    //-Compute fcenter.
    double3 fcenter=ftocenter[cf];
    fcenter.x+=dt*fvel.x;
    fcenter.y+=dt*fvel.y;
    fcenter.z+=dt*fvel.z;
    //-Compute fvel.
    fvel.x=float(dt*face.x+fvel.x);
    fvel.y=float(dt*face.y+fvel.y);
    fvel.z=float(dt*face.z+fvel.z);
    //-Store data to update floating. | Guarda datos para actualizar floatings.
    ftoforcesres[cf*2]=fomega;
    ftoforcesres[cf*2+1]=fvel;
    ftocenterres[cf]=fcenter;
  }
}

//==============================================================================
/// Computes forces on floatings.
/// Calcula fuerzas sobre floatings.
//==============================================================================
void FtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt
  ,const float3 *ftovelace,const double3 *ftocenter,const float3 *ftoforces
  ,float3 *ftoforcesres,double3 *ftocenterres)
{
  if(ftcount){
    dim3 sgrid=GetSimpleGridSize(ftcount,SPHBSIZE);
    KerFtCalcForcesRes <<<sgrid,SPHBSIZE>>> (ftcount,simulate2d,dt,ftovelace,ftocenter,ftoforces,ftoforcesres,ftocenterres);
  }
}


//------------------------------------------------------------------------------
/// Applies motion constraints.
/// Aplica restricciones de movimiento.
//------------------------------------------------------------------------------
__global__ void KerFtApplyConstraints(unsigned ftcount,const byte *ftoconstraints
  ,float3 *ftoforces,float3 *ftoforcesres)
{
  const unsigned cf=blockIdx.x*blockDim.x + threadIdx.x; //-Floating number.
  if(cf<ftcount){
    //-Applies motion constraints.
    const byte constr=ftoconstraints[cf];
    if(constr!=0){
      const unsigned cf2=cf*2;
      const unsigned cf21=cf2+1;
      float3 face=ftoforces[cf2];
      float3 fomegaace=ftoforces[cf21];
      float3 fomega=ftoforcesres[cf2];
      float3 fvel=ftoforcesres[cf21];
      //-Updates values.
      face.x=(constr&FTCON_MoveX? 0: face.x);
      face.y=(constr&FTCON_MoveY? 0: face.y);
      face.z=(constr&FTCON_MoveZ? 0: face.z);
      fomegaace.x=(constr&FTCON_RotateX? 0: fomegaace.x);
      fomegaace.y=(constr&FTCON_RotateY? 0: fomegaace.y);
      fomegaace.z=(constr&FTCON_RotateZ? 0: fomegaace.z);
      fvel.x=(constr&FTCON_MoveX? 0: fvel.x);
      fvel.y=(constr&FTCON_MoveY? 0: fvel.y);
      fvel.z=(constr&FTCON_MoveZ? 0: fvel.z);
      fomega.x=(constr&FTCON_RotateX? 0: fomega.x);
      fomega.y=(constr&FTCON_RotateY? 0: fomega.y);
      fomega.z=(constr&FTCON_RotateZ? 0: fomega.z);
      //-Stores updated values.
      ftoforces[cf2]=face;
      ftoforces[cf21]=fomegaace;
      ftoforcesres[cf2]=fomega;
      ftoforcesres[cf21]=fvel;
    }
  }
}

//==============================================================================
/// Applies motion constraints.
/// Aplica restricciones de movimiento.
//==============================================================================
void FtApplyConstraints(unsigned ftcount,const byte *ftoconstraints
  ,float3 *ftoforces,float3 *ftoforcesres)
{
  if(ftcount){
    dim3 sgrid=GetSimpleGridSize(ftcount,SPHBSIZE);
    KerFtApplyConstraints <<<sgrid,SPHBSIZE>>> (ftcount,ftoconstraints,ftoforces,ftoforcesres);
  }
}


//------------------------------------------------------------------------------
/// Updates information and particles of floating bodies.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerFtUpdate(bool predictor,double dt //ftodata={pini,np,radius,massp}
  ,unsigned nft,const float4 *ftodatp,const float3 *ftoforcesres
  ,double3 *ftocenterres,const unsigned *ftridp
  ,double3 *ftocenter,float3 *ftoangles,float3 *ftovelace
  ,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  const unsigned tid=threadIdx.x;  //-Thread number.
  const unsigned cf=blockIdx.x;    //-Floating number.
  //-Obtains floating data.
  const float3 fomega=ftoforcesres[cf*2];
  const float3 fvel=ftoforcesres[cf*2+1];
  const double3 fcenter=ftocenterres[cf];
  float4 rfdata=ftodatp[cf];
  const unsigned fpini=(unsigned)__float_as_int(rfdata.x);
  const unsigned fnp=(unsigned)__float_as_int(rfdata.y);
  const float fradius=rfdata.z;
  //-Updates floating particles.
  const unsigned nfor=unsigned((fnp+blockDim.x-1)/blockDim.x);
  for(unsigned cfor=0;cfor<nfor;cfor++){
    unsigned fp=cfor*blockDim.x+tid;
    if(fp<fnp){
      const unsigned p=ftridp[fp+fpini];
      if(p!=UINT_MAX){
        double2 rposxy=posxy[p];
        double rposz=posz[p];
        float4 rvel=velrhop[p];
        //-Computes and stores position displacement.
        const double dx=dt*double(rvel.x);
        const double dy=dt*double(rvel.y);
        const double dz=dt*double(rvel.z);
        KerUpdatePos<periactive>(rposxy,rposz,dx,dy,dz,false,p,posxy,posz,dcell,code);
        //-Computes and stores new velocity.
        float disx,disy,disz;
        KerFtPeriodicDist<periactive>(rposxy.x+dx,rposxy.y+dy,rposz+dz,fcenter.x,fcenter.y,fcenter.z,fradius,disx,disy,disz);
        rvel.x=fvel.x+(fomega.y*disz-fomega.z*disy);
        rvel.y=fvel.y+(fomega.z*disx-fomega.x*disz);
        rvel.z=fvel.z+(fomega.x*disy-fomega.y*disx);
        velrhop[p]=rvel;
      }
    }
  }

  //-Stores floating data.
  __syncthreads();
  if(!tid && !predictor){
    ftocenter[cf]=(periactive? KerUpdatePeriodicPos(fcenter): fcenter);
    float3 rangles=ftoangles[cf];
    rangles.x=float(double(rangles.x)+double(fomega.x)*dt);
    rangles.y=float(double(rangles.y)+double(fomega.y)*dt);
    rangles.z=float(double(rangles.z)+double(fomega.z)*dt);
    ftoangles[cf]=rangles;
    //-Linear velocity and acceleration.
    float3 v=ftovelace[cf];
    v.x=(fvel.x-v.x)/float(dt);
    v.y=(fvel.y-v.y)/float(dt);
    v.z=(fvel.z-v.z)/float(dt);
    ftovelace[cf]=fvel;
    ftovelace[nft+nft+cf]=v;
    //-Angular velocity and acceleration.
    v=ftovelace[nft+cf];
    v.x=(fomega.x-v.x)/float(dt);
    v.y=(fomega.y-v.y)/float(dt);
    v.z=(fomega.z-v.z)/float(dt);
    ftovelace[nft+cf]=fomega;
    ftovelace[nft*3+cf]=v;
  }
}

//==============================================================================
/// Updates information and particles of floating bodies.
//==============================================================================
void FtUpdate(bool periactive,bool predictor,unsigned ftcount,double dt
  ,const float4 *ftodatp,const float3 *ftoforcesres,double3 *ftocenterres,const unsigned *ftridp
  ,double3 *ftocenter,float3 *ftoangles,float3 *ftovelace
  ,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(ftcount){
    const unsigned bsize=128; 
    dim3 sgrid=GetSimpleGridSize(ftcount*bsize,bsize);
    if(periactive)KerFtUpdate<true>  <<<sgrid,bsize>>> (predictor,dt,ftcount,ftodatp,ftoforcesres,ftocenterres,ftridp,ftocenter,ftoangles,ftovelace,posxy,posz,dcell,velrhop,code);
    else          KerFtUpdate<false> <<<sgrid,bsize>>> (predictor,dt,ftcount,ftodatp,ftoforcesres,ftocenterres,ftridp,ftocenter,ftoangles,ftovelace,posxy,posz,dcell,velrhop,code);
  }
}


//<vs_ftmottionsv_ini>
//------------------------------------------------------------------------------
/// Get reference position of floating bodies.
//------------------------------------------------------------------------------
__global__ void KerFtGetPosRef(unsigned np,const unsigned *idpref
  ,const unsigned *ftridp,const double2 *posxy,const double *posz,double *posref)
{
  unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(cp<np){
    bool ok=false;
    const unsigned cid=idpref[cp];
    if(cid!=UINT_MAX){
      const unsigned p=ftridp[cid];
      if(p!=UINT_MAX){
        const double2 rxy=posxy[p];
        const unsigned c=cp*3;
        posref[c  ]=rxy.x;
        posref[c+1]=rxy.y;
        posref[c+2]=posz[p];
        ok=true;
      }
    }
    if(!ok)posref[cp*3]=DBL_MAX;
  }
}
//==============================================================================
/// Get reference position of floating bodies.
//==============================================================================
void FtGetPosRef(unsigned np,const unsigned *idpref,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,double *posref)
{
  if(np){
    const unsigned bsize=128; 
    dim3 sgrid=GetSimpleGridSize(np,bsize);
    KerFtGetPosRef <<<sgrid,bsize>>> (np,idpref,ftridp,posxy,posz,posref);
  }
}
//<vs_ftmottionsv_end>



//##############################################################################
//# Kernels for Periodic conditions
//# Kernels para Periodic conditions
//##############################################################################
//------------------------------------------------------------------------------
/// Marks current periodics to be ignored.
/// Marca las periodicas actuales como ignorar.
//------------------------------------------------------------------------------
__global__ void KerPeriodicIgnore(unsigned n,typecode *code)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    //-Checks code of particles.
    //-Comprueba codigo de particula.
    const typecode rcode=code[p];
    if(CODE_IsPeriodic(rcode))code[p]=CODE_SetOutIgnore(rcode);
  }
}

//==============================================================================
/// Marks current periodics to be ignored.
/// Marca las periodicas actuales como ignorar.
//==============================================================================
void PeriodicIgnore(unsigned n,typecode *code){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicIgnore <<<sgrid,SPHBSIZE>>> (n,code);
  }
}

//------------------------------------------------------------------------------
/// Create list of new periodic particles to be duplicated and 
/// marks old periodics to be ignored.
///
/// Crea lista de nuevas particulas periodicas a duplicar y con delper activado
/// marca las periodicas viejas para ignorar.
//------------------------------------------------------------------------------
__global__ void KerPeriodicMakeList(unsigned n,unsigned pini,unsigned nmax
  ,double3 mapposmin,double3 mapposmax,double3 perinc
  ,const double2 *posxy,const double *posz,const typecode *code,unsigned *listp)
{
  extern __shared__ unsigned slist[];
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p2=p+pini;
    //-Inteacts with normal or periodic particles.
    //-Se queda con particulas normales o periodicas.
    if(CODE_GetSpecialValue(code[p2])<=CODE_PERIODIC){
      //-Obtains particle position.
      const double2 rxy=posxy[p2];
      const double rx=rxy.x,ry=rxy.y;
      const double rz=posz[p2];
      double rx2=rx+perinc.x,ry2=ry+perinc.y,rz2=rz+perinc.z;
      if(mapposmin.x<=rx2 && mapposmin.y<=ry2 && mapposmin.z<=rz2 && rx2<mapposmax.x && ry2<mapposmax.y && rz2<mapposmax.z){
        unsigned cp=atomicAdd(slist,1);  slist[cp+1]=p2;
      }
      rx2=rx-perinc.x; ry2=ry-perinc.y; rz2=rz-perinc.z;
      if(mapposmin.x<=rx2 && mapposmin.y<=ry2 && mapposmin.z<=rz2 && rx2<mapposmax.x && ry2<mapposmax.y && rz2<mapposmax.z){
        unsigned cp=atomicAdd(slist,1);  slist[cp+1]=(p2|0x80000000);
      }
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((listp+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    unsigned cp=slist[0]+threadIdx.x;
    if(cp<nmax)listp[cp]=slist[threadIdx.x+1];
  }
  if(blockDim.x+threadIdx.x<ns){ //-There may be twice as many periodics per thread. | Puede haber el doble de periodicas que threads.
    unsigned cp=blockDim.x+slist[0]+threadIdx.x;
    if(cp<nmax)listp[cp]=slist[blockDim.x+threadIdx.x+1];
  }
}

//==============================================================================
/// Create list of new periodic particles to be duplicated.
/// With stable activated reorders perioc list.
///
/// Crea lista de nuevas particulas periodicas a duplicar.
/// Con stable activado reordena lista de periodicas.
//==============================================================================
unsigned PeriodicMakeList(unsigned n,unsigned pini,bool stable,unsigned nmax
  ,tdouble3 mapposmin,tdouble3 mapposmax,tdouble3 perinc
  ,const double2 *posxy,const double *posz,const typecode *code,unsigned *listp)
{
  unsigned count=0;
  if(n){
    //-lspg size list initialized to zero.
    //-Inicializa tamanho de lista lspg a cero.
    hipMemset(listp+nmax,0,sizeof(unsigned));
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE*2+1)*sizeof(unsigned); //-Each particle can leave two new periodic over the counter position. | De cada particula pueden salir 2 nuevas periodicas mas la posicion del contador.
    KerPeriodicMakeList <<<sgrid,SPHBSIZE,smem>>> (n,pini,nmax,Double3(mapposmin),Double3(mapposmax),Double3(perinc),posxy,posz,code,listp);
    hipMemcpy(&count,listp+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list(listp);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}

//------------------------------------------------------------------------------
/// Doubles the position of the indicated particle using a displacement.
/// Duplicate particles are considered valid and are always within
/// the domain.
/// This kernel applies to single-GPU and multi-GPU because the calculations are made
/// from domposmin.
/// It controls the cell coordinates not exceed the maximum.
///
/// Duplica la posicion de la particula indicada aplicandole un desplazamiento.
/// Las particulas duplicadas se considera que siempre son validas y estan dentro
/// del dominio.
/// Este kernel vale para single-gpu y multi-gpu porque los calculos se hacen 
/// a partir de domposmin.
/// Se controla que las coordendas de celda no sobrepasen el maximo.
//------------------------------------------------------------------------------
__device__ void KerPeriodicDuplicatePos(unsigned pnew,unsigned pcopy
  ,bool inverse,double dx,double dy,double dz,uint3 cellmax
  ,double2 *posxy,double *posz,unsigned *dcell)
{
  //-Obtains position of the particle to be duplicated.
  //-Obtiene pos de particula a duplicar.
  double2 rxy=posxy[pcopy];
  double rz=posz[pcopy];
  //-Applies displacement.
  rxy.x+=(inverse? -dx: dx);
  rxy.y+=(inverse? -dy: dy);
  rz+=(inverse? -dz: dz);
  //-Computes cell coordinates within the domain.
  //-Calcula coordendas de celda dentro de dominio.
  unsigned cx=unsigned((rxy.x-CTE.domposminx)/CTE.scell);
  unsigned cy=unsigned((rxy.y-CTE.domposminy)/CTE.scell);
  unsigned cz=unsigned((rz-CTE.domposminz)/CTE.scell);
  //-Adjust cell coordinates if they exceed the maximum.
  //-Ajusta las coordendas de celda si sobrepasan el maximo.
  cx=(cx<=cellmax.x? cx: cellmax.x);
  cy=(cy<=cellmax.y? cy: cellmax.y);
  cz=(cz<=cellmax.z? cz: cellmax.z);
  //-Stores position and cell of the new particles.
  //-Graba posicion y celda de nuevas particulas.
  posxy[pnew]=rxy;
  posz[pnew]=rz;
  dcell[pnew]=PC__Cell(CTE.cellcode,cx,cy,cz);
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
__global__ void KerPeriodicDuplicateVerlet(unsigned n,unsigned pini,uint3 cellmax,double3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,float4 *velrhopm1)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    velrhopm1[pnew]=velrhopm1[pcopy];
    if(spstau)spstau[pnew]=spstau[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateVerlet(unsigned n,unsigned pini,tuint3 domcells,tdouble3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,float4 *velrhopm1)
{
  if(n){
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicDuplicateVerlet <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,velrhopm1);
  }
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
template<bool varspre> __global__ void KerPeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,uint3 cellmax,double3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    if(varspre){
      posxypre[pnew]=posxypre[pcopy];
      poszpre[pnew]=poszpre[pcopy];
      velrhoppre[pnew]=velrhoppre[pcopy];
    }
    if(spstau)spstau[pnew]=spstau[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,tuint3 domcells,tdouble3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  if(n){
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    if(posxypre!=NULL)KerPeriodicDuplicateSymplectic<true>  <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,posxypre,poszpre,velrhoppre);
    else              KerPeriodicDuplicateSymplectic<false> <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,posxypre,poszpre,velrhoppre);
  }
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
__global__ void KerPeriodicDuplicateNormals(unsigned n,unsigned pini,const unsigned *listp,float3 *normals,float3 *motionvel)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    normals[pnew]=normals[pcopy];
    if(motionvel)motionvel[pnew]=motionvel[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateNormals(unsigned n,unsigned pini,const unsigned *listp,float3 *normals,float3 *motionvel)
{
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicDuplicateNormals <<<sgrid,SPHBSIZE>>> (n,pini,listp,normals,motionvel);
  }
}


//##############################################################################
//# Kernels for Damping.
//##############################################################################
//------------------------------------------------------------------------------
/// Returns TRUE when code==NULL or particle is normal and fluid.
//------------------------------------------------------------------------------
__device__ bool KerIsNormalFluid(const typecode *code,unsigned p){
  if(code){//-Descarta particulas floating o periodicas.
    const typecode cod=code[p];
    return(CODE_IsNormal(cod) && CODE_IsFluid(cod));
  }
  return(true);
}
//------------------------------------------------------------------------------
/// Solves point on the plane.
/// Resuelve punto en el plano.
//------------------------------------------------------------------------------
__device__ double KerPointPlane(const double4 &pla,double px,double py,double pz)
{
  return(pla.x*px+pla.y*py+pla.z*pz+pla.w);
}
//------------------------------------------------------------------------------
/// Solves point on the plane.
/// Resuelve punto en el plano.
//------------------------------------------------------------------------------
__device__ double KerPointPlane(const double4 &pla,const double3 &pt)
{
  return(pla.x*pt.x+pla.y*pt.y+pla.z*pt.z+pla.w);
}

//------------------------------------------------------------------------------
/// Applies Damping.
/// Aplica Damping.
//------------------------------------------------------------------------------
__global__ void KerComputeDamping(unsigned n,unsigned pini
  ,double dt,double4 plane,float dist,float over,float3 factorxyz,float redumax
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      double vdis=KerPointPlane(plane,rposxy.x,rposxy.y,rposz);  //fgeo::PlanePoint(plane,ps);
      if(0<vdis && vdis<=dist+over){
        const double fdis=(vdis>=dist? 1.: vdis/dist);
        const double redudt=dt*(fdis*fdis)*redumax;
        double redudtx=(1.-redudt*factorxyz.x);
        double redudty=(1.-redudt*factorxyz.y);
        double redudtz=(1.-redudt*factorxyz.z);
        redudtx=(redudtx<0? 0.: redudtx);
        redudty=(redudty<0? 0.: redudty);
        redudtz=(redudtz<0? 0.: redudtz);
        float4 rvel=velrhop[p1];
        rvel.x=float(redudtx*rvel.x); 
        rvel.y=float(redudty*rvel.y); 
        rvel.z=float(redudtz*rvel.z); 
        velrhop[p1]=rvel;
      }
    }
  }
}
//==============================================================================
/// Applies Damping.
/// Aplica Damping.
//==============================================================================
void ComputeDamping(double dt,tdouble4 plane,float dist,float over,tfloat3 factorxyz,float redumax
  ,unsigned n,unsigned pini,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeDamping <<<sgridf,SPHBSIZE>>> (n,pini,dt,Double4(plane),dist,over,Float3(factorxyz),redumax
      ,posxy,posz,code,velrhop);
  }
}

//------------------------------------------------------------------------------
/// Applies Damping to limited domain.
/// Aplica Damping limitado a un dominio.
//------------------------------------------------------------------------------
__global__ void KerComputeDampingPla(unsigned n,unsigned pini
  ,double dt,double4 plane,float dist,float over,float3 factorxyz,float redumax
  ,double zmin,double zmax,double4 pla0,double4 pla1,double4 pla2,double4 pla3
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      const double3 ps=make_double3(rposxy.x,rposxy.y,rposz);
      double vdis=KerPointPlane(plane,ps);  //fgeo::PlanePoint(plane,ps);
      if(0<vdis && vdis<=dist+over){
        if(ps.z>=zmin && ps.z<=zmax && KerPointPlane(pla0,ps)<=0 && KerPointPlane(pla1,ps)<=0 && KerPointPlane(pla2,ps)<=0 && KerPointPlane(pla3,ps)<=0){
          const double fdis=(vdis>=dist? 1.: vdis/dist);
          const double redudt=dt*(fdis*fdis)*redumax;
          double redudtx=(1.-redudt*factorxyz.x);
          double redudty=(1.-redudt*factorxyz.y);
          double redudtz=(1.-redudt*factorxyz.z);
          redudtx=(redudtx<0? 0.: redudtx);
          redudty=(redudty<0? 0.: redudty);
          redudtz=(redudtz<0? 0.: redudtz);
          float4 rvel=velrhop[p1];
          rvel.x=float(redudtx*rvel.x); 
          rvel.y=float(redudty*rvel.y); 
          rvel.z=float(redudtz*rvel.z); 
          velrhop[p1]=rvel;
        }
      }
    }
  }
}
//==============================================================================
/// Applies Damping to limited domain.
/// Aplica Damping limitado a un dominio.
//==============================================================================
void ComputeDampingPla(double dt,tdouble4 plane,float dist,float over,tfloat3 factorxyz,float redumax
  ,double zmin,double zmax,tdouble4 pla0,tdouble4 pla1,tdouble4 pla2,tdouble4 pla3
  ,unsigned n,unsigned pini,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetSimpleGridSize(n,SPHBSIZE);
    KerComputeDampingPla <<<sgridf,SPHBSIZE>>> (n,pini,dt,Double4(plane),dist,over,Float3(factorxyz),redumax
      ,zmin,zmax,Double4(pla0),Double4(pla1),Double4(pla2),Double4(pla3)
      ,posxy,posz,code,velrhop);
  }
}


}


//##############################################################################
//# Kernels for InOut (JSphInOut) and BoundCorr (JSphBoundCorr).
//# Kernels para InOut (JSphInOut) y BoundCorr (JSphBoundCorr).
//##############################################################################
#include "JSphGpu_InOut_iker.cu"



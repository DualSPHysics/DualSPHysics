#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2023 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public License 
 as published by the Free Software Foundation; either version 2.1 of the License, or (at your option) any later version.
 
 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more details. 

 You should have received a copy of the GNU Lesser General Public License along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JDsGauge_ker.cu \brief Implements functions and CUDA kernels for classes JGauge.

#include "JDsGauge_ker.h"
#include "Functions.h"
#include "FunctionsCuda.h"
#include <float.h>
#include <hip/hip_math_constants.h>
//:#include "JDgKerPrint.h"
//:#include "JDgKerPrint_ker.h"
#include <cstdio>
#include <string>

//#include "TypesDef.h"
//#include <hip/hip_runtime_api.h>

namespace cugauge{
#include "FunctionsBasic_iker.h"
#include "FunSphKernel_iker.h"
#include "FunSphEos_iker.h"
#include "JCellSearch_iker.h"

//##############################################################################
//# Kernels for gauge interaction.
//##############################################################################
//------------------------------------------------------------------------------
/// Performs interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// It includes visco artificial/laminar and floatings SPH/DEM.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings SPH/DEM.
//------------------------------------------------------------------------------
template<TpKernel tker> __global__ void KerInteractionGaugeVel(float aker
  ,double3 ptpos,int scelldiv,int4 nc,int3 cellzero,const int2* beginendcellfluid
  ,unsigned axis,unsigned cellcode,double3 domposmin,float scell,float poscellsize
  ,float kernelsize2,float kernelh,float massf
  ,const double2* posxy,const double* posz,const typecode* code,const float4* velrhop
  ,float3* ptvel)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle
  if(!p){
    const double px=ptpos.x;
    const double py=ptpos.y;
    const double pz=ptpos.z;

    double sumwab=0;
    double3 sumvel=make_double3(0,0,0);

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::Initsp(px,py,pz,axis,domposmin,scell,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    //ini3+=cellfluid; fin3+=cellfluid; //cellfluid is included in *beginendcellfluid.
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
      if(pfin)for(int p2=pini;p2<pfin;p2++){
        const double2 pxyp2=posxy[p2];
        const float drx=float(px-pxyp2.x);
        const float dry=float(py-pxyp2.y);
        const float drz=float(pz-posz[p2]);
        const float rr2=(drx*drx + dry*dry + drz*drz);
        //-Interaction with real neighboring fluid particles.
        if(rr2<=kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){
          float wab=cufsph::GetKernel_Wab<tker>(rr2,kernelh,aker);
          const float4 velrhopp2=velrhop[p2];
          wab*=massf/velrhopp2.w;
          sumwab+=wab;
          sumvel.x+=wab*velrhopp2.x;
          sumvel.y+=wab*velrhopp2.y;
          sumvel.z+=wab*velrhopp2.z;

        }
      }
    }
    //-Applies kernel correction.
    //if(sumwab){
    //  sumvel.x/=sumwab;
    //  sumvel.y/=sumwab;
    //  sumvel.z/=sumwab;
    //}
    //-Stores result. | Guarda resultado.
    ptvel[0]=make_float3(float(sumvel.x),float(sumvel.y),float(sumvel.z));
  }
}
//==============================================================================
/// Calculates velocity in indicated point.
//==============================================================================
void Interaction_GaugeVel(const StCteSph& CSP,const StDivDataGpu& dvd
  ,tdouble3 ptpos,const double2* posxy,const double* posz,const typecode* code
  ,const float4* velrhop,float3* ptvel)
  //,tdouble3 domposmin,float scell,float kernelsize2,float h,float massf)
{
  //-Interaction Fluid-Fluid & Fluid-Bound.
  const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
  const unsigned bsize=32;
  dim3 sgrid=GetSimpleGridSize(1,bsize);
  //:JDgKerPrint info;
  //:byte* ik=NULL; //info.GetInfoPointer(sgridf,bsfluid);
  switch(CSP.tkernel){
    case KERNEL_Cubic:   //Kernel Cubic is not available.
    case KERNEL_Wendland:{ const float aker=CSP.kwend.awen;
      KerInteractionGaugeVel<KERNEL_Wendland> <<<sgrid,bsize>>> (aker,Double3(ptpos)
        ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
        ,dvd.axis,dvd.domcellcode,dvd.domposmin,dvd.scell,dvd.poscellsize
        ,dvd.kernelsize2,CSP.kernelh,CSP.massfluid,posxy,posz,code,velrhop,ptvel);
    }break;
    default: throw "Kernel unknown at Interaction_GaugeVel().";
  }
  //:info.PrintValuesFull(true); //info.PrintValuesInfo();
}

//------------------------------------------------------------------------------
/// Calculates mass value at one point by interacting with the fluid.
/// Calcula valor de masa en un punto mediante la interaccion con el fluido.
//------------------------------------------------------------------------------
template<TpKernel tker> __device__ float KerCalculeMass(float aker
  ,double px,double py,double pz,float kernelsize2,float kernelh,float massf
  ,int scelldiv,int4 nc,int3 cellzero,const int2* beginendcellfluid
  ,unsigned axis,unsigned cellcode,double3 domposmin,float scell,float poscellsize
  ,const double2* posxy,const double* posz,const typecode* code
  ,const float4* velrhop)
{
  double summass=0;

  //-Obtains neighborhood search limits.
  int ini1,fin1,ini2,fin2,ini3,fin3;
  cunsearch::Initsp(px,py,pz,axis,domposmin,scell,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

  //-Interaction with fluids.
  for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
    unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
    if(pfin)for(int p2=pini;p2<pfin;p2++){
      const double2 pxyp2=posxy[p2];
      const float drx=float(px-pxyp2.x);
      const float dry=float(py-pxyp2.y);
      const float drz=float(pz-posz[p2]);
      const float rr2=(drx*drx + dry*dry + drz*drz);
      if(rr2<=kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){
        float wab=cufsph::GetKernel_Wab<tker>(rr2,kernelh,aker);
        wab*=massf/velrhop[p2].w;
        summass+=wab*massf;
      }
    }
  }
  return(float(summass));
}

//------------------------------------------------------------------------------
/// Calculates surface water level at indicated line.
//------------------------------------------------------------------------------
template<TpKernel tker> __global__ void KerInteractionGaugeSwl(float aker
  ,double p0x,double p0y,double p0z,double pdirx,double pdiry,double pdirz
  ,unsigned pointnp,float masslimit,float kernelsize2,float kernelh,float massf
  ,int scelldiv,int4 nc,int3 cellzero,const int2* beginendcellfluid
  ,unsigned axis,unsigned cellcode,double3 domposmin,float scell,float poscellsize
  ,const double2* posxy,const double* posz,const typecode* code
  ,const float4* velrhop,float3* ptres)
{
  extern __shared__ float shmass[];
  const unsigned tid=threadIdx.x;
  unsigned cpbase=0;
  float psurfx=FLT_MAX,psurfy=FLT_MAX,psurfz=FLT_MAX;
  float mpre=0;
  while(cpbase<=pointnp){
    //-Saves mass values in shared memory.
    const unsigned cp=cpbase+tid;
    if(cp<=pointnp){
      shmass[tid]=KerCalculeMass<tker>(aker,p0x+pdirx*cp,p0y+pdiry*cp,p0z+pdirz*cp
        ,kernelsize2,kernelh,massf,scelldiv,nc,cellzero,beginendcellfluid
        ,axis,cellcode,domposmin,scell,poscellsize,posxy,posz,code,velrhop);
    }
    else shmass[tid]=0;
    __syncthreads();
    //-Checks mass values.
    if(!tid){
      for(unsigned c=0;c<blockDim.x;c++){
        const float mass=shmass[c];
        if(mass>masslimit)mpre=mass;
        if(mass<masslimit && mpre){
          const float fxm1=((masslimit-mpre)/(mass-mpre)-1)+float(cpbase+c);
          psurfx=p0x+pdirx*fxm1;
          psurfy=p0y+pdiry*fxm1;
          psurfz=p0z+pdirz*fxm1;
          shmass[0]=FLT_MAX;
          break;
        }
      }
    }
    __syncthreads();
    if(shmass[0]==FLT_MAX)break;
    cpbase+=blockDim.x;
  }
  //-Stores result.
  if(!tid){
    if(psurfx==FLT_MAX){
      const unsigned cp=(mpre? pointnp: 0);
      psurfx=p0x+(pdirx*cp);
      psurfy=p0y+(pdiry*cp);
      psurfz=p0z+(pdirz*cp);
    }
    ptres[0]=make_float3(psurfx,psurfy,psurfz);
  }
}
//==============================================================================
/// Calculates surface water level at indicated line.
//==============================================================================
void Interaction_GaugeSwl(const StCteSph& CSP,const StDivDataGpu& dvd
  ,tdouble3 point0,tdouble3 pointdir,unsigned pointnp,float masslimit
  ,const double2* posxy,const double* posz,const typecode* code
  ,const float4* velrhop,float3* ptres)
{
  const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
  const unsigned bsize=128;
  dim3 sgrid=GetSimpleGridSize(bsize,bsize);
  const unsigned smem=sizeof(float)*(bsize+1);
  switch(CSP.tkernel){
    case KERNEL_Cubic:   //Kernel Cubic is not available.
    case KERNEL_Wendland:{ const float aker=CSP.kwend.awen;
      KerInteractionGaugeSwl<KERNEL_Wendland> <<<sgrid,bsize,smem>>> (aker
        ,point0.x,point0.y,point0.z,pointdir.x,pointdir.y,pointdir.z
        ,pointnp,masslimit,dvd.kernelsize2,CSP.kernelh,CSP.massfluid
        ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
        ,dvd.axis,dvd.domcellcode,dvd.domposmin,dvd.scell,dvd.poscellsize
        ,posxy,posz,code,velrhop,ptres);
    }break;
    default: throw "Kernel unknown at Interaction_GaugeSwl().";
  }
}


//------------------------------------------------------------------------------
/// Calculates maximum z of fluid at distance of a vertical line.
//------------------------------------------------------------------------------
__global__ void KerInteractionGaugeMaxz(double p0x,double p0y,float maxdist2
  ,int cxini,int cxfin,int yini,int yfin,int zini,int zfin
  ,int4 nc,unsigned cellfluid,const int2* begincell
  ,const double2* posxy,const double* posz,const typecode* code
  ,float3* ptres)
{
  if(threadIdx.x==0){
    unsigned pmax=UINT_MAX;
    float zmax=-FLT_MAX;
    //-Interaction with fluid particles. | Interaccion con fluidas.
    for(int z=zfin-1;z>=zini && pmax==UINT_MAX;z--){
      int zmod=(nc.w)*z+cellfluid; //-Sum from start of fluid cells. | Le suma donde empiezan las celdas de fluido.
      for(int y=yini;y<yfin;y++){
        int ymod=zmod+nc.x*y;
        unsigned pini,pfin=0;
        for(int x=cxini;x<cxfin;x++){
          int2 cbeg=begincell[x+ymod];
          if(cbeg.y){
            if(!pfin)pini=cbeg.x;
            pfin=cbeg.y;
          }
        }
        if(pfin)for(unsigned p2=pini;p2<pfin;p2++){
          const double posz2=posz[p2];
          if(posz2>zmax){
            const double2 posxy2=posxy[p2];
            const float drx=float(p0x-posxy2.x);
            const float dry=float(p0y-posxy2.y);
            const float rr2=drx*drx+dry*dry;
            if(rr2<=maxdist2 && CODE_IsFluid(code[p2])){//-Only with fluid particles.
              zmax=float(posz2);
              pmax=p2;
            }
          }
        }
      }
    }
    //-Stores result.
    ptres[0]=make_float3(0,0,zmax);
  }
}
//==============================================================================
/// Calculates maximum z of fluid at distance of a vertical line.
//==============================================================================
void Interaction_GaugeMaxz(tdouble3 point0,float maxdist2,const StDivDataGpu& dvd
  ,int cxini,int cxfin,int yini,int yfin,int zini,int zfin
  ,const double2* posxy,const double* posz,const typecode* code
  ,float3* ptres)
{
  const unsigned bsize=128;
  dim3 sgrid=GetSimpleGridSize(1,bsize);
  KerInteractionGaugeMaxz <<<sgrid,bsize>>> (point0.x,point0.y,maxdist2
    ,cxini,cxfin,yini,yfin,zini,zfin,dvd.nc,dvd.cellfluid,dvd.beginendcell
    ,posxy,posz,code,ptres);
}



//<vs_meeshdat_ini>
//------------------------------------------------------------------------------
/// Calculates data interpolation according to fluid particle data.
//------------------------------------------------------------------------------
template<TpKernel tker,bool tvel> __global__ void KerComputeGaugeMesh(unsigned nptot
  ,unsigned npt12,unsigned npt1,double3 ptref
  ,double3 vdp1,double3 vdp2,double3 vdp3,float3 vdir
  ,int scelldiv,int4 nc,int3 cellzero,const int2* beginendcellfluid
  ,unsigned axis,unsigned cellcode,double3 domposmin,float scell,float poscellsize
  ,float aker,float kernelsize2,float kernelh,float massf,float kclimit,float kcdummy
  ,const double2* posxy,const double* posz,const typecode* code
  ,const float4* velrhop,float3* ptvel,float* ptvdir,float* ptrhop
  ,float* ptmass)
{
  unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of position.
  if(cp<nptot){
    const unsigned cp3=cp/npt12;
    const unsigned cp3r=cp-cp3*npt12;
    const unsigned cp2=cp3r/npt1;
    const unsigned cp1=cp3r-cp2*npt1;
    const double px=ptref.x+(vdp1.x*cp1)+(vdp2.x*cp2)+(vdp3.x*cp3);
    const double py=ptref.y+(vdp1.y*cp1)+(vdp2.y*cp2)+(vdp3.y*cp3);
    const double pz=ptref.z+(vdp1.z*cp1)+(vdp2.z*cp2)+(vdp3.z*cp3);
    float sumwab=0;
    float summass=0;
    float sumrhop=0;
    float3 sumvel=make_float3(0,0,0);

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::Initsp(px,py,pz,axis,domposmin,scell,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    //ini3+=cellfluid; fin3+=cellfluid; //cellfluid is included in *beginendcellfluid.
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
      if(pfin)for(int p2=pini;p2<pfin;p2++){
        const double2 pxyp2=posxy[p2];
        const float drx=float(px-pxyp2.x);
        const float dry=float(py-pxyp2.y);
        const float drz=float(pz-posz[p2]);
        const float rr2=(drx*drx + dry*dry + drz*drz);
        //-Interaction with real neighboring fluid particles.
        if(rr2<=kernelsize2 && CODE_IsFluid(code[p2])){
          float wab=cufsph::GetKernel_Wab<tker>(rr2,kernelh,aker);
          const float4 velrhopp2=velrhop[p2];
          wab*=massf/velrhopp2.w;
          sumwab+=wab;
          summass+=wab*massf;
          sumrhop+=wab*velrhopp2.w;
          if(tvel){
            sumvel.x+=wab*velrhopp2.x;
            sumvel.y+=wab*velrhopp2.y;
            sumvel.z+=wab*velrhopp2.z;
          }
        }
      }
    }
    //-Applies kernel correction.
    if(kclimit!=FLT_MAX){
      if(sumwab>=kclimit){
        sumvel.x/=sumwab;
        sumvel.y/=sumwab;
        sumvel.z/=sumwab;
        sumrhop/=sumwab;
      }
      else if(kcdummy!=FLT_MAX){
        sumvel=make_float3(kcdummy,kcdummy,kcdummy);
        sumrhop=kcdummy;
      }
    }
    //-Stores results.
    if(tvel){
      if(ptvel )ptvel [cp]=sumvel;
      if(ptvdir)ptvdir[cp]=(sumvel.x*vdir.x + sumvel.y*vdir.y + sumvel.z*vdir.z);
    }
    if(ptrhop)ptrhop[cp]=sumrhop;
    if(ptmass)ptmass[cp]=summass;
  }
}

//==============================================================================
/// Calculates data interpolation according to fluid particle data.
//==============================================================================
template<TpKernel tker,bool tvel> void ComputeGaugeMeshT(float aker
  ,const StCteSph& CSP,const StDivDataGpu& dvd,const jmsh::StMeshPts& mp
  ,float kclimit,float kcdummy,const double2* posxy,const double* posz
  ,const typecode* code,const float4* velrhop,float3* ptvel,float* ptvdir
  ,float* ptrhop,float* ptmass)
{
  //-Interaction with fluid particles.
  const unsigned npt12=mp.npt1*mp.npt2;
  const unsigned nptot=npt12*mp.npt3;
  if(nptot){
    const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
    const unsigned bsize=128;
    dim3 sgrid=GetSimpleGridSize(nptot,bsize);
    //:JDgKerPrint info;
    //:byte* ik=NULL; //info.GetInfoPointer(sgridf,bsfluid);
    KerComputeGaugeMesh<tker,tvel> <<<sgrid,bsize>>> (nptot,npt12,mp.npt1
     ,Double3(mp.ptref),Double3(mp.vdp1),Double3(mp.vdp2),Double3(mp.vdp3),Float3(mp.dirdat)
     ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
     ,dvd.axis,dvd.domcellcode,dvd.domposmin,dvd.scell,dvd.poscellsize
     ,aker,dvd.kernelsize2,CSP.kernelh,CSP.massfluid,kclimit,kcdummy
     ,posxy,posz,code,velrhop,ptvel,ptvdir,ptrhop,ptmass);
    //:info.PrintValuesFull(true); //info.PrintValuesInfo();
  }
}

//==============================================================================
/// Calculates velocity and/or mass interpolation according to fluid particle data.
//==============================================================================
void ComputeGaugeMesh(const StCteSph& CSP,const StDivDataGpu& dvd
  ,const jmsh::StMeshPts& mp,float kclimit,float kcdummy,const double2* posxy
  ,const double* posz,const typecode* code,const float4* velrhop
  ,float3* ptvel,float* ptvdir,float* ptrhop,float* ptmass)
{
  const bool tvel=(ptvel!=NULL || ptvdir!=NULL);
  switch(CSP.tkernel){
    case KERNEL_Cubic:   //Kernel Cubic is not available.
    case KERNEL_Wendland:{ const float aker=CSP.kwend.awen;
      if(tvel)ComputeGaugeMeshT<KERNEL_Wendland,true >(aker,CSP,dvd,mp,kclimit,kcdummy,posxy,posz,code,velrhop,ptvel,ptvdir,ptrhop,ptmass);
      else    ComputeGaugeMeshT<KERNEL_Wendland,false>(aker,CSP,dvd,mp,kclimit,kcdummy,posxy,posz,code,velrhop,ptvel,ptvdir,ptrhop,ptmass);
    }break;
    default: throw "Kernel unknown at ComputeGaugeMesh().";
  }
}

//------------------------------------------------------------------------------
/// Calculates Z surface water level according mass interpolation values on fluid.
//------------------------------------------------------------------------------
__global__ void KerComputeGaugeMeshZsurf(unsigned npt12,unsigned npt1,unsigned npt3
  ,float masslimit,double ptrefz,double vdp1z,double vdp2z,double vdp3z
  ,const float* ptmass,float* ptzsurf)
{
  const unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of position.
  if(cp<npt12){
    const unsigned cp2=cp/npt1;
    const unsigned cp1=cp-cp2*npt1;
    float masspre=0;
    unsigned cpsurf=0;
    float    fsurf=0;
    unsigned cpp=cp;
    for(unsigned cp3=0;cp3<npt3 && !cpsurf;cp3++){
      const float mass=ptmass[cpp];
      if(mass>masslimit)masspre=mass;
      if(mass<masslimit && masspre){
        fsurf=(masslimit-masspre)/(mass-masspre);
        cpsurf=cp3;
      }
      cpp+=npt12;
    }
    float zsurf=float(ptrefz+(vdp1z*cp1)+(vdp2z*cp2));    //-Minimum zsurf.
    if(cpsurf==0 && masspre)zsurf+=float(vdp3z*(npt3-1)); //-Maximum zsurf.
    if(cpsurf){
      zsurf+=float((vdp3z*(cpsurf-1))+(vdp3z*fsurf));     //-Found zsurf.
    }
    ptzsurf[cp]=zsurf;
  }
}

//==============================================================================
/// Calculates Z surface water level according mass interpolation values on fluid.
//==============================================================================
void ComputeGaugeMeshZsurf(float masslimit,const jmsh::StMeshPts& mp
  ,const float* ptmass,float* ptzsurf)
{
  //-Interaction with fluid particles.
  const unsigned npt12=mp.npt1*mp.npt2;
  if(npt12){
    const unsigned bsize=128;
    dim3 sgrid=GetSimpleGridSize(npt12,bsize);
    //:JDgKerPrint info;
    //:byte* ik=NULL; //info.GetInfoPointer(sgridf,bsfluid);
    hipMemset(ptzsurf,0,sizeof(float)*npt12);
    KerComputeGaugeMeshZsurf <<<sgrid,bsize>>> (npt12,mp.npt1,mp.npt3
      ,masslimit,mp.ptref.z,mp.vdp1.z,mp.vdp2.z,mp.vdp3.z,ptmass,ptzsurf);
    //:info.PrintValuesFull(true); //info.PrintValuesInfo();
  }
}
//<vs_meeshdat_end>


//<vs_flowdat_ini>
//------------------------------------------------------------------------------
/// Calculates data interpolation according to fluid particle data.
//------------------------------------------------------------------------------
template<TpKernel tker,bool tvel> __global__ void KerComputeGaugeFlow(unsigned nptot
  ,unsigned npt12,unsigned npt1,double3 ptref
  ,double3 vdp1,double3 vdp2,double3 vdp3,float3 vdir
  ,int scelldiv,int4 nc,int3 cellzero,const int2* beginendcellfluid
  ,unsigned axis,unsigned cellcode,double3 domposmin,float scell,float poscellsize
  ,float aker,float kernelsize2,float kernelh,float massf,float kclimit
  ,const double2* posxy,const double* posz,const typecode* code
  ,const float4* velrhop,float* ptvdir)
{
  unsigned cp=blockIdx.x*blockDim.x + threadIdx.x; //-Number of position.
  if(cp<nptot){
    const unsigned cp3=cp/npt12;
    const unsigned cp3r=cp-cp3*npt12;
    const unsigned cp2=cp3r/npt1;
    const unsigned cp1=cp3r-cp2*npt1;
    const double px=ptref.x+(vdp1.x*cp1)+(vdp2.x*cp2)+(vdp3.x*cp3);
    const double py=ptref.y+(vdp1.y*cp1)+(vdp2.y*cp2)+(vdp3.y*cp3);
    const double pz=ptref.z+(vdp1.z*cp1)+(vdp2.z*cp2)+(vdp3.z*cp3);
    float sumwab=0;
    float3 sumvel=make_float3(0,0,0);

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::Initsp(px,py,pz,axis,domposmin,scell,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    //ini3+=cellfluid; fin3+=cellfluid; //cellfluid is included in *beginendcellfluid.
    for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
      if(pfin)for(int p2=pini;p2<pfin;p2++){
        const double2 pxyp2=posxy[p2];
        const float drx=float(px-pxyp2.x);
        const float dry=float(py-pxyp2.y);
        const float drz=float(pz-posz[p2]);
        const float rr2=(drx*drx + dry*dry + drz*drz);
        //-Interaction with real neighboring fluid particles.
        if(rr2<=kernelsize2 && CODE_IsFluid(code[p2])){
          float wab=cufsph::GetKernel_Wab<tker>(rr2,kernelh,aker);
          const float4 velrhopp2=velrhop[p2];
          wab*=massf/velrhopp2.w;
          sumwab+=wab;
          if(tvel){
            sumvel.x+=wab*velrhopp2.x;
            sumvel.y+=wab*velrhopp2.y;
            sumvel.z+=wab*velrhopp2.z;
          }
        }
      }
    }
    //-Applies kernel correction.
    if(kclimit!=FLT_MAX){
      if(sumwab>=kclimit){
        sumvel.x/=sumwab;
        sumvel.y/=sumwab;
        sumvel.z/=sumwab;        
      }
      else {
        sumvel=make_float3(0,0,0);        
      }
    }
    //-Stores results.
    if(ptvdir)ptvdir[cp]=(sumvel.x*vdir.x + sumvel.y*vdir.y + sumvel.z*vdir.z);
  }
}

//==============================================================================
/// Calculates data interpolation according to fluid particle data.
//==============================================================================
template<TpKernel tker,bool tvel> void ComputeGaugeFlowT(float aker
  ,const StCteSph& CSP,const StDivDataGpu& dvd,const jmsh::StMeshPts& mp
  ,float kclimit,const double2* posxy,const double* posz
  ,const typecode* code,const float4* velrhop,float* ptvdir)
{
  //-Interaction with fluid particles.
  const unsigned npt12=mp.npt1*mp.npt2;
  const unsigned nptot=npt12*mp.npt3;
  //const unsigned nptot=mp.npt1*mp.npt2;
  if(nptot){
    const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
    const unsigned bsize=128;
    dim3 sgrid=GetSimpleGridSize(nptot,bsize);
    //:JDgKerPrint info;
    //:byte* ik=NULL; //info.GetInfoPointer(sgridf,bsfluid);
    KerComputeGaugeFlow<tker,tvel> <<<sgrid,bsize>>> (nptot,npt12,mp.npt1
     ,Double3(mp.ptref),Double3(mp.vdp1),Double3(mp.vdp2),Double3(mp.vdp3),Float3(mp.dirdat)
     ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
     ,dvd.axis,dvd.domcellcode,dvd.domposmin,dvd.scell,dvd.poscellsize
     ,aker,dvd.kernelsize2,CSP.kernelh,CSP.massfluid,kclimit
     ,posxy,posz,code,velrhop,ptvdir);
    //:info.PrintValuesFull(true); //info.PrintValuesInfo();
  }
}

//==============================================================================
/// Calculates velocity according to fluid particle data.
//==============================================================================
void ComputeGaugeFlow(const StCteSph& CSP,const StDivDataGpu& dvd
  ,const jmsh::StMeshPts& mp,float kclimit,const double2* posxy
  ,const double* posz,const typecode* code,const float4* velrhop
  ,float* ptvdir)
{  
  switch(CSP.tkernel){
    case KERNEL_Cubic:   //Kernel Cubic is not available.
    case KERNEL_Wendland:{ const float aker=CSP.kwend.awen;
      ComputeGaugeFlowT<KERNEL_Wendland,true >(aker,CSP,dvd,mp,kclimit,posxy,posz,code,velrhop,ptvdir);
    }break;
    default: throw "Kernel unknown at ComputeGaugeMesh().";
  }
}
//<vs_flowdat_end>


//------------------------------------------------------------------------------
/// Calculates force on selected fixed or moving particles using only fluid particles.
/// Ignores periodic boundary particles to avoid race condition problems.
//------------------------------------------------------------------------------
template<TpKernel tker> __global__ void KerInteractionGaugeForce(float bhker
  ,unsigned n,unsigned idbegin,typecode codesel
  ,int scelldiv,int4 nc,int3 cellzero,const int2* beginendcellfluid
  ,unsigned axis,unsigned cellcode,double3 domposmin,float scell,float poscellsize
  ,float kernelsize2,float kernelh,float massf,float cteb,float rhopzero,float gamma,float cs0
  ,const double2* posxy,const double* posz,const typecode* code
  ,const unsigned* idp,const float4* velrhop,float3* partace)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const typecode code1=code[p];
    if(CODE_GetTypeAndValue(code1)==codesel && CODE_IsNormal(code1)){
      const double2 ptposxy=posxy[p];
      const double px=ptposxy.x;
      const double py=ptposxy.y;
      const double pz=posz[p];
      const float rhop1=velrhop[p].w;
      const float press1=cufsph::ComputePress(rhop1,rhopzero,cteb,gamma,cs0);
      float3 ace=make_float3(0,0,0);

      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::Initsp(px,py,pz,axis,domposmin,scell,scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);
   
      //-Interaction with fluids.
      for(int c3=ini3;c3<fin3;c3+=nc.w)for(int c2=ini2;c2<fin2;c2+=nc.x){
        unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
        if(pfin)for(int p2=pini;p2<pfin;p2++){
          double2 pxyp2=posxy[p2];
          const float drx=float(px-pxyp2.x);
          const float dry=float(py-pxyp2.y);
          const float drz=float(pz-posz[p2]);
          const float rr2=(drx*drx + dry*dry + drz*drz);
          //-Interaction with real neighboring fluid particles.
          if(rr2<=kernelsize2 && rr2>=ALMOSTZERO && CODE_IsFluid(code[p2])){
            const float fac=cufsph::GetKernel_Fac<tker>(rr2,kernelh,bhker);
            const float frx=fac*drx;
            const float fry=fac*dry;
            const float frz=fac*drz;

            //-Velocity derivative (Momentum equation).
            const float mass2=massf;
            const float rhop2=velrhop[p2].w;
            const float press2=cufsph::ComputePress(rhop2,rhopzero,cteb,gamma,cs0);
            const float prs=(press1+press2)/(rhop1*rhop2);
            {//-Adds aceleration.
              const float p_vpm1=-prs*mass2;
              ace.x+=p_vpm1*frx;  ace.y+=p_vpm1*fry;  ace.z+=p_vpm1*frz;
            }
          }
        }
      }
      //-Saves ace.
      partace[idp[p]-idbegin]=ace;
    }
  }
}

//==============================================================================
/// Calculates force on selected fixed or moving particles using only fluid particles.
/// Ignores periodic boundary particles to avoid race condition problems.
//==============================================================================
void Interaction_GaugeForce(const StCteSph& CSP,const StDivDataGpu& dvd
  ,unsigned n,unsigned idbegin,typecode codesel,const double2* posxy
  ,const double* posz,const typecode* code,const unsigned* idp
  ,const float4* velrhop,float3* partace)
{
  //const float ovrhopzero=1.f/rhopzero;
  //-Interaction bound-Fluid.
  if(n){
    const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
    const unsigned bsize=128;
    dim3 sgrid=GetSimpleGridSize(n,bsize);
    switch(CSP.tkernel){
      case KERNEL_Cubic:   //Kernel Cubic is not available.
      case KERNEL_Wendland:{ const float bhker=CSP.kwend.bwenh;
        KerInteractionGaugeForce<KERNEL_Wendland> <<<sgrid,bsize>>>(bhker,n,idbegin,codesel
          ,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid
          ,dvd.axis,dvd.domcellcode,dvd.domposmin,dvd.scell,dvd.poscellsize
          ,dvd.kernelsize2,CSP.kernelh,CSP.massfluid,CSP.cteb,CSP.rhopzero,CSP.gamma,float(CSP.cs0)
          ,posxy,posz,code,idp,velrhop,partace);
      }break;
      default: throw "Kernel unknown at Interaction_GaugeForce().";
    }
  }
}


}


